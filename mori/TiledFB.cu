#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "mori/TiledFB.h"
#include "owl/owl.h"

namespace mori {

  TiledFB::SP TiledFB::create(DeviceContext *device)
  {
    return std::make_shared<TiledFB>(device);
  }
  
  TiledFB::TiledFB(DeviceContext *device)
    : device(device)
  {}

  __global__ void setTileCoords(TileDesc *tileDescs,
                                int numActiveTiles,
                                vec2i numTiles,
                                int tileIndexOffset,
                                int tileIndexScale)
  {
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if (tid >= numActiveTiles)
      return;
    
    int tileID = tid * tileIndexScale + tileIndexOffset;

    int tile_x = tileID % numTiles.x;
    int tile_y = tileID / numTiles.x;
    tileDescs[tid].lower = vec2i(tile_x*tileSize,tile_y*tileSize);
  }
  
  void TiledFB::resize(vec2i newSize)
  {
    SetActiveGPU forDuration(device);
    if (accumTiles)  {
      MORI_CUDA_CALL(Free(accumTiles));
      // MORI_CUDA_CALL(FreeAsync(accumTiles,device->stream));
      accumTiles = nullptr;
    }
    if (finalTiles) {
      MORI_CUDA_CALL(Free(finalTiles));
      // MORI_CUDA_CALL(FreeAsync(finalTiles,device->stream));
      finalTiles = nullptr;
    }
    if (tileDescs) {
      MORI_CUDA_CALL(Free(tileDescs));
      // MORI_CUDA_CALL(FreeAsync(tileDescs,device->stream));
      tileDescs = nullptr;
    }
    
    numPixels = newSize;
    numTiles  = divRoundUp(numPixels,vec2i(tileSize));
    numActiveTiles
      = divRoundUp(numTiles.x*numTiles.y - device->tileIndexOffset,
                   device->tileIndexScale);
    MORI_CUDA_CALL(Malloc(&accumTiles, numActiveTiles * sizeof(AccumTile)));
    MORI_CUDA_CALL(Malloc(&finalTiles, numActiveTiles * sizeof(FinalTile)));
    MORI_CUDA_CALL(MallocManaged(&tileDescs, numActiveTiles * sizeof(TileDesc)));
    // MORI_CUDA_CALL(MallocAsync(&accumTiles, numActiveTiles * sizeof(AccumTile),
    //                            device->stream));
    // MORI_CUDA_CALL(MallocAsync(&finalTiles, numActiveTiles * sizeof(FinalTile),
    //                            device->stream));
    // MORI_CUDA_CALL(MallocAsync(&tileDescs, numActiveTiles * sizeof(TileDesc),
    //                            device->stream));
    MORI_CUDA_SYNC_CHECK();
    if (numActiveTiles)
      setTileCoords<<<divRoundUp(numActiveTiles,1024),1024,0,device->stream>>>
        (tileDescs,numActiveTiles,numTiles,
         device->tileIndexOffset,device->tileIndexScale);
    MORI_CUDA_SYNC_CHECK();
  }

  __global__ void g_finalizeTiles(FinalTile *finalTiles,
                                  AccumTile *accumTiles)
  {
    int pixelID = threadIdx.x+tileSize*threadIdx.y;
    int tileID = blockIdx.x;
    finalTiles[tileID].rgba[pixelID]
      = owl::make_rgba(vec4f(accumTiles[tileID].accum[pixelID]));
  }

  __global__ void g_writeFinalPixels(uint32_t  *finalFB,
                                     vec2i      numPixels,
                                     FinalTile *finalTiles,
                                     TileDesc  *tileDescs)
  {
    int tileID = blockIdx.x;
    int ix = threadIdx.x + tileDescs[tileID].lower.x;
    int iy = threadIdx.y + tileDescs[tileID].lower.y;
    if (ix >= numPixels.x) return;
    if (iy >= numPixels.y) return;
    finalFB[ix + numPixels.x*iy]
      = finalTiles[tileID].rgba[threadIdx.x+threadIdx.y*tileSize];
  }
                                 
  void TiledFB::writeFinalPixels(uint32_t  *finalFB,
                                 vec2i      numPixels,
                                 FinalTile *finalTiles,
                                 TileDesc  *tileDescs,
                                 int        numTiles,
                                 hipStream_t stream)
  {
    g_writeFinalPixels
      <<<numTiles,vec2i(tileSize),0,stream>>>
      (finalFB,numPixels,
       finalTiles,tileDescs);
  }
  
  /*! write this tiledFB's tiles into given "final" frame buffer
    (i.e., a plain 2D array of numPixels.x*numPixels.y RGBA8
    pixels) */
  void TiledFB::finalizeTiles()
  {
    SetActiveGPU forDuration(device);
    g_finalizeTiles<<<numActiveTiles,vec2i(tileSize),0,device->stream>>>
      (finalTiles,accumTiles);
  }
}
