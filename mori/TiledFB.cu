#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "mori/TiledFB.h"
#include "owl/owl.h"

namespace mori {

  TiledFB::SP TiledFB::create(DeviceContext *device)
  {
    return std::make_shared<TiledFB>(device);
  }
  
  TiledFB::TiledFB(DeviceContext *device)
    : device(device)
  {}

  __global__ void setTileCoords(TileDesc *tileDescs,
                                int numActiveTiles,
                                vec2i numTiles,
                                int tileIndexOffset,
                                int tileIndexScale)
  {
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if (tid >= numActiveTiles)
      return;
    
    int tileID = tid * tileIndexScale + tileIndexOffset;

    int tile_x = tileID % numTiles.x;
    int tile_y = tileID / numTiles.x;
    tileDescs[tid].lower = vec2i(tile_x*tileSize,tile_y*tileSize);
  }
  
  void TiledFB::resize(vec2i newSize)
  {
    SetActiveGPU forDuration(device);
    if (accumTiles)  {
      MORI_CUDA_CALL(Free(accumTiles));
      // MORI_CUDA_CALL(FreeAsync(accumTiles,device->stream));
      accumTiles = nullptr;
    }
    if (finalTiles) {
      MORI_CUDA_CALL(Free(finalTiles));
      // MORI_CUDA_CALL(FreeAsync(finalTiles,device->stream));
      finalTiles = nullptr;
    }
    if (tileDescs) {
      MORI_CUDA_CALL(Free(tileDescs));
      // MORI_CUDA_CALL(FreeAsync(tileDescs,device->stream));
      tileDescs = nullptr;
    }
    
    numPixels = newSize;
    numTiles  = divRoundUp(numPixels,vec2i(tileSize));
    numActiveTiles
      = divRoundUp(numTiles.x*numTiles.y - device->tileIndexOffset,
                   device->tileIndexScale);
    MORI_CUDA_CALL(Malloc(&accumTiles, numActiveTiles * sizeof(AccumTile)));
    MORI_CUDA_CALL(Malloc(&finalTiles, numActiveTiles * sizeof(FinalTile)));
    MORI_CUDA_CALL(MallocManaged(&tileDescs, numActiveTiles * sizeof(TileDesc)));
    // MORI_CUDA_CALL(MallocAsync(&accumTiles, numActiveTiles * sizeof(AccumTile),
    //                            device->stream));
    // MORI_CUDA_CALL(MallocAsync(&finalTiles, numActiveTiles * sizeof(FinalTile),
    //                            device->stream));
    // MORI_CUDA_CALL(MallocAsync(&tileDescs, numActiveTiles * sizeof(TileDesc),
    //                            device->stream));
    MORI_CUDA_SYNC_CHECK();
    if (numActiveTiles)
      setTileCoords<<<divRoundUp(numActiveTiles,1024),1024,0,device->stream>>>
        (tileDescs,numActiveTiles,numTiles,
         device->tileIndexOffset,device->tileIndexScale);
    MORI_CUDA_SYNC_CHECK();
  }

  // ==================================================================

  __global__ void g_finalizeTiles(FinalTile *finalTiles,
                                  AccumTile *accumTiles)
  {
    int pixelID = threadIdx.x;
    int tileID  = blockIdx.x;

    uint32_t rgba32
      = owl::make_rgba(vec4f(accumTiles[tileID].accum[pixelID]));
    
    if (tileID == 0 && pixelID == 33)
      printf("### writing final tile:pixel %i:%i, value %i\n",
             tileID,pixelID,rgba32);
             
    finalTiles[tileID].rgba[pixelID] = rgba32;
  }

  /*! write this tiledFB's tiles into given "final" frame buffer
    (i.e., a plain 2D array of numPixels.x*numPixels.y RGBA8
    pixels) */
  void TiledFB::finalizeTiles()
  {
    SetActiveGPU forDuration(device);
    PING; PRINT(numActiveTiles);
    if (numActiveTiles > 0)
      g_finalizeTiles<<<numActiveTiles,pixelsPerTile,0,device->stream>>>
      (finalTiles,accumTiles);
  }


  // ==================================================================

  __global__ void g_writeFinalPixels(uint32_t  *finalFB,
                                     vec2i      numPixels,
                                     FinalTile *finalTiles,
                                     TileDesc  *tileDescs)
  {
    int tileID = blockIdx.x;
    int ix = threadIdx.x + tileDescs[tileID].lower.x;
    int iy = threadIdx.y + tileDescs[tileID].lower.y;
    if (ix >= numPixels.x) return;
    if (iy >= numPixels.y) return;

    uint32_t pixelValue
      = finalTiles[tileID].rgba[threadIdx.x + tileSize*threadIdx.y];
    
    if (ix == 1100 && iy == 700)
      printf("pixel %i %i tile %i lower %i %i value %i\n",
             ix,iy,tileID,tileDescs[tileID].lower.x,tileDescs[tileID].lower.y,
             pixelValue);
             

    finalFB[ix + numPixels.x*iy] = pixelValue;
  }
                                 
  void TiledFB::writeFinalPixels(DeviceContext *device,
                                 uint32_t  *finalFB,
                                 vec2i      numPixels,
                                 FinalTile *finalTiles,
                                 TileDesc  *tileDescs,
                                 int        numTiles)
  {
    if (finalFB == 0) throw std::runtime_error("invalid finalfb of null!");

    SetActiveGPU forDuration(device);
    if (numTiles > 0)
      g_writeFinalPixels
        <<<numTiles,vec2i(tileSize),0,device->stream>>>
        (finalFB,numPixels,
         finalTiles,tileDescs);
  }
  
}
