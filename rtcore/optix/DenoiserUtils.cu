#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2025 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "DenoiserUtils.h"
#include <hip/hip_runtime.h>

// Use the common CUDA header that includes proper type definitions
#include "rtcore/cudaCommon/cuda-common.h"

namespace rtc {
  namespace optix {

    __device__ float linear_to_srgb_device(float x) {
      return (x <= 0.0031308f) ? 12.92f * x : 1.055f * powf(x, 1.f/2.4f) - 0.055f;
    }

    __device__ uint32_t pack_rgba(float r, float g, float b, float a) {
      uint32_t ri = min(255, max(0, int(r * 256.f)));
      uint32_t gi = min(255, max(0, int(g * 256.f)));
      uint32_t bi = min(255, max(0, int(b * 256.f)));
      uint32_t ai = min(255, max(0, int(a * 256.f)));
      return (ri << 0) | (gi << 8) | (bi << 16) | (ai << 24);
    }

    __global__ void convert_float4_to_rgba_kernel(
        const float4* input,
        uint32_t* output,
        int width,
        int height,
        bool srgb)
    {
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      int idy = blockIdx.y * blockDim.y + threadIdx.y;
      
      if (idx >= width || idy >= height) return;
      
      int pixel_idx = idy * width + idx;
      float4 pixel = input[pixel_idx];
      
      if (srgb) {
        // Convert linear to sRGB
        pixel.x = linear_to_srgb_device(pixel.x);
        pixel.y = linear_to_srgb_device(pixel.y);
        pixel.z = linear_to_srgb_device(pixel.z);
      }
      
      output[pixel_idx] = pack_rgba(pixel.x, pixel.y, pixel.z, pixel.w);
    }

    void convert_float4_to_rgba(
        const void* input,
        void* output,
        int width,
        int height,
        bool srgb,
        hipStream_t stream)
    {
      dim3 blockSize(16, 16);
      dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                    (height + blockSize.y - 1) / blockSize.y);
      
      convert_float4_to_rgba_kernel<<<gridSize, blockSize, 0, stream>>>(
          (const float4*)input,
          (uint32_t*)output,
          width,
          height,
          srgb);
    }

  } // namespace optix
} // namespace rtc
