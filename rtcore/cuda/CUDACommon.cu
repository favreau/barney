#include "hip/hip_runtime.h"
#include "rtcore/cuda/CUDACommon.h"

namespace rtc {
  namespace cuda_common {

    // BaseBackend::BaseBackend()
    // {
    //   hipFree(0);
    //   BARNEY_CUDA_CALL(GetDeviceCount(&numPhysicalDevices));
    // }
    
    int Device::setActive() const
    {
      int oldActive = 0;
      BARNEY_CUDA_CHECK(hipGetDevice(&oldActive));
      BARNEY_CUDA_CHECK(hipSetDevice(physicalID));
      return oldActive;
    }
    
    void Device::restoreActive(int oldActive) const
    {
      BARNEY_CUDA_CHECK(hipSetDevice(oldActive));
    }
    
    void *Device::allocMem(size_t numBytes)
    {
      SetActiveGPU forDuration(this);
      void *ptr = 0;
      BARNEY_CUDA_CALL(Malloc((void **)&ptr,numBytes));
      assert(ptr);
      return ptr;
    }
    
    void *Device::allocHost(size_t numBytes) 
    {
      SetActiveGPU forDuration(this);
      void *ptr = 0;
      BARNEY_CUDA_CALL(MallocHost(&ptr,numBytes));
      return ptr;
    }
      
    void Device::freeHost(void *mem) 
    {
      SetActiveGPU forDuration(this);
      BARNEY_CUDA_CALL(FreeHost(mem));
    }
      
    void Device::freeMem(void *mem) 
    {
      SetActiveGPU forDuration(this);
      BARNEY_CUDA_CALL(Free(mem));
    }
      
    void Device::memsetAsync(void *mem,int value, size_t size) 
    {
      SetActiveGPU forDuration(this);
      BARNEY_CUDA_CALL(MemsetAsync(mem,value,size,stream));
    }
      

    void Device::copyAsync(void *dst, const void *src, size_t numBytes) 
    {
      SetActiveGPU forDuration(this);
      BARNEY_CUDA_CALL(MemcpyAsync(dst,src,numBytes,hipMemcpyDefault,stream));
    }
      
    void Device::sync() 
    {
      SetActiveGPU forDuration(this);
      BARNEY_CUDA_CALL(StreamSynchronize(stream));
    }

    void Device::freeTextureData(TextureData *td)
    {
      if (td) delete td;
    }
    
    void Device::freeTexture(Texture *tex)
    {
      if (tex) delete tex;
    }
    
    TextureData::TextureData(Device *device,
                             vec3i dims,
                             rtc::DataType format,
                             const void *texels)
      : device(device), dims(dims), format(format)
    {
      hipChannelFormatDesc desc;
      size_t sizeOfScalar;
      size_t numScalarsPerTexel;
      switch (format) {
      case rtc::FLOAT:
        desc         = hipCreateChannelDesc<float>();
        sizeOfScalar = 4;
        readMode     = hipReadModeElementType;
        numScalarsPerTexel = 1;
        break;
      case rtc::FLOAT4:
        desc         = hipCreateChannelDesc<float4>();
        sizeOfScalar = 4;
        readMode     = hipReadModeElementType;
        numScalarsPerTexel = 4;
        break;
      case rtc::UCHAR:
        desc         = hipCreateChannelDesc<uint8_t>();
        sizeOfScalar = 1;
        readMode     = hipReadModeNormalizedFloat;
        numScalarsPerTexel = 1;
        break;
      case rtc::UCHAR4:
        desc         = hipCreateChannelDesc<uchar4>();
        sizeOfScalar = 1;
        readMode     = hipReadModeNormalizedFloat;
        numScalarsPerTexel = 4;
        break;
      case rtc::USHORT:
        desc         = hipCreateChannelDesc<uint16_t>();
        sizeOfScalar = 2;
        readMode     = hipReadModeNormalizedFloat;
        numScalarsPerTexel = 1;
        break;
      default:
        assert(0);
      };

      if (dims.z != 0) {
        unsigned int padded_x = (unsigned)dims.x;
        unsigned int padded_y = std::max(1u,(unsigned)dims.y);
        unsigned int padded_z = std::max(1u,(unsigned)dims.z);
        hipExtent extent{padded_x,padded_y,padded_z};
        BARNEY_CUDA_CALL(Malloc3DArray(&array,&desc,extent,0));
        hipMemcpy3DParms copyParms;
        memset(&copyParms,0,sizeof(copyParms));
        copyParms.srcPtr
          = make_hipPitchedPtr((void *)texels,
                                (size_t)padded_x*sizeOfScalar*numScalarsPerTexel,
                                (size_t)padded_x,
                                (size_t)padded_y);
        copyParms.dstArray = array;
        copyParms.extent   = extent;
        copyParms.kind     = hipMemcpyHostToDevice;
        BARNEY_CUDA_CALL(Memcpy3D(&copyParms));
      } else if (dims.y != 0) {
        BARNEY_CUDA_CALL(MallocArray(&array,&desc,dims.x,dims.y,0));
        BARNEY_CUDA_CALL(Memcpy2DToArray(array,0,0,
                                         (void *)texels,
                                         (size_t)dims.x*sizeOfScalar*numScalarsPerTexel,
                                         (size_t)dims.x*sizeOfScalar*numScalarsPerTexel,
                                         (size_t)dims.y,
                                         hipMemcpyHostToDevice));
      } else {
        assert(0);
      }
    }

    TextureData::~TextureData()
    {
      BARNEY_CUDA_CALL_NOTHROW(FreeArray(array));
      array = 0;
    }
    
    TextureData *
    Device::createTextureData(vec3i dims,
                              rtc::DataType format,
                              const void *texels) 
    {
      SetActiveGPU forDuration(this);
      return new TextureData(this,dims,format,texels);
    }

    inline hipTextureFilterMode toCUDA(FilterMode mode)
    {
      return (mode == FILTER_MODE_POINT)
        ? hipFilterModePoint
        : hipFilterModeLinear;
    }
    
    inline hipTextureAddressMode toCUDA(AddressMode mode)
    {
      switch (mode) {
      case MIRROR:
        return hipAddressModeMirror;
      case CLAMP:
        return hipAddressModeClamp;
      case WRAP:
        return hipAddressModeWrap;
      case BORDER:
        return hipAddressModeBorder;
      };
      // just to make teh compiler happy:
      return hipAddressModeMirror;
    }

    Texture::Texture(TextureData *data,
                     const TextureDesc &desc)
      : data(data)//, desc(desc)
    {
      hipResourceDesc resourceDesc;
      memset(&resourceDesc,0,sizeof(resourceDesc));
      resourceDesc.resType         = hipResourceTypeArray;
      resourceDesc.res.array.array = data->array;
      
      hipTextureDesc textureDesc;
      memset(&textureDesc,0,sizeof(textureDesc));
      textureDesc.addressMode[0] = toCUDA(desc.addressMode[0]);
      textureDesc.addressMode[1] = toCUDA(desc.addressMode[1]);
      textureDesc.addressMode[2] = toCUDA(desc.addressMode[2]);
      textureDesc.filterMode     = toCUDA(desc.filterMode);
      textureDesc.readMode       = data->readMode;
      textureDesc.borderColor[0] = desc.borderColor.x;
      textureDesc.borderColor[1] = desc.borderColor.y;
      textureDesc.borderColor[2] = desc.borderColor.z;
      textureDesc.borderColor[3] = desc.borderColor.w;
      textureDesc.normalizedCoords = desc.normalizedCoords;
      
      BARNEY_CUDA_CALL(CreateTextureObject(&textureObject,
                                           &resourceDesc,
                                           &textureDesc,0));
    }

    Texture::~Texture()
    {
      BARNEY_CUDA_CALL_NOTHROW(DestroyTextureObject(textureObject));
      textureObject = 0;
    }
    
    Texture *TextureData::createTexture(const TextureDesc &desc) 
    {
      SetActiveGPU forDuration(device);
      return new Texture(this,desc);
    }    
    
  }
}

