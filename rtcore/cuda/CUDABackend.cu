#include "rtcore/cuda/CUDABackend.h"

namespace barney {
  namespace cuda {

    BaseBackend::BaseBackend()
    {
      hipFree(0);
      BARNEY_CUDA_CALL(GetDeviceCount(&numPhysicalDevices));
    }
    
    BaseDevGroup::BaseDevGroup(BaseBackend *backend,
                               const std::vector<int> &gpuIDs,
                               size_t sizeOfGlobals)
      : rtc::DevGroup(backend)
    {}
    

    int BaseDevice::setActive() const
    {
      int oldActive = 0;
      BARNEY_CUDA_CHECK(hipGetDevice(&oldActive));
      BARNEY_CUDA_CHECK(hipSetDevice(physicalID));
      return oldActive;
    }
    
    void BaseDevice::restoreActive(int oldActive) const
    {
      BARNEY_CUDA_CHECK(hipSetDevice(oldActive));
    }
    
    rtc::DevGroup *CUDABackend
    ::createDevGroup(const std::vector<int> &gpuIDs,
                     size_t sizeOfGlobals)
    {
      BARNEY_NYI();
    }
    
  }

  namespace rtc {
    Backend *createBackend_cuda()
    {
      return new barney::cuda::CUDABackend;
    }
  }
}

  
  



