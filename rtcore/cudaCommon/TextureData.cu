#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2025 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "rtcore/cudaCommon/Device.h"
#include "rtcore/cudaCommon/TextureData.h"

namespace rtc {
  namespace cuda_common {

    TextureData::TextureData(Device *device,
                             vec3i dims,
                             rtc::DataType format,
                             const void *texels)
      : device(device), dims(dims), format(format)
    {
      hipChannelFormatDesc desc;
      size_t sizeOfScalar;
      size_t numScalarsPerTexel;
      switch (format) {
      case rtc::FLOAT:
        desc         = hipCreateChannelDesc<float>();
        sizeOfScalar = 4;
        readMode     = hipReadModeElementType;
        numScalarsPerTexel = 1;
        break;
      case rtc::FLOAT4:
        desc         = hipCreateChannelDesc<float4>();
        sizeOfScalar = 4;
        readMode     = hipReadModeElementType;
        numScalarsPerTexel = 4;
        break;
      case rtc::UCHAR:
        desc         = hipCreateChannelDesc<uint8_t>();
        sizeOfScalar = 1;
        readMode     = hipReadModeNormalizedFloat;
        numScalarsPerTexel = 1;
        break;
      case rtc::UCHAR4:
        desc         = hipCreateChannelDesc<uchar4>();
        sizeOfScalar = 1;
        readMode     = hipReadModeNormalizedFloat;
        numScalarsPerTexel = 4;
        break;
      case rtc::USHORT:
        desc         = hipCreateChannelDesc<uint16_t>();
        sizeOfScalar = 2;
        readMode     = hipReadModeNormalizedFloat;
        numScalarsPerTexel = 1;
        break;
      default:
        assert(0);
      };

      if (dims.z != 0) {
        unsigned int padded_x = (unsigned)dims.x;
        unsigned int padded_y = std::max(1u,(unsigned)dims.y);
        unsigned int padded_z = std::max(1u,(unsigned)dims.z);
        hipExtent extent{padded_x,padded_y,padded_z};
        BARNEY_CUDA_CALL(Malloc3DArray(&array,&desc,extent,0));
        hipMemcpy3DParms copyParms;
        memset(&copyParms,0,sizeof(copyParms));
        copyParms.srcPtr
          = make_hipPitchedPtr((void *)texels,
                                (size_t)padded_x*sizeOfScalar*numScalarsPerTexel,
                                (size_t)padded_x,
                                (size_t)padded_y);
        copyParms.dstArray = array;
        copyParms.extent   = extent;
        copyParms.kind     = hipMemcpyHostToDevice;
        BARNEY_CUDA_CALL(Memcpy3D(&copyParms));
      } else if (dims.y != 0) {
        BARNEY_CUDA_CALL(MallocArray(&array,&desc,dims.x,dims.y,0));
        BARNEY_CUDA_CALL(Memcpy2DToArray(array,0,0,
                                         (void *)texels,
                                         (size_t)dims.x*sizeOfScalar*numScalarsPerTexel,
                                         (size_t)dims.x*sizeOfScalar*numScalarsPerTexel,
                                         (size_t)dims.y,
                                         hipMemcpyHostToDevice));
      } else {
        assert(0);
      }
    }

    TextureData::~TextureData()
    {
      BARNEY_CUDA_CALL_NOTHROW(FreeArray(array));
      array = 0;
    }
    
  }
}
