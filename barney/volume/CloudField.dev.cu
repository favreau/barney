#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/volume/CloudField.h"
#include "barney/volume/MCAccelerator.h"
#include "rtcore/TraceInterface.h"

RTC_DECLARE_GLOBALS(BARNEY_NS::render::OptixGlobals);

namespace BARNEY_NS {

  struct MCAccel_Cloud_Programs {
    static inline __rtc_device
    void bounds(const rtc::TraceInterface &ti,
                const void *geomData,
                owl::common::box3f &bounds,  
                const int32_t primID)
    {
#if RTC_DEVICE_CODE
      MCVolumeAccel<CloudSampler>
        ::boundsProg(ti,geomData,bounds,primID);
#endif
    }
    
    static inline __rtc_device
    void intersect(rtc::TraceInterface &ti)
    {
#if RTC_DEVICE_CODE
      // Custom Cloud sphere intersection - only trace within sphere bounds
      const void *pd = ti.getProgramData();
      const MCVolumeAccel<CloudSampler>::DD &self = 
        *(typename MCVolumeAccel<CloudSampler>::DD*)pd;
      const render::World::DD &world = render::OptixGlobals::get(ti).world;
      
      // Get ray in object space
      Ray &ray = *(Ray*)ti.getPRD();
      vec3f obj_org = ti.getObjectRayOrigin();
      vec3f obj_dir = ti.getObjectRayDirection();
      
      // Extract Cloud parameters from sampler
      const CloudSampler::DD &cloudSampler = self.volume.sfSampler;
      float planetRadius = cloudSampler.planetRadius;
      float atmosphereThickness = cloudSampler.atmosphereThickness;
      
      // Check ray-sphere intersection for atmospheric shell (between planetRadius and totalRadius)
      vec3f oc = obj_org;
      float totalRadius = planetRadius + atmosphereThickness;
      float a = dot(obj_dir, obj_dir);
      float b = 2.0f * dot(oc, obj_dir);
      
      // Outer sphere intersection
      float c_outer = dot(oc, oc) - totalRadius * totalRadius;
      float discriminant_outer = b * b - 4.0f * a * c_outer;
      if (discriminant_outer < 0.0f)
        return; // No intersection with outer atmosphere sphere
        
      float sqrt_disc_outer = sqrtf(discriminant_outer);
      float t_outer_near = (-b - sqrt_disc_outer) / (2.0f * a);
      float t_outer_far = (-b + sqrt_disc_outer) / (2.0f * a);
      
      // Inner sphere intersection (planet surface)
      float c_inner = dot(oc, oc) - planetRadius * planetRadius;
      float discriminant_inner = b * b - 4.0f * a * c_inner;
      
      range1f tRange = { ti.getRayTmin(), ti.getRayTmax() };
      
      if (discriminant_inner >= 0.0f) {
        // Ray intersects inner planet sphere - exclude interior
        float sqrt_disc_inner = sqrtf(discriminant_inner);
        float t_inner_near = (-b - sqrt_disc_inner) / (2.0f * a);
        float t_inner_far = (-b + sqrt_disc_inner) / (2.0f * a);
        
        // Ray-marching only in atmospheric shell - exclude planet interior
        if (t_inner_near > tRange.lower && t_inner_near < tRange.upper) {
          // Ray enters planet from outside - march until planet surface
          tRange.upper = min(tRange.upper, t_inner_near);
        }
        if (t_inner_far > tRange.lower && t_inner_far < tRange.upper) {
          // Ray exits planet - start marching from planet surface
          tRange.lower = max(tRange.lower, t_inner_far);
        }
        if (t_inner_near <= tRange.lower && t_inner_far >= tRange.upper) {
          // Ray entirely inside planet - no atmospheric ray-marching
          return;
        }
      }
      
      // Constrain to outer atmosphere boundary
      tRange.lower = max(tRange.lower, t_outer_near);
      tRange.upper = min(tRange.upper, t_outer_far);
      
      if (tRange.lower >= tRange.upper)
        return; // No valid intersection range in atmospheric shell
      
      // Convert to macro cell grid space for DDA traversal
      vec3f mcGridOrigin = self.mcGrid.gridOrigin;
      vec3f mcGridSpacing = self.mcGrid.gridSpacing;
      
      vec3f dda_org = (obj_org - mcGridOrigin) * rcp(mcGridSpacing);
      vec3f dda_dir = obj_dir * rcp(mcGridSpacing);
      
      Random rng(ray.rngSeed.next(hash(ti.getRTCInstanceIndex(),
                                       ti.getGeometryIndex(),
                                       ti.getPrimitiveIndex())));
      
      // DDA traversal through macro cells within sphere bounds
      dda::dda3(dda_org, dda_dir, tRange.upper,
                vec3ui(self.mcGrid.dims),
                [&](const vec3i &cellIdx, float t0, float t1) -> bool
                {
                  const float majorant = self.mcGrid.majorant(cellIdx);
                  
                  if (majorant == 0.f) return true;
                  
                  vec4f sample = 0.f;
                  range1f cellTRange = {max(t0, tRange.lower), min(t1, tRange.upper)};
                  
                  // Ensure we're within the atmospheric shell for this cell
                  vec3f cellOrg = obj_org + cellTRange.lower * obj_dir;
                  vec3f cellEnd = obj_org + cellTRange.upper * obj_dir;
                  float distOrg = length(cellOrg);
                  float distEnd = length(cellEnd);
                  
                  // Skip cell if entirely outside atmosphere
                  if (distOrg > totalRadius && distEnd > totalRadius)
                    return true; // Entire cell is outside atmosphere
                  
                  // Skip cell if entirely inside planet
                  if (distOrg < planetRadius && distEnd < planetRadius)
                    return true; // Entire cell is inside planet
                  
                  if (!Woodcock::sampleRange(sample,
                                           self.volume,
                                           obj_org,
                                           obj_dir,
                                           cellTRange,
                                           majorant,
                                           rng,
                                           ray.dbg)) 
                    return true;
                  
                  vec3f P_obj = obj_org + cellTRange.upper * obj_dir;
                  vec3f P = ti.transformPointFromObjectToWorldSpace(P_obj);
                  
                  ray.setVolumeHit(P, obj_dir,
                                   cellTRange.upper,
                                   getPos(sample));
                  ti.reportIntersection(cellTRange.upper, 0);
                  return false;
                },
                /*NO debug:*/false
                );
#endif
    }
    
    static inline __rtc_device
    void closestHit(rtc::TraceInterface &ti)
    {
#if RTC_DEVICE_CODE
      // Volume hit already processed in intersect
#endif
    }
    
    static inline __rtc_device
    bool anyHit(rtc::TraceInterface &ti)
    {
#if RTC_DEVICE_CODE
      return true; // Accept all volume hits
#endif
      return true;
    }
  };
  
  RTC_EXPORT_USER_GEOM(CloudMC,CloudField::DD,MCAccel_Cloud_Programs,false,false);
} 