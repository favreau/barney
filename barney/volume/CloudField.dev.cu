#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/volume/CloudField.h"
#include "barney/volume/MCAccelerator.h"
#include "rtcore/TraceInterface.h"

RTC_DECLARE_GLOBALS(BARNEY_NS::render::OptixGlobals);

namespace BARNEY_NS {

  struct MCAccel_Cloud_Programs {
    static inline __rtc_device
    void bounds(const rtc::TraceInterface &ti,
                const void *geomData,
                owl::common::box3f &bounds,  
                const int32_t primID)
    {
#if RTC_DEVICE_CODE
      MCVolumeAccel<CloudSampler>
        ::boundsProg(ti,geomData,bounds,primID);
#endif
    }
    
    static inline __rtc_device
    void intersect(rtc::TraceInterface &ti)
    {
#if RTC_DEVICE_CODE
      // Custom Cloud sphere intersection - only trace within sphere bounds
      const void *pd = ti.getProgramData();
      const MCVolumeAccel<CloudSampler>::DD &self = 
        *(typename MCVolumeAccel<CloudSampler>::DD*)pd;
      const render::World::DD &world = render::OptixGlobals::get(ti).world;
      
      // Get ray in object space
      Ray &ray = *(Ray*)ti.getPRD();
      vec3f obj_org = ti.getObjectRayOrigin();
      vec3f obj_dir = ti.getObjectRayDirection();
      
      // Extract Cloud parameters from sampler
      const CloudSampler::DD &cloudSampler = self.volume.sfSampler;
      float sphereRadius = cloudSampler.sphereRadius;
      float maxHeight = cloudSampler.maxHeight;
      vec3f sphereCenter = cloudSampler.sphereCenter;
      
      // Check ray-sphere intersection for outer cloud boundary
      vec3f oc = obj_org - sphereCenter;
      float totalRadius = sphereRadius + maxHeight;
      float a = dot(obj_dir, obj_dir);
      float b = 2.0f * dot(oc, obj_dir);
      float c = dot(oc, oc) - totalRadius * totalRadius;
      
      float discriminant = b * b - 4.0f * a * c;
      if (discriminant < 0.0f)
        return; // No intersection with cloud sphere
        
      float sqrt_disc = sqrtf(discriminant);
      float t_near = (-b - sqrt_disc) / (2.0f * a);
      float t_far = (-b + sqrt_disc) / (2.0f * a);
      
      // Clamp to valid ray range
      range1f tRange = { max(ti.getRayTmin(), t_near), min(ti.getRayTmax(), t_far) };
      if (tRange.lower >= tRange.upper)
        return; // No valid intersection range
      
      // Convert to macro cell grid space for DDA traversal
      vec3f mcGridOrigin = self.mcGrid.gridOrigin;
      vec3f mcGridSpacing = self.mcGrid.gridSpacing;
      
      vec3f dda_org = (obj_org - mcGridOrigin) * rcp(mcGridSpacing);
      vec3f dda_dir = obj_dir * rcp(mcGridSpacing);
      
      Random rng(ray.rngSeed.next(hash(ti.getRTCInstanceIndex(),
                                       ti.getGeometryIndex(),
                                       ti.getPrimitiveIndex())));
      
      // DDA traversal through macro cells within sphere bounds
      dda::dda3(dda_org, dda_dir, tRange.upper,
                vec3ui(self.mcGrid.dims),
                [&](const vec3i &cellIdx, float t0, float t1) -> bool
                {
                  const float majorant = self.mcGrid.majorant(cellIdx);
                  
                  if (majorant == 0.f) return true;
                  
                  vec4f sample = 0.f;
                  range1f cellTRange = {max(t0, tRange.lower), min(t1, tRange.upper)};
                  
                  // Ensure we're within the sphere for this cell
                  vec3f cellOrg = obj_org + cellTRange.lower * obj_dir;
                  vec3f cellEnd = obj_org + cellTRange.upper * obj_dir;
                  float distOrg = length(cellOrg - sphereCenter);
                  float distEnd = length(cellEnd - sphereCenter);
                  
                  if (distOrg > totalRadius && distEnd > totalRadius)
                    return true; // Entire cell is outside sphere
                  
                  if (!Woodcock::sampleRange(sample,
                                           self.volume,
                                           obj_org,
                                           obj_dir,
                                           cellTRange,
                                           majorant,
                                           rng,
                                           ray.dbg)) 
                    return true;
                  
                  vec3f P_obj = obj_org + cellTRange.upper * obj_dir;
                  vec3f P = ti.transformPointFromObjectToWorldSpace(P_obj);
                  
                  ray.setVolumeHit(P, obj_dir,
                                   cellTRange.upper,
                                   getPos(sample));
                  ti.reportIntersection(cellTRange.upper, 0);
                  return false;
                },
                /*NO debug:*/false
                );
#endif
    }
    
    static inline __rtc_device
    void closestHit(rtc::TraceInterface &ti)
    {
#if RTC_DEVICE_CODE
      // Volume hit already processed in intersect
#endif
    }
    
    static inline __rtc_device
    bool anyHit(rtc::TraceInterface &ti)
    {
#if RTC_DEVICE_CODE
      return true; // Accept all volume hits
#endif
      return true;
    }
  };
  
  RTC_EXPORT_USER_GEOM(CloudMC,CloudField::DD,MCAccel_Cloud_Programs,false,false);
} 