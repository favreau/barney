// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/Context.h"
#include "barney/volume/StructuredData.h"
#include "barney/common/Texture.h"
#include "rtcore/ComputeInterface.h"

namespace BARNEY_NS {

  RTC_IMPORT_USER_GEOM(/*file*/StructuredData,/*name*/StructuredMC,
                       /*geomtype device data */
                       MCVolumeAccel<StructuredDataSampler>::DD,false,false);
  RTC_IMPORT_USER_GEOM(/*file*/StructuredData,/*name*/StructuredMC_Iso,
                       /*geomtype device data */
                       MCIsoSurfaceAccel<StructuredDataSampler>::DD,false,false);
  // RTC_IMPORT_COMPUTE3D(StructuredData_computeMCs);

  // StructuredData::PLD *StructuredData::getPLD(Device *device) 
  // { return &perLogical[device->contextRank()]; } 

  /*! how many cells (in each dimension) will go into a macro
      cell. eg, a value of 8 will mean that eachmacrocell covers 8x8x8
      cells, and a 128^3 volume (with 127^3 cells...) will this have a
      macrocell grid of 16^3 macro cells (out of which the
      right/back/top one will only have 7x7x7 of its 8x8x8 covered by
      actual cells */
  enum { cellsPerMC = 8 };

//   /*! compute kernel that computes macro-cell information for a 3D
//       structured data grid */
//   struct StructuredData_ComputeMCs {
// #if RTC_DEVICE_CODE
//     /* kernel CODE */
//     inline __rtc_device void run(const rtc::ComputeInterface &rtCore)
//     {
//       vec3i mcID
//         = vec3i(rtCore.getThreadIdx())
//         + vec3i(rtCore.getBlockIdx())
//         * vec3i(rtCore.getBlockDim());
//       if (mcID.x >= mcGrid.dims.x) return;
//       if (mcID.y >= mcGrid.dims.y) return;
//       if (mcID.z >= mcGrid.dims.z) return;
        
//       range1f scalarRange;
//       for (int iiz=0;iiz<=cellsPerMC;iiz++)
//         for (int iiy=0;iiy<=cellsPerMC;iiy++)
//           for (int iix=0;iix<=cellsPerMC;iix++) {
//             vec3i scalarID = mcID*int(cellsPerMC) + vec3i(iix,iiy,iiz);
//             if (scalarID.x >= numScalars.x) continue;
//             if (scalarID.y >= numScalars.y) continue;
//             if (scalarID.z >= numScalars.z) continue;
//             float f = rtc::tex3D<float>(scalars,
//                                    (float)scalarID.x,
//                                    (float)scalarID.y,
//                                    (float)scalarID.z);
//             scalarRange.extend(f);
//           }
//       int mcIdx = mcID.x + mcGrid.dims.x*(mcID.y+mcGrid.dims.y*(mcID.z));
//       mcGrid.scalarRanges[mcIdx] = scalarRange;
//     }
// #endif      
//     /* kernel ARGS */
//     MCGrid::DD mcGrid;
//     vec3i numScalars;
//     rtc::TextureObject scalars;
//   };


  /*! compute kernel that computes macro-cell information for a 3D
      structured data grid */
  __rtc_global
  void StructuredData_computeMCs(const rtc::ComputeInterface &ci,
                                 /* kernel ARGS */
                                 MCGrid::DD mcGrid,
                                 vec3i numScalars,
                                 rtc::TextureObject scalars)
  {
    vec3i mcDims = mcGrid.dims;
    int tid = ci.launchIndex().x;
    if (tid >= mcDims.x*mcDims.y*mcDims.z) return;
    vec3i mcID(tid % mcDims.x,
               (tid / mcDims.x) % mcDims.y,
               tid / (mcDims.x*mcDims.y));
    
    range1f scalarRange;
    for (int iiz=0;iiz<=cellsPerMC;iiz++)
      for (int iiy=0;iiy<=cellsPerMC;iiy++)
        for (int iix=0;iix<=cellsPerMC;iix++) {
          vec3i scalarID = mcID*int(cellsPerMC) + vec3i(iix,iiy,iiz);
          if (scalarID.x >= numScalars.x) continue;
          if (scalarID.y >= numScalars.y) continue;
          if (scalarID.z >= numScalars.z) continue;
          float f = rtc::tex3D<float>(scalars,
                                      (float)scalarID.x,
                                      (float)scalarID.y,
                                      (float)scalarID.z);
          scalarRange.extend(f);
        }
    int mcIdx = mcID.x + mcGrid.dims.x*(mcID.y+mcGrid.dims.y*(mcID.z));
    mcGrid.scalarRanges[mcIdx] = scalarRange;
  }

  
  StructuredData::StructuredData(Context *context,
                                 const DevGroup::SP &devices)
    : ScalarField(context,devices)
  {
    // perLogical.resize(devices->numLogical);
    // if (mcID.x >= mcGrid.dims.x) return;
    // if (mcID.y >= mcGrid.dims.y) return;
    // if (mcID.z >= mcGrid.dims.z) return;
    
    // for (auto device : *devices)
    //   getPLD(device)->computeMCs
    //     = createCompute_StructuredData_computeMCs(device->rtc);
  }


  MCGrid::SP StructuredData::buildMCs() 
  {
    PING;
    MCGrid::SP mcGrid = std::make_shared<MCGrid>(devices);
    vec3i mcDims = divRoundUp(numCells,vec3i(cellsPerMC));
    PRINT(mcDims);
    mcGrid->resize(mcDims);
    // vec3ui blockSize(4);
    // vec3ui numBlocks = divRoundUp(vec3ui(mcDims),blockSize);
    mcGrid->gridOrigin = worldBounds.lower;
    mcGrid->gridSpacing = vec3f(cellsPerMC) * this->gridSpacing;
    for (auto device : *devices) {
      // PLD *pld = getPLD(device);
      // StructuredData_ComputeMCs args = {
      //   mcGrid->getDD(device),
      //   numScalars,
      //   textureNN->getDD(device)
      // };
      // pld->computeMCs->launch(numBlocks,blockSize,
      //                         &args);
      int lc = mcDims.x*mcDims.y*mcDims.z;
      PRINT(lc);
      int bs = 128;
      int nb = divRoundUp(lc,bs);
      __rtc_launch(device->rtc,
                   StructuredData_computeMCs,
                   nb,bs,
                   mcGrid->getDD(device),
                   numScalars,
                   textureNN->getDD(device));
    }
    PING;
    for (auto device : *devices)
      device->sync();
    PING;
    return mcGrid;
  }
  
  StructuredDataSampler::DD StructuredDataSampler::getDD(Device *device)
  {
    DD dd;
    dd.texObj = sf->texture->getDD(device);
    dd.cellGridOrigin = sf->gridOrigin;
    dd.cellGridSpacing = sf->gridSpacing;
    dd.numCells = sf->numCells;
    return dd;
  }
  
  VolumeAccel::SP StructuredData::createAccel(Volume *volume) 
  {
    auto sampler = std::make_shared<StructuredDataSampler>(this);
    return std::make_shared<MCVolumeAccel<StructuredDataSampler>>
      (volume,
       createGeomType_StructuredMC,
       sampler);
  }
  
  IsoSurfaceAccel::SP StructuredData::createIsoAccel(IsoSurface *isoSurface) 
  {
    auto sampler = std::make_shared<StructuredDataSampler>(this);
    return std::make_shared<MCIsoSurfaceAccel<StructuredDataSampler>>
      (isoSurface,
       createGeomType_StructuredMC_Iso,
       sampler);
  }
  
  // ==================================================================
  bool StructuredData::set3f(const std::string &member,
                             const vec3f &value) 
  {
    if (member == "gridOrigin") {
      gridOrigin = value;
      return true;
    }
    if (member == "gridSpacing") {
      gridSpacing = value;
      return true;
    }
    return false;
  }

  // ==================================================================
  bool StructuredData::set3i(const std::string &member,
                             const vec3i &value) 
  {
    if (member == "dims") {
      numScalars = value;
      numCells   = value - vec3i(1);
      return true;
    }
    return false;
  }

  // ==================================================================
  bool StructuredData::setObject(const std::string &member,
                                 const Object::SP &value) 
  {
    if (member == "textureData") {
      scalars = value->as<TextureData>();
      BNTextureAddressMode addressModes[3] = {
        BN_TEXTURE_CLAMP,BN_TEXTURE_CLAMP,BN_TEXTURE_CLAMP
      };
      texture = std::make_shared<Texture>((Context*)context,scalars,
                                          BN_TEXTURE_LINEAR,
                                          addressModes,
                                          BN_COLOR_SPACE_LINEAR);
      textureNN = std::make_shared<Texture>((Context*)context,scalars,
                                            BN_TEXTURE_NEAREST,
                                            addressModes,
                                            BN_COLOR_SPACE_LINEAR);
      return true;
    }
    return false;
  }

  // ==================================================================
  void StructuredData::commit() 
  {
    worldBounds.lower = gridOrigin;
    worldBounds.upper = gridOrigin + gridSpacing * vec3f(numCells);
    PRINT(numCells);
    PRINT(worldBounds);
  }
  
  // RTC_EXPORT_COMPUTE3D(StructuredData_computeMCs,StructuredData_ComputeMCs);
}

