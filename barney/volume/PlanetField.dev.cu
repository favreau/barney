#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2025 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/volume/PlanetField.h"
#include "barney/volume/MCAccelerator.h"
#include "rtcore/TraceInterface.h"

RTC_DECLARE_GLOBALS(BARNEY_NS::render::OptixGlobals);

namespace BARNEY_NS {

  struct MCAccel_Planet_Programs {
    static inline __rtc_device
    void bounds(const rtc::TraceInterface &ti,
                const void *geomData,
                owl::common::box3f &bounds,  
                const int32_t primID)
    {
#if RTC_DEVICE_CODE
      MCVolumeAccel<PlanetSampler>
        ::boundsProg(ti,geomData,bounds,primID);
#endif
    }
    
    static inline __rtc_device
    void intersect(rtc::TraceInterface &ti)
    {
#if RTC_DEVICE_CODE
      // Custom Planet sphere intersection - only trace within sphere bounds
      const void *pd = ti.getProgramData();
      const MCVolumeAccel<PlanetSampler>::DD &self = 
        *(typename MCVolumeAccel<PlanetSampler>::DD*)pd;
      const render::World::DD &world = render::OptixGlobals::get(ti).world;
      
      // Get ray in object space
      Ray &ray = *(Ray*)ti.getPRD();
      vec3f obj_org = ti.getObjectRayOrigin();
      vec3f obj_dir = ti.getObjectRayDirection();
      
      // Extract Planet parameters from sampler
      const PlanetSampler::DD &planetSampler = self.volume.sfSampler;
      float sphereRadius = planetSampler.sphereRadius;
      
      // Check ray-sphere intersection for outer atmosphere boundary
      vec3f oc = obj_org;
      float totalRadius = 1.f;
      float a = dot(obj_dir, obj_dir);
      float b = 2.0f * dot(oc, obj_dir);
      float c = dot(oc, oc) - totalRadius * totalRadius;
      
      float discriminant = b * b - 4.0f * a * c;
      if (discriminant < 0.0f)
        return; // No intersection with atmosphere
        
      float sqrt_disc = sqrtf(discriminant);
      float t_near = (-b - sqrt_disc) / (2.0f * a);
      float t_far = (-b + sqrt_disc) / (2.0f * a);
      
      // Clamp to valid ray range
      range1f tRange = { max(ti.getRayTmin(), t_near), min(ti.getRayTmax(), t_far) };
      if (tRange.lower >= tRange.upper)
        return; // No valid intersection range
      
      // Convert to macro cell grid space for DDA traversal
      vec3f mcGridOrigin = self.mcGrid.gridOrigin;
      vec3f mcGridSpacing = self.mcGrid.gridSpacing;
      
      vec3f dda_org = (obj_org - mcGridOrigin) * rcp(mcGridSpacing);
      vec3f dda_dir = obj_dir * rcp(mcGridSpacing);
      
      Random rng(ray.rngSeed.next(hash(ti.getRTCInstanceIndex(),
                                       ti.getGeometryIndex(),
                                       ti.getPrimitiveIndex())));
      
      // DDA traversal through macro cells within sphere bounds
      dda::dda3(dda_org, dda_dir, tRange.upper,
                vec3ui(self.mcGrid.dims),
                [&](const vec3i &cellIdx, float t0, float t1) -> bool
                {
                  const float majorant = self.mcGrid.majorant(cellIdx);
                  
                  if (majorant == 0.f) return true;
                  
                  vec4f sample = 0.f;
                  range1f cellTRange = {max(t0, tRange.lower), min(t1, tRange.upper)};
                  
                  // Ensure we're within the sphere for this cell
                  vec3f cellOrg = obj_org + cellTRange.lower * obj_dir;
                  vec3f cellEnd = obj_org + cellTRange.upper * obj_dir;
                  float distOrg = length(cellOrg);
                  float distEnd = length(cellEnd);
                  
                  if (distOrg > totalRadius && distEnd > totalRadius)
                    return true; // Entire cell is outside sphere
                  
                  if (!Woodcock::sampleRange(sample,
                                           self.volume,
                                           obj_org,
                                           obj_dir,
                                           cellTRange,
                                           majorant,
                                           rng,
                                           ray.dbg)) 
                    return true;
                  
                  vec3f P_obj = obj_org + cellTRange.upper * obj_dir;
                  vec3f P = ti.transformPointFromObjectToWorldSpace(P_obj);
                  
                  vec3f normal = normalize(P);
                  if (planetSampler.normalTex) {
                    const float theta = acosf(clamp(normal.y, -1.0f, 1.0f));
                    const float phi = atan2f(normal.z, normal.x) + M_PI;
                    const vec2f uv(phi / (2.0f * M_PI), theta / M_PI);
                    const vec4f normal_tex = rtc::tex2D<vec4f>(planetSampler.normalTex, uv.x, uv.y);
                    normal = normalize(vec3f(normal_tex.x, normal_tex.y, normal_tex.z));
                  }
                  ray.setVolumeHit(P, normal,
                                  cellTRange.upper,
                                  getPos(sample));

                  ti.reportIntersection(cellTRange.upper, 0);
                  return false;
                },
                /*NO debug:*/false
                );
#endif
    }
    
    static inline __rtc_device
    void closestHit(rtc::TraceInterface &ti)
    {
#if RTC_DEVICE_CODE
      // Volume hit already processed in intersect
#endif
    }
    
    static inline __rtc_device
    bool anyHit(rtc::TraceInterface &ti)
    {
#if RTC_DEVICE_CODE
      return true; // Accept all volume hits
#endif
      return true;
    }
  };
  
  RTC_EXPORT_USER_GEOM(PlanetMC,PlanetField::DD,MCAccel_Planet_Programs,false,false);
} 