#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/Context.h"
#include "barney/volume/CloudField.h"
#include "barney/common/Texture.h"
#include "rtcore/ComputeInterface.h"

namespace BARNEY_NS {

  RTC_IMPORT_USER_GEOM(/*file*/CloudField,/*name*/CloudMC,
                       /*geomtype device data */
                       MCVolumeAccel<CloudSampler>::DD,false,false);
  RTC_IMPORT_COMPUTE3D(CloudField_computeMCs);

  CloudField::CLD *CloudField::getCLD(Device *device) 
  { return &perLogical[device->contextRank()]; } 

  /*! how many cells (in each dimension) will go into a macro
      cell for spherical volumes */
  enum { cellsPerMC = 8 };

  /*! compute kernel that computes macro-cell information for Cloud volumes */
  struct CloudField_ComputeMCs {
#if RTC_DEVICE_CODE
    /* kernel CODE */
    inline __rtc_device void run(const rtc::ComputeInterface &rtCore)
    {
      vec3i mcID
        = vec3i(rtCore.getThreadIdx())
        + vec3i(rtCore.getBlockIdx())
        * vec3i(rtCore.getBlockDim());
      if (mcID.x >= mcGrid.dims.x) return;
      if (mcID.y >= mcGrid.dims.y) return;
      if (mcID.z >= mcGrid.dims.z) return;
        
      range1f scalarRange;
      
      // Sample density values in a grid pattern within this macro cell
      vec3f mcOrigin = mcGrid.gridOrigin + vec3f(mcID) * mcGrid.gridSpacing;
      vec3f mcSize = mcGrid.gridSpacing;
      
      for (int iiz=0; iiz<=cellsPerMC; iiz++)
        for (int iiy=0; iiy<=cellsPerMC; iiy++)
          for (int iix=0; iix<=cellsPerMC; iix++) {
            vec3f samplePos = mcOrigin + vec3f(iix, iiy, iiz) * (mcSize / float(cellsPerMC));
            float density = cloudSampler.sample(samplePos);
            scalarRange.extend(density);
          }
          
      int mcIdx = mcID.x + mcGrid.dims.x*(mcID.y+mcGrid.dims.y*(mcID.z));
      mcGrid.scalarRanges[mcIdx] = scalarRange;
    }
#endif      
    /* kernel ARGS */
    MCGrid::DD mcGrid;
    CloudSampler::DD cloudSampler;
  };
  
  
  CloudField::CloudField(Context *context,
                         const DevGroup::SP &devices)
    : ScalarField(context,devices)
  {
    perLogical.resize(devices->numLogical);
    for (auto device : *devices)
      getCLD(device)->computeMCs
        = createCompute_CloudField_computeMCs(device->rtc);
  }

  void CloudField::buildMCs(MCGrid &mcGrid) 
  {
    // Create macro cell grid that covers the spherical volume
    // The sphere is centered at origin with radius planetRadius + atmosphereThickness
    float totalRadius = planetRadius + atmosphereThickness;
    vec3f boxMin = -vec3f(totalRadius);
    vec3f boxMax = vec3f(totalRadius);
    
    worldBounds = box3f(boxMin, boxMax);
    
    vec3f boxSize = boxMax - boxMin;
    float cellSize = totalRadius * 2.0f / static_cast<float>(cellsPerMC);
    vec3i mcDims = vec3i(ceil(boxSize.x / cellSize), ceil(boxSize.y / cellSize), ceil(boxSize.z / cellSize));
    mcDims = max(mcDims, vec3i(4)); // Minimum grid size
    
    mcGrid.resize(mcDims);
    vec3ui blockSize(4);
    vec3ui numBlocks = divRoundUp(vec3ui(mcDims),blockSize);
    mcGrid.gridOrigin = boxMin;
    mcGrid.gridSpacing = boxSize / vec3f(mcDims);
    
    for (auto device : *devices) {
      CLD *cld = getCLD(device);
      auto tempSampler = std::make_shared<CloudSampler>(this);
      CloudField_ComputeMCs args = {
        mcGrid.getDD(device),
        tempSampler->getDD(device)
      };
      cld->computeMCs->launch(numBlocks,blockSize,
                              &args);
    }
    for (auto device : *devices)
      device->sync();
  }
  
  CloudSampler::DD CloudSampler::getDD(Device *device)
  {
    DD dd;
    
    // Set texture object and dimensions
    if (sf->cloudData) {
      dd.cloudDataTex = sf->cloudData->getDD(device);
      dd.textureDims = sf->cloudTextureData->dims; // Get texture dimensions for coordinate calculation
    }
    
    // Set sphere parameters
    dd.planetRadius = sf->planetRadius;
    dd.atmosphereThickness = sf->atmosphereThickness;
    
    return dd;
  }
  
  VolumeAccel::SP CloudField::createAccel(Volume *volume) 
  {
    auto sampler = std::make_shared<CloudSampler>(this);
    return std::make_shared<MCVolumeAccel<CloudSampler>>
      (volume,
       createGeomType_CloudMC,
       sampler);
  }
  
  // ==================================================================
  bool CloudField::set1f(const std::string &member,
                         const float &value) 
  {
    if (member == "planetRadius") {
      planetRadius = value;
      return true;
    }
    if (member == "atmosphereThickness") {
      atmosphereThickness = value;
      return true;
    }
    return false;
  }

  // ==================================================================
  bool CloudField::setObject(const std::string &member,
                             const Object::SP &value) 
  {
    BNTextureAddressMode addressModes[3] = {
      BN_TEXTURE_CLAMP, BN_TEXTURE_CLAMP, BN_TEXTURE_CLAMP
    };
    
    if (member == "cloudData") {
      cloudTextureData = value->as<TextureData>();
      cloudData = std::make_shared<Texture>((Context*)context, cloudTextureData,
                                           BN_TEXTURE_LINEAR,
                                           addressModes,
                                           BN_COLOR_SPACE_LINEAR);
      return true;
    }
    return false;
  }

  // ==================================================================
  void CloudField::commit() 
  {
    float totalRadius = planetRadius + atmosphereThickness;
    worldBounds.lower = -vec3f(totalRadius);
    worldBounds.upper = vec3f(totalRadius);
  }

  CloudField::DD CloudField::getDD(Device *device)
  {
    DD dd;
    dd.worldBounds = worldBounds;
    dd.planetRadius = planetRadius;
    dd.atmosphereThickness = atmosphereThickness;
    return dd;
  }
  
  RTC_EXPORT_COMPUTE3D(CloudField_computeMCs,CloudField_ComputeMCs);
} 