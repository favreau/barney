#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/Context.h"
#include "barney/volume/PlanetField.h"
#include "barney/common/Texture.h"
#include "rtcore/ComputeInterface.h"

namespace BARNEY_NS {

  RTC_IMPORT_USER_GEOM(/*file*/PlanetField,/*name*/PlanetMC,
                       /*geomtype device data */
                       MCVolumeAccel<PlanetSampler>::DD,false,false);
  RTC_IMPORT_COMPUTE3D(PlanetField_computeMCs);

  PlanetField::PLD *PlanetField::getPLD(Device *device) 
  { return &perLogical[device->contextRank()]; } 

  /*! how many cells (in each dimension) will go into a macro
      cell for spherical volumes */
  enum { cellsPerMC = 8 };

  /*! compute kernel that computes macro-cell information for Planet volumes */
  struct PlanetField_ComputeMCs {
#if RTC_DEVICE_CODE
    /* kernel CODE */
    inline __rtc_device void run(const rtc::ComputeInterface &rtCore)
    {
      vec3i mcID
        = vec3i(rtCore.getThreadIdx())
        + vec3i(rtCore.getBlockIdx())
        * vec3i(rtCore.getBlockDim());
      if (mcID.x >= mcGrid.dims.x) return;
      if (mcID.y >= mcGrid.dims.y) return;
      if (mcID.z >= mcGrid.dims.z) return;
        
      range1f scalarRange;
      
      // Sample density values in a grid pattern within this macro cell
      vec3f mcOrigin = mcGrid.gridOrigin + vec3f(mcID) * mcGrid.gridSpacing;
      vec3f mcSize = mcGrid.gridSpacing;
      
      for (int iiz=0; iiz<=cellsPerMC; iiz++)
        for (int iiy=0; iiy<=cellsPerMC; iiy++)
          for (int iix=0; iix<=cellsPerMC; iix++) {
            vec3f samplePos = mcOrigin + vec3f(iix, iiy, iiz) * (mcSize / float(cellsPerMC));
            float density = planetSampler.sample(samplePos);
            scalarRange.extend(density);
          }
          
      int mcIdx = mcID.x + mcGrid.dims.x*(mcID.y+mcGrid.dims.y*(mcID.z));
      mcGrid.scalarRanges[mcIdx] = scalarRange;
    }
#endif      
    /* kernel ARGS */
    MCGrid::DD mcGrid;
    PlanetSampler::DD planetSampler;
  };
  
  PlanetField::PlanetField(Context *context,
                         const DevGroup::SP &devices)
    : ScalarField(context,devices)
  {
    perLogical.resize(devices->numLogical);
    for (auto device : *devices)
      getPLD(device)->computeMCs
        = createCompute_PlanetField_computeMCs(device->rtc);
  }

  void PlanetField::buildMCs(MCGrid &mcGrid) 
  {
    float totalRadius = 1.f;
    vec3f boxMin = -vec3f(totalRadius);
    vec3f boxMax = vec3f(totalRadius);
    
    worldBounds = box3f(boxMin, boxMax);
    
    vec3f boxSize = boxMax - boxMin;
    float cellSize = totalRadius * 2.0f / static_cast<float>(cellsPerMC);
    vec3i mcDims = vec3i(ceil(boxSize.x / cellSize), ceil(boxSize.y / cellSize), ceil(boxSize.z / cellSize));
    mcDims = max(mcDims, vec3i(4)); // Minimum grid size
    
    mcGrid.resize(mcDims);
    vec3ui blockSize(4);
    vec3ui numBlocks = divRoundUp(vec3ui(mcDims),blockSize);
    mcGrid.gridOrigin = boxMin;
    mcGrid.gridSpacing = boxSize / vec3f(mcDims);
    
    for (auto device : *devices) {
      PLD *pld = getPLD(device);
      auto tempSampler = std::make_shared<PlanetSampler>(this);
      PlanetField_ComputeMCs args = {
        mcGrid.getDD(device),
        tempSampler->getDD(device)
      };
      pld->computeMCs->launch(numBlocks,blockSize,
                              &args);
    }
    for (auto device : *devices)
      device->sync();
  }
  
  PlanetSampler::DD PlanetSampler::getDD(Device *device)
  {
    DD dd;
    
    // Set texture objects
    if (sf->elevationMap)
      dd.elevationTex = sf->elevationMap->getDD(device);
    if (sf->diffuseMap)
      dd.diffuseTex = sf->diffuseMap->getDD(device);
    if (sf->normalMap)
      dd.normalTex = sf->normalMap->getDD(device);
    
    // Set sphere parameters
    dd.planetRadius = sf->planetRadius;
    dd.elevationScale = sf->elevationScale;
    
    return dd;
  }
  
  VolumeAccel::SP PlanetField::createAccel(Volume *volume) 
  {
    auto sampler = std::make_shared<PlanetSampler>(this);
    return std::make_shared<MCVolumeAccel<PlanetSampler>>
      (volume,
       createGeomType_PlanetMC,
       sampler);
  }
  
  // ==================================================================
  bool PlanetField::set1f(const std::string &member,
                         const float &value) 
  {
    if (member == "planetRadius") {
      planetRadius = value;
      return true;
    }
    if (member == "elevationScale") {
      elevationScale = value;
      return true;
    }
    return false;
  }

  // ==================================================================
  bool PlanetField::setObject(const std::string &member,
                             const Object::SP &value) 
  {
    BNTextureAddressMode addressModes[3] = {
      BN_TEXTURE_WRAP, BN_TEXTURE_CLAMP, BN_TEXTURE_CLAMP
    };
    
    if (member == "elevationMap") {
      elevationData = value->as<TextureData>();
      elevationMap = std::make_shared<Texture>((Context*)context, elevationData,
                                              BN_TEXTURE_LINEAR,
                                              addressModes,
                                              BN_COLOR_SPACE_LINEAR);
      return true;
    }
    if (member == "diffuseMap") {
      diffuseData = value->as<TextureData>();
      diffuseMap = std::make_shared<Texture>((Context*)context, diffuseData,
                                            BN_TEXTURE_LINEAR,
                                            addressModes,
                                            BN_COLOR_SPACE_LINEAR);
      return true;
    }
    if (member == "normalMap") {
      normalData = value->as<TextureData>();
      normalMap = std::make_shared<Texture>((Context*)context, normalData,
                                           BN_TEXTURE_LINEAR,
                                           addressModes,
                                           BN_COLOR_SPACE_LINEAR);
      return true;
    }
    return false;
  }

  // ==================================================================
  void PlanetField::commit() 
  {
    const float totalRadius = planetRadius + elevationScale;
    worldBounds.lower = -vec3f(totalRadius);
    worldBounds.upper = vec3f(totalRadius);
  }

  PlanetField::DD PlanetField::getDD(Device *device)
  {
    DD dd;
    dd.worldBounds = worldBounds;
    dd.planetRadius = planetRadius;
    dd.elevationScale = elevationScale;
    return dd;
  }
  
  RTC_EXPORT_COMPUTE3D(PlanetField_computeMCs,PlanetField_ComputeMCs);
} 