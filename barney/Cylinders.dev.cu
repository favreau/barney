#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/Cylinders.h"
#include "owl/owl_device.h"

namespace barney {
  
  OPTIX_BOUNDS_PROGRAM(CylindersBounds)(const void *geomData,
                                        owl::common::box3f &bounds,  
                                        const int32_t primID)
  {
    const Cylinders::DD &geom = *(const Cylinders::DD *)geomData;
    vec2i idx = geom.indices[primID];
    float r   = geom.radii[primID];
    vec3f a = geom.points[idx.x];
    vec3f b = geom.points[idx.y];
    bounds.lower = min(a,b)-r;
    bounds.upper = max(a,b)+r;
  }

  OPTIX_CLOSEST_HIT_PROGRAM(CylindersCH)()
  {
    auto &ray = owl::getPRD<Ray>();
    auto &self = owl::getProgramData<Cylinders::DD>();
    int primID = optixGetPrimitiveIndex();
    
    ray.hadHit = true;
    ray.tMax = optixGetRayTmax();

    vec3f org = optixGetWorldRayOrigin();
    vec3f dir = optixGetWorldRayDirection();
    vec3f hitPos = org + ray.tMax * dir;
    vec3f baseColor = owl::randomColor(primID);
    ray.hit.baseColor = baseColor;
    // ray.hit.N = n;
  }
  
  OPTIX_INTERSECT_PROGRAM(CylindersIsec)()
  {
    // capped
    const int primID
      = optixGetPrimitiveIndex();
    const auto &self
      = owl::getProgramData<Cylinders::DD>();

    const vec2i idx = self.indices[primID];
    const vec3f v0  = self.points[idx.x];
    const vec3f v1  = self.points[idx.y];
    const float radius = self.radii[primID];

    const vec3f ray_org  = optixGetObjectRayOrigin();
    const vec3f ray_dir  = optixGetObjectRayDirection();
    float hit_t      = optixGetRayTmax();
    const float ray_tmin = optixGetRayTmin();
    const float ray_tmax = optixGetRayTmax();

    const vec3f d = ray_dir;
    const vec3f s = v1 - v0; // axis
    const vec3f sxd = cross(s, d);
    const float a = dot(sxd, sxd); // (s x d)^2
    if (a == 0.f)
      return;

    const vec3f f = v0 - ray_org;
    const vec3f sxf = cross(s, f);
    const float ra = 1.0f/a;
    const float ts = dot(sxd, sxf) * ra; // (s x d)(s x f) / (s x d)^2, in ray-space
    const vec3f fp = f - ts * d; // f' = v0 - closest point to axis

    const float s2 = dot(s, s); // s^2
    const vec3f perp = cross(s, fp); // s x f'
    const float c = radius*radius * s2 - dot(perp, perp); //  r^2 s^2 - (s x f')^2
    if (c < 0.f)
      return;

    float td = sqrtf(c * ra);
    const float tube_t0 = ts - td;
    const float tube_t1 = ts + td;
      
    // clip to cylinder caps
    const float sf = dot(s, f);
    const float sd = dot(s, d);

    float cap_t0 = -1e20f;
    float cap_t1 = -1e20f;
    if (sd == 0.f) {
      if (dot(ray_org-v0,v1-v0) < 0.f) return;
      if (dot(ray_org-v1,v0-v1) < 0.f) return;
    } else {
      const float rsd = 1.f/(sd);
      const float cap_t_v0 = sf * rsd;
      const float cap_t_v1 = cap_t_v0 + s2 * rsd;
      cap_t0 = min(cap_t_v0,cap_t_v1);
      cap_t1 = max(cap_t_v0,cap_t_v1);
    }
      
    // bool onCap_t0 = cap_t0 >= tube_t0;
    // bool onCap_t1 = cap_t1 <= tube_t1;
    const float t0 = max(cap_t0,tube_t0);
    const float t1 = min(cap_t1,tube_t1);
    if (t0 > t1) return;

    Ray &ray    = getPRD<Ray>();
    if (ray_tmin <= t0 && t0 <= ray_tmax) {
      // front side hit:
      ray.tMax = t0;
      td *= -1.f;
      float hit_surf_u = (ray.tMax * sd - sf) * 1.f/(s2);
      ray.hit.N
        = (t0 == cap_t0)
        ? s
        : (td * d - fp - hit_surf_u * s);
      
    } else if (ray_tmin <= t1 && t1 <= ray_tmax) {
      ray.tMax = t1;
      float hit_surf_u = (ray.tMax * sd - sf) * 1.f/(s2);
      ray.hit.N
        = (t1 == cap_t1)
        ? -s
        : (td * d - fp - hit_surf_u * s);
    } else
      return;

    ray.hit.P = ray_org + ray.tMax * ray_dir;
    optixReportIntersection(ray.tMax, 0);
    // const int primID = optixGetPrimitiveIndex();
    // const auto &self
    //   = owl::getProgramData<Cylinders::DD>();

    // vec3f center = self.origins[primID];
    // float radius = self.defaultRadius;
    
    // const vec3f org  = optixGetObjectRayOrigin();
    // const vec3f dir  = optixGetObjectRayDirection();
    // const float tmin = optixGetRayTmin();
    // float hit_t      = optixGetRayTmax();
    
    // const vec3f oc = org - center;
    // const float a = dot(dir,dir);
    // const float b = dot(oc, dir);
    // const float c = dot(oc, oc) - radius * radius;
    // const float discriminant = b * b - a * c;
    
    // if (discriminant < 0.f) return;

    // {
    //   float temp = (-b - sqrtf(discriminant)) / a;
    //   if (temp < hit_t && temp > tmin) 
    //     hit_t = temp;
    // }
      
    // {
    //   float temp = (-b + sqrtf(discriminant)) / a;
    //   if (temp < hit_t && temp > tmin) 
    //     hit_t = temp;
    // }
    // if (hit_t < optixGetRayTmax()) {
    //   optixReportIntersection(hit_t, 0);
    // }
  }
  
}
