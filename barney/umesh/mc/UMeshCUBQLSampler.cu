// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/umesh/mc/UMeshCUBQLSampler.h"
#include "cuBQL/builder/hip/hip_runtime.h"
#include "cuBQL/builder/cpu.h"

namespace barney {

  struct UMeshReorderElements {
    Element        *out;
    Element        *in;
    const uint32_t *primIDs;
    int             numElements;
    
    template<typename CI>
    inline __both__ void run(const CI &ci)
    {
      int li = ci.launchIndex().x;
      if (li >= numElements) return;

      out[li] = in[primIDs[li]];
    }
  };
  
  UMeshCUBQLSampler::UMeshCUBQLSampler(UMeshField *mesh)
    : mesh(mesh),
      devices(mesh->devices)
  {
    perLogical.resize(devices->size());
  }

  UMeshCUBQLSampler::PLD *UMeshCUBQLSampler::getPLD(Device *device) 
  {
    assert(device);
    assert(device->contextRank >= 0);
    assert(device->contextRank < perLogical.size());
    return &perLogical[device->contextRank];
  }
  
  UMeshCUBQLSampler::DD UMeshCUBQLSampler::getDD(Device *device)
  {
    DD dd;
    (UMeshField::DD &)dd = mesh->getDD(device);
    dd.bvhNodes = getPLD(device)->bvhNodes;
    return dd;
  }

  void UMeshCUBQLSampler::build()
  {
    int numElements = mesh->numElements;
    for (auto device : *devices) {
      PLD *pld = getPLD(device);
      if (pld->bvhNodes != 0)
        /* BVH already built! */
        continue;

      std::cout << "------------------------------------------" << std::endl;
      std::cout << "building UMeshCUBQL BVH!" << std::endl;
      std::cout << "------------------------------------------" << std::endl;
      
      bvh_t bvh;
      box3f *primBounds
        = (box3f*)device->rtc->allocMem(numElements*sizeof(box3f));
      range1f *valueRanges
        = (range1f*)device->rtc->allocMem(numElements*sizeof(range1f));
      mesh->computeElementBBs(device,
                              primBounds,valueRanges);
      device->rtc->sync();
      
      SetActiveGPU forDuration(device);
      if (device->rtc->computeType() == "cuda")
        cuBQL::gpuBuilder(bvh,
                          (const cuBQL::box_t<float,3>*)primBounds,
                          numElements,
                          cuBQL::BuildConfig());
      else
        cuBQL::cpu::spatialMedian(bvh,
                                  (const cuBQL::box_t<float,3>*)primBounds,
                                  numElements,
                                  cuBQL::BuildConfig());
      device->rtc->sync();
      device->rtc->freeMem(primBounds);
      device->rtc->freeMem(valueRanges);
    
      Element *reorderedElements
        = (Element *)device->rtc->allocMem(numElements*sizeof(Element));
      UMeshReorderElements args =
        {
          // Element  *out;
          reorderedElements,
          // Element  *in;
          mesh->getPLD(device)->elements,
          // uint32_t *primIDs;
          bvh.primIDs,
          // int numElements;
          numElements
        };
      int bs = 128;
      int nb = divRoundUp(numElements,bs);
      device->umeshReorderElements->launch(nb,bs,&args);
      device->rtc->copy(mesh->getPLD(device)->elements,
                        reorderedElements,
                        numElements*sizeof(Element));
      device->rtc->sync();
      device->rtc->freeMem(reorderedElements);

      // "save the node"
      pld->bvhNodes
        = (node_t *)device->rtc->allocMem(bvh.numNodes*sizeof(node_t));
      device->rtc->copy(pld->bvhNodes,bvh.nodes,bvh.numNodes*sizeof(node_t));
      device->rtc->sync();
      
      // ... and kill whatever else cubql may have in the bvh
      if (device->rtc->computeType() == "cuda")
        cuBQL::cuda::free(bvh,0);
      else
        cuBQL::cpu::freeBVH(bvh);
      
      std::cout << OWL_TERMINAL_LIGHT_GREEN
                << "#bn.umesh: cubql bvh built ..."
                << OWL_TERMINAL_DEFAULT << std::endl;
    }
  }
  
}

RTC_DECLARE_COMPUTE(umeshReorderElements,barney::UMeshReorderElements);
