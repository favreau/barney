#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/umesh/common/UMeshField.h"
#include "barney/Context.h"
#include "barney/volume/MCGrid.cuh"
// just to be able to create these accelerators:
// #include "barney/umesh/mc/UMeshMCAccelerator.h"
#include "barney/umesh/mc/UMeshCUBQLSampler.h"
#include "barney/umesh/os/RTXObjectSpace.h"
#include "barney/umesh/os/AWT.h"

#define BUFFER_CREATE owlDeviceBufferCreate
// #define BUFFER_CREATE owlManagedMemoryBufferCreate

namespace barney {

  extern "C" char UMeshMC_ptx[];
  
  // this is probably waaaay overkill for smallish voluems, but those
  // are fast, anyway. and this helps for large ones...
  // enum { MC_GRID_SIZE = 256 };

  inline __device__ float length3(vec4f v)
  { return length(getPos(v)); }
  
  template<int D> inline __device__
  void rasterTet(MCGrid::DD grid,
                 vec4f a,
                 vec4f b,
                 vec4f c,
                 vec4f d)
  {
    float lab = length3(b-a);
    float lac = length3(c-a);
    float lad = length3(d-a);
    float lbc = length3(c-b);
    float lbd = length3(d-b);
    float lcd = length3(d-c);
    float maxLen = max(max(max(max(max(lab,lac),lad),lbc),lbd),lcd);

    vec4f ab = 0.5f*(a+b);
    vec4f ac = 0.5f*(a+c);
    vec4f ad = 0.5f*(a+d);
    vec4f bc = 0.5f*(b+c);
    vec4f bd = 0.5f*(b+d);
    vec4f cd = 0.5f*(c+d);

#if 1
    vec4f oa,ob,oc,od0,od1;
    if (lab >= maxLen) {
      oa = ab;
      ob = c;
      oc = d;
      od0 = a;
      od1 = a;
      // rasterTet<D-1>(grid,ab,c,d,a);
      // rasterTet<D-1>(grid,ab,c,d,b);
    } else if (lac >= maxLen) {
      oa = ac;
      ob = b;
      oc = d;
      od0 = a;
      od1 = c;
      // rasterTet<D-1>(grid,ac,b,d,a);
      // rasterTet<D-1>(grid,ac,b,d,c);
    } else if (lad >= maxLen) {
      oa = ad;
      ob = b;
      oc = c;
      od0 = a;
      od1 = d;
      // rasterTet<D-1>(grid,ad,b,c,a);
      // rasterTet<D-1>(grid,ad,b,c,d);
    } else if (lbc >= maxLen) {
      oa = bc;
      ob = a;
      oc = d;
      od0 = b;
      od1 = c;
      // rasterTet<D-1>(grid,bc,a,d,b);
      // rasterTet<D-1>(grid,bc,a,d,c);
    } else if (lbd >= maxLen) {
      oa = bd;
      ob = a;
      oc = c;
      od0 = b;
      od1 = d;
      // rasterTet<D-1>(grid,bd,a,c,b);
      // rasterTet<D-1>(grid,bd,a,c,d);
    } else {
      oa = cd;
      ob = a;
      oc = b;
      od0 = c;
      od1 = d;
      // rasterTet<D-1>(grid,cd,a,b,c);
      // rasterTet<D-1>(grid,cd,a,b,d);
    }
    rasterTet<D-1>(grid,oa,ob,oc,od0);
    rasterTet<D-1>(grid,oa,ob,oc,od1);
#else
    if (lab >= maxLen) {
      rasterTet<D-1>(grid,ab,c,d,a);
      rasterTet<D-1>(grid,ab,c,d,b);
    } else if (lac >= maxLen) {
      rasterTet<D-1>(grid,ac,b,d,a);
      rasterTet<D-1>(grid,ac,b,d,c);
    } else if (lad >= maxLen) {
      rasterTet<D-1>(grid,ad,b,c,a);
      rasterTet<D-1>(grid,ad,b,c,d);
    } else if (lbc >= maxLen) {
      rasterTet<D-1>(grid,bc,a,d,b);
      rasterTet<D-1>(grid,bc,a,d,c);
    } else if (lbd >= maxLen) {
      rasterTet<D-1>(grid,bd,a,c,b);
      rasterTet<D-1>(grid,bd,a,c,d);
    } else {
      rasterTet<D-1>(grid,cd,a,b,c);
      rasterTet<D-1>(grid,cd,a,b,d);
    }
#endif
  }
  
  template<> inline __device__
  void rasterTet<0>(MCGrid::DD grid,
                    vec4f a,
                    vec4f b,
                    vec4f c,
                    vec4f d)
  {
    box4f bb;
    bb.extend(a);
    bb.extend(b);
    bb.extend(c);
    bb.extend(d);
    rasterBox(grid,bb);
  }
  
  __global__ void rasterElements(MCGrid::DD grid,
                                 UMeshField::DD mesh)
  {
    const int eltIdx = blockIdx.x*blockDim.x + threadIdx.x;
    if (eltIdx >= mesh.numElements) return;    

    auto elt = mesh.elements[eltIdx];
    if (elt.type == Element::TET) {
      const vec4i indices = *(const vec4i *)&mesh.indices[elt.ofs0];
      vec4f a = make_vec4f(mesh.vertices[indices.x]);
      vec4f b = make_vec4f(mesh.vertices[indices.y]);
      vec4f c = make_vec4f(mesh.vertices[indices.z]);
      vec4f d = make_vec4f(mesh.vertices[indices.w]);
      rasterTet<5>(grid,a,b,c,d);
      return;
    }
//     if (elt.type == Element::GRID) {
//       int primID = elt.ID;

//       const box3f bounds = box3f((const vec3f &)mesh.gridDomains[primID].lower,
//                                  (const vec3f &)mesh.gridDomains[primID].upper);

//       vec3i numScalars = mesh.gridDims[primID]+1;
//       vec3f cellSize = bounds.size()/vec3f(mesh.gridDims[primID]);

//       const float *scalars = mesh.gridScalars + mesh.gridOffsets[primID];

//       auto linearIndex = [numScalars](const int x, const int y, const int z) {
//         return z*numScalars.y*numScalars.x + y*numScalars.x + x;
//       };

//       for (int z=0;z<mesh.gridDims[primID].z;z++) {
//         for (int y=0;y<mesh.gridDims[primID].y;y++) {
//           for (int x=0;x<mesh.gridDims[primID].x;x++) {
//             vec3i imin(x,y,z);
//             vec3i imax(x+1,y+1,z+1);

//             float f1 = scalars[linearIndex(imin.x,imin.y,imin.z)];
//             float f2 = scalars[linearIndex(imax.x,imin.y,imin.z)];
//             float f3 = scalars[linearIndex(imin.x,imax.y,imin.z)];
//             float f4 = scalars[linearIndex(imax.x,imax.y,imin.z)];

//             float f5 = scalars[linearIndex(imin.x,imin.y,imax.z)];
//             float f6 = scalars[linearIndex(imax.x,imin.y,imax.z)];
//             float f7 = scalars[linearIndex(imin.x,imax.y,imax.z)];
//             float f8 = scalars[linearIndex(imax.x,imax.y,imax.z)];

// #define EMPTY(x) isnan(x)
//             if (EMPTY(f1) || EMPTY(f2) || EMPTY(f3) || EMPTY(f4) ||
//                 EMPTY(f5) || EMPTY(f6) || EMPTY(f7) || EMPTY(f8))
//               continue;

//             float fmin = min(f1,min(f2,min(f3,min(f4,min(f5,min(f6,min(f7,f8)))))));
//             float fmax = max(f1,max(f2,max(f3,max(f4,max(f5,max(f6,max(f7,f8)))))));

//             const box4f cellBounds(vec4f(bounds.lower+vec3f(imin),fmin),
//                                    vec4f(bounds.lower+vec3f(imax),fmax));
//             rasterBox(grid,getBox(mesh.worldBounds),cellBounds);
//           }
//         }
//       }
//     } else
    {
      const box4f eltBounds = mesh.eltBounds(elt);
      rasterBox(grid,getBox(mesh.worldBounds),eltBounds);
    }
  }

  void UMeshField::buildMCs(MCGrid &grid)
  {
    buildInitialMacroCells(grid);
  }
  
  /*! build *initial* macro-cell grid (ie, the scalar field min/max
    ranges, but not yet the majorants) over a umesh */
  void UMeshField::buildInitialMacroCells(MCGrid &grid)
  {
    if (grid.built()) {
      // initial grid already built
      return;
    }
    
    float maxWidth = reduce_max(getBox(worldBounds).size());
    int MC_GRID_SIZE
      = 128 + int(sqrtf((float)elements.size())/30);
    vec3i dims = 1+vec3i(getBox(worldBounds).size() * ((MC_GRID_SIZE-1) / maxWidth));
    std::cout << OWL_TERMINAL_BLUE
              << "#bn.um: building initial macro cell grid of " << dims << " MCs"
              << OWL_TERMINAL_DEFAULT << std::endl;
    grid.resize(dims);

    grid.gridOrigin
      = worldBounds.lower;
    grid.gridSpacing
      = worldBounds.size() * rcp(vec3f(dims));
    
    grid.clearCells();
    
    const vec3i bs = 4;
    const vec3i nb = divRoundUp(dims,bs);
    for (auto dev : getDevices()) {
      assert(dev); assert(dev.get());
      SetActiveGPU forDuration(dev);
      auto d_mesh = getDD(dev);
      auto d_grid = grid.getDD(dev);
      rasterElements
        <<<divRoundUp(int(elements.size()),128),128>>>
        (d_grid,d_mesh);
      BARNEY_CUDA_SYNC_CHECK();
    }
  }
    
  
  /*! computes - ON CURRENT DEVICE - the given mesh's prim bounds and
    per-prim scalar ranges, and writes those into givne
    pre-allocated device mem location */
  __global__
  void g_computeElementBoundingBoxes(box3f *d_primBounds,
                                     range1f *d_primRanges,
                                     UMeshField::DD mesh)
  {
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if (tid >= mesh.numElements) return;

    auto elt = mesh.elements[tid];
    box4f eb = mesh.eltBounds(elt);
    d_primBounds[tid] = getBox(eb);
    if (d_primRanges) d_primRanges[tid] = getRange(eb);
  }

  /*! computes, on specified device, the bounding boxes and - if
    d_primRanges is non-null - the primitmives ranges. d_primBounds
    and d_primRanges (if non-null) must be pre-allocated and
    writeaable on specified device */
  void UMeshField::computeElementBBs(const Device::SP &device,
                                     box3f *d_primBounds,
                                     range1f *d_primRanges)
  {
    assert(device); assert(device.get());
    SetActiveGPU forDuration(device);
    int bs = 1024;
    int nb = divRoundUp(int(elements.size()),bs);
    g_computeElementBoundingBoxes
      <<<nb,bs>>>(d_primBounds,d_primRanges,getDD(device));
    BARNEY_CUDA_SYNC_CHECK();
  }

  UMeshField::UMeshField(Context *context, int slot,
                         std::vector<vec4f>   &_vertices,
                         std::vector<int>     &_indices,
                         std::vector<Element> &_elements,
                         const box3f &domain)
    : ScalarField(context,slot,domain),
      vertices(std::move(_vertices)),
      indices(std::move(_indices)),
      elements(std::move(_elements))
  {
    for (auto vtx : vertices) worldBounds.extend(getPos(vtx));

    if (!domain.empty())
      worldBounds = intersection(worldBounds,domain);

    assert(!elements.empty());

    verticesBuffer
      = BUFFER_CREATE(getOWL(),
                      OWL_FLOAT4,
                      vertices.size(),
                      vertices.data());
    indicesBuffer
      = BUFFER_CREATE(getOWL(),
                      OWL_INT,
                      indices.size(),
                      indices.data());

    elementsBuffer
      = BUFFER_CREATE(getOWL(),
                      OWL_INT,
                      elements.size(),
                      elements.data());
  }

  UMeshField::DD UMeshField::getDD(const Device::SP &device)
  {
    assert(device.get());
    UMeshField::DD dd;
    int devID = device->owlID;
    assert(verticesBuffer);
    assert(indicesBuffer);
    assert(elementsBuffer);
    dd.vertices
      = (const float4  *)owlBufferGetPointer(verticesBuffer,devID);
    dd.indices
      = (const int     *)owlBufferGetPointer(indicesBuffer,devID);
    dd.elements
      = (const Element *)owlBufferGetPointer(elementsBuffer,devID);
    dd.numElements
      = (int)elements.size();
    dd.worldBounds
      = worldBounds;
    
    return dd;
  }
  

  ScalarField::SP UMeshField::create(Context *context, int slot,
                                     const vec4f   *_vertices, int numVertices,
                                     const int     *_indices,  int numIndices,
                                     const int     *elementOffsets,
                                     int      numElements,
                                     const box3f &domain)
  {
    std::vector<Element> elements;
    for (int i=0;i<numElements;i++) {
      Element elt;
      elt.ofs0 = elementOffsets[i];
      if (elt.ofs0 != elementOffsets[i])
        throw std::runtime_error("not enough bits to encode element offset");

      int eltEnd
        = (i==(numElements-1))
        ? numIndices
        : elementOffsets[i+1];
      int numEltIndices = eltEnd - elt.ofs0;

      switch (numEltIndices) {
      case 4:
        elt.type = Element::TET;
        break;
      case 5:
        elt.type = Element::PYR;
        break;
      case 6:
        elt.type = Element::WED;
        break;
      case 8:
        elt.type = Element::HEX;
        break;
      default:
        throw std::runtime_error("non-supported element type with "
                                 +std::to_string(numEltIndices)+" indices");
      }
      elements.push_back(elt);
    }
    std::vector<vec4f> vertices(numVertices);
    std::copy(_vertices,_vertices+numVertices,vertices.data());
    std::vector<int> indices(numIndices);
    std::copy(_indices,_indices+numIndices,indices.data());
    ScalarField::SP sf
      = std::make_shared<UMeshField>(context,slot,
                                     vertices,
                                     indices,
                                     elements,
                                     domain);
    return sf;
  }
  
  void UMeshField::setVariables(OWLGeom geom)
  {
    ScalarField::setVariables(geom);
    
    owlGeomSetBuffer(geom,"umesh.vertices",verticesBuffer);
    owlGeomSetBuffer(geom,"umesh.indices",indicesBuffer);
    owlGeomSetBuffer(geom,"umesh.elements",elementsBuffer);
  }
  
  void UMeshField::DD::addVars(std::vector<OWLVarDecl> &vars, int base)
  {
    ScalarField::DD::addVars(vars,base);
    std::vector<OWLVarDecl> mine = 
      {
        { "umesh.vertices",    OWL_BUFPTR, base+OWL_OFFSETOF(DD,vertices) },
        { "umesh.indices" ,    OWL_BUFPTR, base+OWL_OFFSETOF(DD,indices) },
        { "umesh.elements",    OWL_BUFPTR, base+OWL_OFFSETOF(DD,elements) },
      };
    for (auto var : mine)
      vars.push_back(var);
  }

  VolumeAccel::SP UMeshField::createAccel(Volume *volume)
  {
#if 1
    const char *methodFromEnv = getenv("BARNEY_UMESH");
    std::string method = (methodFromEnv ? methodFromEnv : "DDA");

    if (method == "DDA" || method == "MCDDA" || method == "dda") {
      return std::make_shared<MCDDAVolumeAccel<UMeshCUBQLSampler>::Host>
        (this,volume,UMeshMC_ptx);
    }

    if (method == "MCRTX")
      return std::make_shared<MCRTXVolumeAccel<UMeshCUBQLSampler>::Host>
        (this,volume,UMeshMC_ptx);
    
    if (method == "OS" || method == "os")
      return std::make_shared<RTXObjectSpace::Host>
        (this,volume);
    
    if (method == "AWT" || method == "awt")
      return std::make_shared<UMeshAWT::Host>
        (this,volume);
    
    throw std::runtime_error("unknown BARNEY_UMESH accelerator method");
#else
    const char *methodFromEnv = getenv("BARNEY_UMESH");
    std::string method = (methodFromEnv ? methodFromEnv : "object-space");
    if (method == "macro-cells" || method == "spatial" || method == "mc")
      return std::make_shared<UMeshAccel_MC_CUBQL>(this,volume);
    else if (method == "AWT" || method == "awt")
      return std::make_shared<UMeshAWT>(this,volume);
    else if (method == "object-space" || method == "os")
      return std::make_shared<RTXObjectSpace>(this,volume);
    else
      throw std::runtime_error("found BARNEY_METHOD env-var, but didn't recognize its value. allowed values are 'awt', 'object-space', and 'macro-cells'");
#endif
  }


}
  
