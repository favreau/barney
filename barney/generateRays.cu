#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/DeviceContext.h"
#include "barney/Ray.h"

namespace barney {

  /*! generates a new wave-front of rays, to be written to
      'rayQueue[]', at (atomically incrementable) positoin
      *d_count. This kernel operates on *tiles* (not complete frames);
      the list of tiles to generate rays for is passed in 'tileDescs';
      there will be one cuda block per tile */
  __global__
  void g_generateRays(/*! the camera used for generating the rays */
                      Camera camera,
                      /*! a unique random number seed value for pixel
                          and lens jitter; probably just accumID */
                      int rngSeed,
                      /*! full frame buffer size, to check if a given
                          tile's pixel ID is still valid */
                      vec2i fbSize,
                      /*! pointer to a device-side int that tracks the
                          next write position in the 'write' ray
                          queue; can be atomically incremented on the
                          device */
                      int *d_count,
                      /*! pointer to device-side ray queue to write
                          newly generated raysinto */
                      Ray *rayQueue,
                      /*! tile descriptors for the tiles that the
                          frame buffer owns on this device; rays
                          should only get generated for these tiles */
                      TileDesc *tileDescs)
  {
    __shared__ int l_count;
    if (threadIdx.x == 0)
      l_count = 0;

    // ------------------------------------------------------------------
    __syncthreads();
    
    int tileID = blockIdx.x;
    
    vec2i tileOffset = tileDescs[tileID].lower;
    int ix = (threadIdx.x % tileSize) + tileOffset.x;
    int iy = (threadIdx.x / tileSize) + tileOffset.y;

    // bool dbg = ((ix == 0) || (ix == fbSize.x-1)) && ((iy==0) || (iy == fbSize.y-1));
    
    Ray ray;
    ray.org  = camera.lens_00;
    ray.dir
      = camera.dir_00
      + (ix+.5f)*camera.dir_du
      + (iy+.5f)*camera.dir_dv;
    ray.dir = normalize(ray.dir);

    ray.centerPixel = ((ix == fbSize.x/2) && (iy == fbSize.y/2));
    ray.dbg = ray.centerPixel;
    // if (dbg) {
    //   vec3f ctr = normalize(camera.dbg_vi - camera.dbg_vp);
    //   float angle = dot(ctr,ray.direction);
    //   printf("pixel (%4i %4i) org %.1f %.1f %.1f dir %6.3f %6.3f %6.3f ctr %6.3f %6.3f %6.3f angle %f\n",
    //          ix,iy,
    //          ray.origin.x,
    //          ray.origin.y,
    //          ray.origin.z,
    //          ray.direction.x,
    //          ray.direction.y,
    //          ray.direction.z,ctr.x,ctr.y,ctr.z,angle);
    // }
    
    ray.tMax = 1e30f;
    ray.instID  = -1;
    ray.geomID  = -1;
    ray.primID  = -1;
    ray.u       = 0.f;
    ray.v       = 0.f;
    ray.pixelID = tileID * (tileSize*tileSize) + threadIdx.x;
    Random rand(rngSeed,ray.pixelID);
    ray.rngSeed = rand.state;
    ray.hadHit = false;

    const float t = (iy+.5f)/float(fbSize.y);
    ray.color = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
    
    int pos = -1;
    if (ix < fbSize.x && iy < fbSize.y) 
      pos = atomicAdd(&l_count,1);

    // ------------------------------------------------------------------
    __syncthreads();
    if (threadIdx.x == 0) 
      l_count = atomicAdd(d_count,l_count);
    
    // ------------------------------------------------------------------
    __syncthreads();
    if (pos >= 0) 
      rayQueue[l_count + pos] = ray;
  }
  
}
