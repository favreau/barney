#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/fb/TiledFB.h"
#include "barney/fb/FrameBuffer.h"
// #include <optix.h>
// #include <optix_function_table.h>
// #include <optix_stubs.h>

namespace BARNEY_NS {

  TiledFB::SP TiledFB::create(Device *device, FrameBuffer *owner)
  {
    return std::make_shared<TiledFB>(device, owner);
  }

  TiledFB::TiledFB(Device *device, FrameBuffer *owner)
    : device(device),
      owner(owner)
  {}

  TiledFB::~TiledFB()
  { free(); }

  void TiledFB::free()
  {
    SetActiveGPU forDuration(device);
    if (accumTiles)  {
      device->rtc->freeMem(accumTiles);
      accumTiles = nullptr;
    }
    if (compressedTiles) {
      device->rtc->freeMem(compressedTiles);
      compressedTiles = nullptr;
    }
    if (tileDescs) {
      device->rtc->freeMem(tileDescs);
      tileDescs = nullptr;
    }
  }

  struct SetTileCoords {
    /* kernel ARGS */
    TileDesc *tileDescs;
    int numActiveTiles;
    vec2i numTiles;
    int globalIndex;
    int globalIndexStep;

    /* kernel CODE */
    inline __rtc_device
    void run(const rtc::ComputeInterface &rtCore);
  };

#if RTC_DEVICE_CODE
  /* kernel CODE */
  inline __rtc_device
  void SetTileCoords::run(const rtc::ComputeInterface &rtCore)
  {
    int tid
      = rtCore.getThreadIdx().x
      + rtCore.getBlockIdx().x*rtCore.getBlockDim().x;
    if (tid >= numActiveTiles)
      return;
        
    int tileID = tid * globalIndexStep + globalIndex;
        
    int tile_x = tileID % numTiles.x;
    int tile_y = tileID / numTiles.x;
    tileDescs[tid].lower = vec2i(tile_x*tileSize,tile_y*tileSize);
  }
#endif
  

  void TiledFB::resize(vec2i newSize)
  {
    free();
    SetActiveGPU forDuration(device);

    numPixels = newSize;
    numTiles  = divRoundUp(numPixels,vec2i(tileSize));
    numActiveTiles
      = device
      ? divRoundUp(std::max(0,numTiles.x*numTiles.y - device->globalIndex),
                   device->globalIndexStep)
      : 0;
    auto rtc = device->rtc;
    accumTiles
      = (AccumTile *)rtc->allocMem(numActiveTiles * sizeof(AccumTile));
    compressedTiles
      = (CompressedTile *)rtc->allocMem(numActiveTiles * sizeof(CompressedTile));
    tileDescs
      = (TileDesc *)rtc->allocMem(numActiveTiles * sizeof(TileDesc));
    SetTileCoords args = {
      tileDescs,
      numActiveTiles,
      numTiles,
      device->globalIndex,
      device->globalIndexStep
    };
    if (numActiveTiles > 0)
      device->setTileCoords
        ->launch(divRoundUp(numActiveTiles,1024),1024,
                 &args);
  }

  // ==================================================================


  struct CompressTiles {
    CompressedTile *compressedTiles;
    AccumTile      *accumTiles;
    float           accumScale;
    int             globalIdx;
    int             globalIdxStep;

    inline __rtc_device
    void run(const rtc::ComputeInterface &rtCore);
  };

#if RTC_DEVICE_CODE
  inline __rtc_device
  void CompressTiles::run(const rtc::ComputeInterface &rtCore)
  { 
    int pixelID = rtCore.getThreadIdx().x;
    int tileID  = rtCore.getBlockIdx().x;

    vec4f color = vec4f(accumTiles[tileID].accum[pixelID])*accumScale;
    vec4f org = color;
    float scale = reduce_max(color);
    color *= 1.f/scale;
    compressedTiles[tileID].scale[pixelID] = scale;
    compressedTiles[tileID].normal[pixelID]
      .set(accumTiles[tileID].normal[pixelID]);

      
    uint32_t rgba32
      = make_rgba(color);

    compressedTiles[tileID].rgba[pixelID]
      = rgba32;
    compressedTiles[tileID].depth[pixelID]
      = accumTiles[tileID].depth[pixelID];
  }
#endif
  
  /*! write this tiledFB's tiles into given "compressed" frame buffer
    (i.e., a plain 2D array of numPixels.x*numPixels.y RGBA8
    pixels) */
  void TiledFB::finalizeTiles_launch()
  {
    SetActiveGPU forDuration(device);
    if (numActiveTiles > 0) {
      CompressTiles args = {
        compressedTiles,
        accumTiles,
        1.f/(owner->accumID),
        device->globalIndex,
        device->globalIndexStep,
      };
      device->compressTiles
        ->launch(numActiveTiles,pixelsPerTile,
                 &args);       
    }
  }
  
  RTC_EXPORT_COMPUTE1D(setTileCoords,SetTileCoords);
  RTC_EXPORT_COMPUTE1D(compressTiles,CompressTiles);
}




  
