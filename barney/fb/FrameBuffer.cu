#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/fb/FrameBuffer.h"
#include <hip/hip_runtime.h>
#if BARNEY_HAVE_OIDN
# include <OpenImageDenoise/oidn.h>
#endif

namespace barney {

  inline __device__ float saturate(float f, float lo=0.f, float hi=1.f)
  { return max(lo,min(f,hi)); }
  
  inline __device__ float from_8bit(uint8_t v) {
    return float(v) * (1.f/255.f);
  }
  
  inline __device__ vec4f from_8bit(uint32_t v) {
    return vec4f(from_8bit(uint8_t((v >> 0)&0xff)),
                 from_8bit(uint8_t((v >> 8)&0xff)),
                 from_8bit(uint8_t((v >> 16)&0xff)),
                 from_8bit(uint8_t((v >> 24)&0xff)));
  }
  
  inline __device__ float linear_to_srgb(float x) {
    if (x <= 0.0031308f) {
      return 12.92f * x;
    }
    return 1.055f * pow(x, 1.f/2.4f) - 0.055f;
  }

  inline __device__ uint32_t _make_8bit(const float f)
  {
    return min(255,max(0,int(f*256.f)));
  }

  inline __device__ uint32_t make_rgba8(const vec4f color, bool dbg=false)
  {
    if (dbg)
      printf("col %f %f %f %f\n",
             color.x,
             color.y,
             color.z,
             color.w);
    uint32_t r = _make_8bit(color.x);
    uint32_t g = _make_8bit(color.y);
    uint32_t b = _make_8bit(color.z);
    uint32_t a = 0xff;//_make_8bit(color.w);
    uint32_t ret =
      (r << 0) |
      (g << 8) |
      (b << 16) |
      (a << 24);
    if (dbg) printf("%x %x %x %x all %x\n",
                    r,g,b,a,ret);
    return ret;
      // (_make_8bit(color.x) << 0) +
      // (_make_8bit(color.y) << 8) +
      // (_make_8bit(color.z) << 16) +
      // (_make_8bit(color.w) << 24);
  }
  
  inline __device__ float clamp(float f) { return min(1.f,max(0.f,f)); }

  __global__ void copyPixels(vec2i numPixels,
                             float4 *out,
                             vec3f *in_color,
                             float *in_alpha)
  {
    int ix = threadIdx.x+blockIdx.x*blockDim.x;
    int iy = threadIdx.y+blockIdx.y*blockDim.y;
    if (ix >= numPixels.x) return;
    if (iy >= numPixels.y) return;
    int idx = ix + numPixels.x*iy;
    vec3f color = in_color[idx];
    float alpha = in_alpha[idx];
    out[idx] = vec4f(color,alpha);
  }
  
  struct DenoiserNone : public Denoiser {
    DenoiserNone(FrameBuffer *fb) : Denoiser(fb) {};
    virtual ~DenoiserNone() {}
    void resize() override { }
                                      
    void run() override
    {
      vec2i bs(8,8);
      copyPixels<<<divRoundUp(fb->numPixels,bs),bs>>>
        (fb->numPixels,fb->denoisedColor,fb->linearColor,fb->linearAlpha);
    }
  };

#if !BARNEY_DISABLE_DENOISING
#if BARNEY_HAVE_OIDN
  // __global__ void g_oidnWriteReults(vec2i numPixels,
  //                                   float4 *out,
  //                                   float3 *in_color,
  //                                   float  *in_alpha)
  // {
  //   xxx
  // }
  
  struct DenoiserOIDN : public Denoiser {
    DenoiserOIDN(FrameBuffer *fb)
      : Denoiser(fb)
    {
      int devID = 0;
      hipStream_t stream = 0;
      device = 
        oidnNewCUDADevice(&devID,&stream,1);
      oidnCommitDevice(device);
      filter = oidnNewFilter(device,"RT");
    }
    virtual ~DenoiserOIDN()
    {
      if (colorBuf)  oidnReleaseBuffer(colorBuf);
      if (normalBuf) oidnReleaseBuffer(normalBuf);
      if (outputBuf) oidnReleaseBuffer(outputBuf);
      if (denoiserOutput) BARNEY_CUDA_CALL_NOTHROW(Free(denoiserOutput));
      oidnReleaseFilter(filter);
      oidnReleaseDevice(device);
    }
    
    void resize() override
    {
      vec2i numPixels = fb->numPixels;
      if (denoiserOutput)
        BARNEY_CUDA_CALL(Free(denoiserOutput));
      BARNEY_CUDA_CALL(Malloc((void **)&denoiserOutput,
                              numPixels.x*numPixels.y*sizeof(float3)));
      // if (fb->linearColor)
      //   BARNEY_CUDA_CALL(Free(denoiserInput));
      // BARNEY_CUDA_CALL(Malloc((void **)&denoiserInput,
      //                         numPixels.x*numPixels.y*sizeof(*denoiserInput)));
    
      // if (denoiserAlpha)
      //   BARNEY_CUDA_CALL(Free(denoiserAlpha));
      // BARNEY_CUDA_CALL(Malloc((void **)&denoiserAlpha,
      //                         numPixels.x*numPixels.y*sizeof(*denoiserAlpha)));
      // if (denoiserOutput)
      //   BARNEY_CUDA_CALL(Free(denoiserOutput));
      // BARNEY_CUDA_CALL(Malloc((void **)&denoiserOutput,
      //                         numPixels.x*numPixels.y*sizeof(*denoiserOutput)));
      
      // if (denoiserNormal)
      //   BARNEY_CUDA_CALL(Free(denoiserNormal));

      // BARNEY_CUDA_CALL(Malloc((void **)&denoiserNormal,
      //                         numPixels.x*numPixels.y*sizeof(*denoiserNormal)));
      if (colorBuf)  oidnReleaseBuffer(colorBuf);
      if (normalBuf) oidnReleaseBuffer(normalBuf);
      if (outputBuf) oidnReleaseBuffer(outputBuf);
      colorBuf
        = oidnNewSharedBuffer(device, fb->linearColor,
                              numPixels.x*numPixels.y*sizeof(*fb->linearColor));
      normalBuf
        = oidnNewSharedBuffer(device, fb->linearNormal,
                              numPixels.x*numPixels.y*sizeof(*fb->linearNormal));
      outputBuf
        = oidnNewSharedBuffer(device, denoiserOutput,
                              numPixels.x*numPixels.y*sizeof(*denoiserOutput));
      oidnSetFilterImage(filter,"color",colorBuf,
                         OIDN_FORMAT_FLOAT3,numPixels.x,numPixels.y,0,0,0);
      // oidnSetFilterImage(filter,"normal",normalBuf,
      //                    OIDN_FORMAT_FLOAT3,numPixels.x,numPixels.y,0,0,0);
      // oidnSetFilterImage(filter,"albedo",normalBuf,
                         // OIDN_FORMAT_FLOAT3,numPixels.x,numPixels.y,0,0,0);
      oidnSetFilterImage(filter,"output",outputBuf,
                         OIDN_FORMAT_FLOAT3,numPixels.x,numPixels.y,0,0,0);
      oidnSetFilterBool(filter,"hdr",true);
      oidnCommitFilter(filter);
    }
    void run() override
    {
      oidnExecuteFilter(filter);
      vec2i bs(8,8);
      copyPixels<<<divRoundUp(fb->numPixels,bs),bs>>>
        (fb->numPixels,fb->denoisedColor,denoiserOutput,fb->linearAlpha);
      const char *error;
      oidnGetDeviceError(device,&error);
      if (error)
        PRINT(error);
    }
    
    vec3f    *denoiserOutput   = 0;
    
    OIDNBuffer outputBuf = 0; 
    OIDNBuffer normalBuf = 0; 
    OIDNBuffer colorBuf = 0; 
    OIDNDevice device = 0;
    OIDNFilter filter = 0;
  };
#endif
  
#if OPTIX_VERSION >= 80000  
  struct DenoiserOptix : public Denoiser {
    DenoiserOptix(FrameBuffer *fb)
      : Denoiser(fb)
    {
      denoiserOptions.guideAlbedo = 0;
      denoiserOptions.guideNormal = 1;
      denoiserOptions.denoiseAlpha
        = OPTIX_DENOISER_ALPHA_MODE_DENOISE;
        
      auto device = fb->context->getDevice(0);

      OptixDeviceContext optixContext
        = owlContextGetOptixContext(device->devGroup->owl,0);
      optixDenoiserCreate(optixContext,
                          OPTIX_DENOISER_MODEL_KIND_HDR,
                          &denoiserOptions,
                          &denoiser);
    }      
    virtual ~DenoiserOptix();
    void resize() override
    {
      Denoiser::resize();
      vec2i numPixels = fb->numPixels;
      // if (denoiserInput)
      //   BARNEY_CUDA_CALL(Free(denoiserInput));
      // BARNEY_CUDA_CALL(Malloc((void **)&denoiserInput,
      //                         numPixels.x*numPixels.y*sizeof(*denoiserInput)));
    
      if (denoiserOutput)
        BARNEY_CUDA_CALL(Free(denoiserOutput));
      BARNEY_CUDA_CALL(Malloc((void **)&denoiserOutput,
                              numPixels.x*numPixels.y*sizeof(*denoiserOutput)));
      
      // if (fb->denoiserNormal)
      //   BARNEY_CUDA_CALL(Free(denoiserNormal));
      // BARNEY_CUDA_CALL(Malloc((void **)&denoiserNormal,
      //                         numPixels.x*numPixels.y*sizeof(*denoiserNormal)));
      denoiserSizes.overlapWindowSizeInPixels = 0;
      optixDenoiserComputeMemoryResources(/*const OptixDenoiser */
                                          denoiser,
                                          // unsigned int        outputWidth,
                                          numPixels.x,
                                          // unsigned int        outputHeight,
                                          numPixels.y,
                                          // OptixDenoiserSizes* returnSizes
                                          &denoiserSizes
                                          );
      if (denoiserScratch) {
        BARNEY_CUDA_CALL(Free(denoiserScratch));
        denoiserScratch = 0;
      }
      BARNEY_CUDA_CALL(Malloc(&denoiserScratch,
                              denoiserSizes.withoutOverlapScratchSizeInBytes));
      if (denoiserState) {
        BARNEY_CUDA_CALL(Free(denoiserState));
        denoiserState = 0;
      }
      BARNEY_CUDA_CALL(Malloc(&denoiserState,
                              denoiserSizes.stateSizeInBytes));
      optixDenoiserSetup(// OptixDenoiser denoiser,
                         denoiser,
                         // hipStream_t      stream,
                         0,//device->launchStream,
                         // unsigned int  inputWidth,
                         numPixels.x,
                         // unsigned int  inputHeight,
                         numPixels.y,
                         // hipDeviceptr_t   denoiserState,
                         (hipDeviceptr_t)denoiserState,
                         // size_t        denoiserStateSizeInBytes,
                         denoiserSizes.stateSizeInBytes,
                         // hipDeviceptr_t   scratch,
                         (hipDeviceptr_t)denoiserScratch,
                         //size_t        scratchSizeInBytes
                         denoiserSizes.withoutOverlapScratchSizeInBytes
                         );
    }
    void run() override
    {
      OptixDenoiserGuideLayer guideLayer = {};
      OptixDenoiserLayer layer = {};
      auto numPixels = fb->numPixels;
      layer.input.format = OPTIX_PIXEL_FORMAT_FLOAT3;
      layer.input.rowStrideInBytes = numPixels.x*sizeof(float3);
      layer.input.pixelStrideInBytes = sizeof(float4);
      layer.input.width = numPixels.x;
      layer.input.height = numPixels.y;
      layer.input.data = (hipDeviceptr_t)fb->linearColor;//denoiserInput;

      guideLayer.normal = layer.input;
      guideLayer.normal.data = (hipDeviceptr_t)fb->linearNormal;//denoiserNormal;
      layer.output = layer.input;
      layer.output.data = (hipDeviceptr_t)denoiserOutput;

      OptixDenoiserParams denoiserParams = {};

      optixDenoiserInvoke
        (
         denoiser,
         0,
         &denoiserParams,
         (hipDeviceptr_t)denoiserState,
         denoiserSizes.stateSizeInBytes,
         &guideLayer,
         &layer,
         1,
         0,
         0,
         (hipDeviceptr_t)denoiserScratch,
         denoiserSizes.withoutOverlapScratchSizeInBytes
         );
    }

    OptixDenoiser        denoiser = {};
    OptixDenoiserOptions denoiserOptions;
    void                *denoiserScratch = 0;
    void                *denoiserState   = 0;
    OptixDenoiserSizes   denoiserSizes;
    
    // float4              *denoiserInput   = 0;
    float3              *denoiserOutput  = 0;
    // float4              *denoiserNormal  = 0;
  };
#endif
#endif

  Denoiser::SP Denoiser::create(FrameBuffer *fb)
  {
#if !BARNEY_DISABLE_DENOISING
# if BARNEY_HAVE_OIDN
    return std::make_shared<DenoiserOIDN>(fb);
# endif
# if OPTIX_VERSION >= 80000
    return std::make_shared<DenoiserOptix>(fb);
# endif
#endif
    return std::make_shared<DenoiserNone>(fb);
  }
  
  
  FrameBuffer::FrameBuffer(Context *context, const bool isOwner)
    : Object(context),
      isOwner(isOwner)
  {
    perDev.resize(context->devices.size());
    for (int localID=0;localID<context->devices.size();localID++) {
      perDev[localID]
        = TiledFB::create(context->getDevice(localID),this);
    }
  }

  FrameBuffer::~FrameBuffer()
  {
    freeResources();
    denoiser = 0;
  }

  bool FrameBuffer::set1i(const std::string &member, const int &value)
  {
    if (member == "showCrosshairs") {
      showCrosshairs = value;
      return true;
    }
    return false;
  }

  void FrameBuffer::freeResources()
  {
    if (denoisedColor) {
      BARNEY_CUDA_CALL(Free(denoisedColor));
      denoisedColor = 0;
    }
    if (linearColor) {
      BARNEY_CUDA_CALL(Free(linearColor));
      linearColor = 0;
    }
    if (linearAlpha) {
      BARNEY_CUDA_CALL(Free(linearAlpha));
      linearAlpha = 0;
    }
    if (linearDepth) {
      BARNEY_CUDA_CALL(Free(linearDepth));
      linearDepth = 0;
    }
    if (linearNormal) {
      BARNEY_CUDA_CALL(Free(linearNormal));
      linearNormal = 0;
    }
  }

  template<bool SRGB>
  __global__
  void toFixed8(uint32_t *out,
                float4 *in,
                vec2i numPixels)
  {
    int ix = threadIdx.x+blockIdx.x*blockDim.x;
    if (ix >= numPixels.x) return;
    int iy = threadIdx.y+blockIdx.y*blockDim.y;
    if (iy >= numPixels.y) return;
    int idx = ix+numPixels.x*iy;

    bool dbg = 0;// (ix == 0 && iy == 0);
    
    float4 v = in[idx];
    v.x = clamp(v.x);
    v.y = clamp(v.y);
    v.z = clamp(v.z);
    if (SRGB) {
      // this doesn't make sense - the color channel has ALREADY been
      // gamma-corrected in tonemap()!?
      v.x = linear_to_srgb(v.x);
      v.y = linear_to_srgb(v.y);
      v.z = linear_to_srgb(v.z);
    }
    out[idx] = make_rgba8(v,dbg);
  }

  __global__ void toneMap(float4 *color, vec2i numPixels)
  {
    int ix = threadIdx.x+blockIdx.x*blockDim.x;
    if (ix >= numPixels.x) return;
    int iy = threadIdx.y+blockIdx.y*blockDim.y;
    if (iy >= numPixels.y) return;
    int idx = ix+numPixels.x*iy;

    float4 v = color[idx];
    // if (ix == 0 && iy == 0)
    //   printf("color %f %f %f %f\n",
    //          v.x,v.y,v.z,v.w);
#if 1
    v.x = linear_to_srgb(v.x);
    v.y = linear_to_srgb(v.y);
    v.z = linear_to_srgb(v.z);
#elif 0
    v.x = sqrtf(v.x);
    v.y = sqrtf(v.y);
    v.z = sqrtf(v.z);
#else
    // v.x = linear_to_srgb(v.x);
    // v.y = linear_to_srgb(v.y);
    // v.z = linear_to_srgb(v.z);
#endif
    color[idx] = v;
  }


  void FrameBuffer::finalizeFrame()
  {
    dirty = true;
    ownerGatherCompressedTiles();
    if (isOwner) {
      unpackTiles();
    }
    
  }


  __global__ void g_unpackTiles(vec2i numPixels,
                                vec3f *colors,
                                float *alphas,
                                vec3f *normals,
                                float *depths,
                                CompressedTile *tiles,
                                TileDesc *descs)
  {
    int tileIdx = blockIdx.x;

    const CompressedTile &tile = tiles[tileIdx];
    const TileDesc        desc = descs[tileIdx];
    
    int subIdx = threadIdx.x;
    int iix = subIdx % tileSize;
    int iiy = subIdx / tileSize;
    int ix = desc.lower.x + iix;
    int iy = desc.lower.y + iiy;
    if (ix >= numPixels.x) return;
    if (iy >= numPixels.y) return;
    int idx = ix + numPixels.x*iy;
    
    vec4f rgba = from_8bit(tile.rgba[subIdx]);
    float alpha = rgba.w;
    float scale = float(tile.scale[subIdx]);
    vec3f color = vec3f(rgba.x,rgba.y,rgba.z)*scale;
    // if (ix == 0 && iy == 0)
    //   printf("rgba %f %f %f scale %f color %f %f %f\n",
    //          rgba.x,rgba.y,rgba.z,scale,
    //          color.x,color.y,color.z);
    vec3f normal = tile.normal[subIdx].get3f();
    float depth = tile.depth[subIdx];

    colors[idx] = color;
    alphas[idx] = alpha;
    depths[idx] = depth;
    normals[idx] = normal;
  }
  
  void FrameBuffer::unpackTiles()
  {
    g_unpackTiles<<<gatheredTilesOnOwner.numActiveTiles,pixelsPerTile>>>
      (numPixels,
       linearColor,
       linearAlpha,
       linearNormal,
       linearDepth,
       gatheredTilesOnOwner.compressedTiles,
       gatheredTilesOnOwner.tileDescs);
  }

  void FrameBuffer::read(BNFrameBufferChannel channel,
                         void *hostPtr,
                         BNDataType requestedFormat)
  {
    if (!isOwner) return;

    if (dirty) {
      denoiser->run();
      vec2i bs(8,8);
      toneMap<<<divRoundUp(numPixels,bs),bs>>>(denoisedColor,numPixels);
      BARNEY_CUDA_SYNC_CHECK();
      dirty = false;
    }
    if (channel == BN_FB_DEPTH && hostPtr && linearDepth) {
      if (requestedFormat != BN_FLOAT)
        throw std::runtime_error("can only read depth channel as BN_FLOAT format");
      if (!linearDepth)
        throw std::runtime_error("requesting to read depth channel, but didn't create one");
      BARNEY_CUDA_CALL(Memcpy(hostPtr,linearDepth,
                              numPixels.x*numPixels.y*sizeof(float),hipMemcpyDefault));
      BARNEY_CUDA_SYNC_CHECK();
      return;
    }

    if (!hostPtr) return;
    
    if (channel != BN_FB_COLOR)
      throw std::runtime_error("trying to read un-known channel!?");

    BARNEY_CUDA_SYNC_CHECK();
    
    switch(requestedFormat) {
    // case BN_FLOAT4_RGBA:
    //   BARNEY_CUDA_CALL(Memcpy(hostPtr,finalColor,
    //                           numPixels.x*numPixels.y*sizeof(float4),hipMemcpyDefault));
    //   break;
    case BN_FLOAT4_RGBA: {
      BARNEY_CUDA_CALL(Memcpy(hostPtr,denoisedColor,
                              numPixels.x*numPixels.y*sizeof(float4),hipMemcpyDefault));
    } break;
    case BN_UFIXED8_RGBA: {
      uint32_t *asFixed8;
      BARNEY_CUDA_SYNC_CHECK();
      BARNEY_CUDA_CALL(MallocAsync((void**)&asFixed8,
                                   numPixels.x*numPixels.y*sizeof(uint32_t),0));
      BARNEY_CUDA_SYNC_CHECK();
      vec2i bs(8,8);
      toFixed8<false>
        <<<divRoundUp(numPixels,bs),bs>>>
        (asFixed8,denoisedColor,numPixels);
      BARNEY_CUDA_CALL(Memcpy(hostPtr,asFixed8,
                              numPixels.x*numPixels.y*sizeof(uint32_t),
                              hipMemcpyDefault));
      BARNEY_CUDA_CALL(FreeAsync(asFixed8,0));
    } break;
    case BN_UFIXED8_RGBA_SRGB: {
      uint32_t *asFixed8;
      BARNEY_CUDA_SYNC_CHECK();
      BARNEY_CUDA_CALL(MallocAsync((void**)&asFixed8,
                                   numPixels.x*numPixels.y*sizeof(uint32_t),0));
      BARNEY_CUDA_SYNC_CHECK();
      vec2i bs(8,8);
      toFixed8<true>
        <<<divRoundUp(numPixels,bs),bs>>>
        (asFixed8,denoisedColor,numPixels);
      BARNEY_CUDA_CALL(Memcpy(hostPtr,asFixed8,
                              numPixels.x*numPixels.y*sizeof(uint32_t),
                              hipMemcpyDefault));
      BARNEY_CUDA_CALL(FreeAsync(asFixed8,0));
    } break;
    default:
      throw std::runtime_error("requested to read color channel in un-supported format #"
                               +std::to_string((int)requestedFormat));
    };
  }
  
  void FrameBuffer::resize(vec2i size,
                           uint32_t channels)
  {
    for (auto &pd: perDev)
      pd->resize(size);
    
    freeResources();
    numPixels = size;

    if (isOwner) {
      BARNEY_CUDA_CALL(Malloc(&denoisedColor,
                              numPixels.x*numPixels.y*sizeof(*denoisedColor)));
      BARNEY_CUDA_CALL(Malloc(&linearDepth,
                              numPixels.x*numPixels.y*sizeof(*linearDepth)));
      BARNEY_CUDA_CALL(Malloc(&linearColor,
                              numPixels.x*numPixels.y*sizeof(*linearColor)));
      BARNEY_CUDA_CALL(Malloc(&linearAlpha,
                              numPixels.x*numPixels.y*sizeof(*linearAlpha)));
      BARNEY_CUDA_CALL(Malloc(&linearNormal,
                              numPixels.x*numPixels.y*sizeof(*linearNormal)));
      
      if (!denoiser) denoiser = Denoiser::create(this);
      denoiser->resize();
    }
  }
    

}
