#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/unstructured/QuickClusters.h"
#include "owl/owl_device.h"

namespace barney {

  inline __device__
  vec4f DeviceXF::map(float s) const
  {
    float f = (s-domain.lower)/domain.span();
    f = clamp(f,0.f,1.f);
    f *= (numValues-1);
    int idx0 = clamp(int(f),0,numValues-1);
    int idx1 = clamp(idx0+1,0,numValues-1);
    f -= idx0;
    vec4f r = (1.f-f)*(vec4f)values[idx0]+f*(vec4f)values[idx1];
    r.w *= baseDensity;
    return r;
  }

  struct MeshSampler
  {
    inline __device__
    MeshSampler(const UMeshQC::DD &mesh) : mesh(mesh) {}

    inline __device__ void operator=(const MeshSampler &other)
    {
      P = other.P;
      mapped = other.mapped;
      scalar = other.scalar;
    }

    // inline __device__
    // void computeGradient()
    // {
    // };
    inline __device__
    bool sample();
    inline __device__
    bool sampleAndMap();
    inline __device__
    bool sampleAndMap(int elts_begin, int elts_end);
      
    UMeshQC::Element element;
    vec3f P;

    float scalar;
    /*! color- and alpha-mapped sample */
    vec4f mapped;
    // vec3f gradient;

    const UMeshQC::DD &mesh;    
  };

  template<typename T>
  inline __device__
  void swap(T &a, T &b) { T c = a; a = b; b = c; }

  inline __device__
  float safeDiv(float a, float b) { return (b==0.f)?0.f:(a/b); }
  
  inline __device__
  float doPlane(vec3f P, vec3f a, vec3f b, vec3f c)
  {
    vec3f n = cross(b-a,c-a);
    return dot(P-a,n);
  }
  
  
  inline __device__
  bool evaluateTet(vec3f v0, float s0,
                   vec3f v1, float s1,
                   vec3f v2, float s2,
                   vec3f v3, float s3,
                   float &scalar,
                   vec3f P)
  {
    float a = doPlane(v3,v0,v1,v2);
    if (a == 0.f) return false;
    if (a < 0.f) {
      swap(v0,v1);
      swap(s0,s1);
      a = -a;
    }
    // clipPlane(v0,v1,v2);
    // // if (tRange.empty()) return;
    // clipPlane(v0,v3,v1);
    // // if (tRange.empty()) return;
    // clipPlane(v0,v2,v3);
    // // if (tRange.empty()) return;
    // clipPlane(v1,v3,v2);

    float w3 = doPlane(P, v0,v1,v2)/a; if (w3 < 0.f) return false;
    float w2 = doPlane(v3,v0,v1,P )/a; if (w2 < 0.f) return false;
    float w1 = doPlane(v3,v0,P ,v2)/a; if (w1 < 0.f) return false;
    float w0 = doPlane(v3,P ,v1,v2)/a; if (w0 < 0.f) return false;

    // float ww = w0+w1+w2+w3;
    // if (ww < .98 || ww > 1.02)
    //   printf("weird w %f\n",ww);
    // float w2 = doPlane(P,v0,v3,v1)/a; if (w2 < 0.f) return false;
    // float w1 = doPlane(P,v0,v2,v3)/a; if (w1 < 0.f) return false;
    // float w0 = doPlane(P,v1,v3,v2)/a; if (w0 < 0.f) return false;
    scalar = w0*s0 + w1*s1 + w2*s2 + w3*s3;
    return true;
  }

  inline __device__
  bool MeshSampler::sample() 
  {
    if (element.type == UMeshQC::TET) {
      auto indices = mesh.tetIndices[element.ID];
      vec4f a = mesh.vertices[indices.x];
      vec4f b = mesh.vertices[indices.y];
      vec4f c = mesh.vertices[indices.z];
      vec4f d = mesh.vertices[indices.w];
      return evaluateTet(getPos(a),a.w,
                         getPos(b),b.w,
                         getPos(c),c.w,
                         getPos(d),d.w,
                         scalar, P);
    }
    return false;
  }
  
  inline __device__
  bool MeshSampler::sampleAndMap() 
  {
    if (element.type == UMeshQC::TET) {
      auto indices = mesh.tetIndices[element.ID];
      vec4f a = mesh.vertices[indices.x];
      vec4f b = mesh.vertices[indices.y];
      vec4f c = mesh.vertices[indices.z];
      vec4f d = mesh.vertices[indices.w];
      if (!evaluateTet(getPos(a),a.w,
                       getPos(b),b.w,
                       getPos(c),c.w,
                       getPos(d),d.w,
                       scalar, P))
        return false;
      mapped = mesh.xf.map(scalar);
      // gradient
      //   = (mesh.xf.map(a.w).w-mapped.w)*(getPos(a)-P)
      //   + (mesh.xf.map(b.w).w-mapped.w)*(getPos(b)-P)
      //   + (mesh.xf.map(c.w).w-mapped.w)*(getPos(c)-P)
      //   + (mesh.xf.map(d.w).w-mapped.w)*(getPos(d)-P);
      
      return true;
    }
    return false;
  }
  
  inline __device__
  bool MeshSampler::sampleAndMap(int elts_begin, int elts_end) 
  {
    for (int i=elts_begin;i<elts_end;i++) {
      element = mesh.elements[i];
      if (sampleAndMap())
        return true;
    }
    return false;
  }


  
      
  
  inline __device__
  float DeviceXF::majorant(range1f r, bool dbg) const
  {
    float f_lo = (r.lower-domain.lower)/domain.span();
    float f_hi = (r.upper-domain.lower)/domain.span();
    f_lo = clamp(f_lo,0.f,1.f);
    f_hi = clamp(f_hi,0.f,1.f);
    f_lo *= (numValues-1);
    f_hi *= (numValues-1);
    int idx0 = clamp(int(f_lo),0,numValues-1);
    int idx1 = clamp(int(f_hi)+1,0,numValues-1);
    float m = 0.f;
    if (dbg) printf("maj range %f %f indices %i %i\n",r.lower,r.upper,idx0,idx1);
    for (int i=idx0;i<=idx1;i++)
      m = max(m,values[i].w);
    // printf("maj [%f %f] domain [%f %f]-> idx [%i %i] max %f dens %f\n",
    //        r.lower,r.upper,domain.lower,domain.upper,idx0,idx1,m,baseDensity);
    return m * baseDensity;
  }

  inline __device__
  bool boxTest(float &t0, float &t1,
               box3f box,
               const vec3f org,
               const vec3f dir)
  {
    vec3f t_lo = (box.lower - org) * rcp(dir);
    vec3f t_hi = (box.upper - org) * rcp(dir);
    vec3f t_nr = min(t_lo,t_hi);
    vec3f t_fr = max(t_lo,t_hi);
    t0 = max(t0,reduce_max(t_nr));
    t1 = min(t1,reduce_min(t_fr));
    return t0 < t1;
  }

  inline __device__
  bool boxTest(float &t0, float &t1,
               box4f box,
               const vec3f org,
               const vec3f dir)
  {
    return boxTest(t0,t1,box3f({box.lower.x,box.lower.y,box.lower.z},
                               {box.upper.x,box.upper.y,box.upper.z}),
                   org,dir);
  }
  
  OPTIX_BOUNDS_PROGRAM(UMeshQCBounds)(const void *geomData,                
                                      owl::common::box3f &bounds,  
                                      const int32_t primID)
  {
    const auto &self = *(const UMeshQC::DD *)geomData;

    box4f clusterBounds;
    int begin = primID * UMeshQC::clusterSize;
    int end   = min(begin+UMeshQC::clusterSize,self.numElements);
    for (int i=begin;i<end;i++) {
      UMeshQC::Element elt = self.elements[i];
      clusterBounds.extend(self.getBounds(elt));
    }

    bool dbg = primID < 10;
    
    bounds = getBox(clusterBounds);

    if (dbg) printf("clusterbounds %f %f %f:%f - %f %f %f:%f, xfnum %i\n",
                    clusterBounds.lower.x,
                    clusterBounds.lower.y,
                    clusterBounds.lower.z,
                    clusterBounds.lower.w,
                    clusterBounds.upper.x,
                    clusterBounds.upper.y,
                    clusterBounds.upper.z,
                    clusterBounds.upper.w,
                    self.xf.numValues);
                    
    Cluster &cluster = self.clusters[primID];
    cluster.bounds = clusterBounds;
    
    if (self.xf.numValues > 0) {
      cluster.majorant = self.xf.majorant(getRange(clusterBounds),dbg);
      if (dbg) printf("domain %f %f majorant is %f\n",self.xf.domain.lower,self.xf.domain.upper,
                      cluster.majorant);
      if (cluster.majorant == 0.f)
        bounds = box3f(bounds.center());
    }
    
    // if (length(bounds.span())>30) {
    // printf("bounds %f %f %f : %f %f %f\n",
    //        bounds.lower.x,
    //        bounds.lower.y,
    //        bounds.lower.z,
    //        bounds.upper.x,
    //        bounds.upper.y,
    //        bounds.upper.z);
    // }
  }

  OPTIX_CLOSEST_HIT_PROGRAM(UMeshQCCH)()
  {
    auto &ray = owl::getPRD<Ray>();
    auto &self = owl::getProgramData<UMeshQC::DD>();
    int primID = optixGetPrimitiveIndex();

    // ray.hadHit = true;
    // ray.color = .8f;//owl::randomColor(primID);
    ray.tMax = optixGetRayTmax();

  }

  OPTIX_INTERSECT_PROGRAM(UMeshQCIsec)()
  {
    const int primID = optixGetPrimitiveIndex();
    const auto &self
      = owl::getProgramData<UMeshQC::DD>();
    
    const vec3f org  = optixGetObjectRayOrigin();
    const vec3f dir  = optixGetObjectRayDirection();
    float ray_t0     = optixGetRayTmin();
    float ray_t1     = optixGetRayTmax();

    float hit_t = INFINITY;
    
    Cluster cluster = self.clusters[primID];
    float t0 = ray_t0;
    float t1 = min(ray_t1,hit_t);
    if (!boxTest(t0,t1,cluster.bounds,org,dir))
      return;

    MeshSampler isec(self);
    int begin = primID * UMeshQC::clusterSize;
    int end   = min(begin+UMeshQC::clusterSize,self.numElements);
    
    auto &ray = owl::getPRD<Ray>();
    Random rand(ray.rngSeed++,primID);

    float t = t0;
    while (true) {
      float dt = - logf(1-rand())/(cluster.majorant);
      t += dt;
      if (t >= t1)
        break;

      isec.P = org+t*dir;
      if (!isec.sampleAndMap(begin,end))
        continue;

      bool accept = (isec.mapped.w > rand()*cluster.majorant);
      if (!accept)
        continue;

      MeshSampler dx0(self);
      MeshSampler dy0(self);
      MeshSampler dz0(self);
      MeshSampler dx1(self);
      MeshSampler dy1(self);
      MeshSampler dz1(self);
      const float delta = .1f;
      dx0.P = isec.P - delta * vec3f(1.f,0.f,0.f);
      dy0.P = isec.P - delta * vec3f(0.f,1.f,0.f);
      dz0.P = isec.P - delta * vec3f(0.f,0.f,1.f);
      dx1.P = isec.P + delta * vec3f(1.f,0.f,0.f);
      dy1.P = isec.P + delta * vec3f(0.f,1.f,0.f);
      dz1.P = isec.P + delta * vec3f(0.f,0.f,1.f);
      if (!dx0.sampleAndMap(begin,end))
        dx0 = isec;
      if (!dx1.sampleAndMap(begin,end))
        dx1 = isec;
      if (!dy0.sampleAndMap(begin,end))
        dy0 = isec;
      if (!dy1.sampleAndMap(begin,end))
        dy1 = isec;
      if (!dz0.sampleAndMap(begin,end))
        dz0 = isec;
      if (!dz1.sampleAndMap(begin,end))
        dz1 = isec;
      
      vec3f N;
      N.x = safeDiv(dx1.mapped.w-dx0.mapped.w,dx1.P.x - dx0.P.x);
      N.y = safeDiv(dy1.mapped.w-dy0.mapped.w,dy1.P.y - dy0.P.y);
      N.z = safeDiv(dz1.mapped.w-dz0.mapped.w,dz1.P.z - dz0.P.z);
      N = normalize
        ((N == vec3f(0.f)) ? dir : N);
      
      ray.hadHit = 1;
      ray.tMax   = t;
      ray.color
        = vec3f(isec.mapped.x,isec.mapped.y,isec.mapped.z)
        * (.3f+.7f*fabsf(dot(normalize(dir),normalize(N))));
      optixReportIntersection(t, 0);
      return;
    }
  }
  
}
