#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/unstructured/QuickClusters.h"
#include "barney/Context.h"

#if CLUSTERS_FROM_QC
# include "hilbert.h"
#else
# include "cuBQL/bvh.h"
#endif

namespace barney {

#if 0
  extern "C" char QuickClusters_ptx[];

  OWLGeomType UMeshQC::createGeomType(DevGroup *devGroup)
  {
    std::cout << OWL_TERMINAL_GREEN
              << "creating 'UMeshQC' geometry type"
              << OWL_TERMINAL_DEFAULT << std::endl;
    
    static OWLVarDecl params[]
      = {
         { "vertices", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.vertices) },
         { "tetIndices", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.tetIndices) },
         { "hexIndices", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.hexIndices) },
         { "elements", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.elements) },
         { "numElements", OWL_INT, OWL_OFFSETOF(DD,mesh.numElements) },
         { "clusters", OWL_BUFPTR, OWL_OFFSETOF(DD,clusters) },
         { "xf.values", OWL_BUFPTR, OWL_OFFSETOF(DD,xf.values) },
         { "xf.domain", OWL_FLOAT2, OWL_OFFSETOF(DD,xf.domain) },
         { "xf.baseDensity", OWL_FLOAT, OWL_OFFSETOF(DD,xf.baseDensity) },
         { "xf.numValues", OWL_INT, OWL_OFFSETOF(DD,xf.numValues) },
         { nullptr }
    };
    OWLModule module = owlModuleCreate
      (devGroup->owl,QuickClusters_ptx);
    OWLGeomType gt = owlGeomTypeCreate
      (devGroup->owl,OWL_GEOM_USER,sizeof(UMeshQC::DD),
       params,-1);
    owlGeomTypeSetBoundsProg(gt,module,"UMeshQCBounds");
    owlGeomTypeSetIntersectProg(gt,/*ray type*/0,module,"UMeshQCIsec");
    owlGeomTypeSetClosestHit(gt,/*ray type*/0,module,"UMeshQCCH");
    owlBuildPrograms(devGroup->owl);
    
    return gt;
  }
  
  UMeshQC::UMeshQC(DataGroup *owner,
                   std::vector<vec4f> &vertices,
                   std::vector<TetIndices> &tetIndices,
                   std::vector<PyrIndices> &pyrIndices,
                   std::vector<WedIndices> &wedIndices,
                   std::vector<HexIndices> &hexIndices)
    : UMeshField(owner,
                 vertices,
                 tetIndices,
                 pyrIndices,
                 wedIndices,
                 hexIndices)
  {
  }

#if CLUSTERS_FROM_QC
  uint64_t UMeshQC::encodeBox(const box4f &box4f)
  {
    box3f box((const vec3f&)box4f.lower,(const vec3f&)box4f.upper);
    
    int maxValue = (1<<numHilbertBits)-1;
    vec3f center = box.center();
    // PRINT(box);
    // PRINT(worldBounds);
    center
      = (center - getPos(worldBounds.lower))
      * rcp(max(vec3f(1e-10f),getPos(worldBounds.size())));
    // PRINT(center);
    center = clamp(center,vec3f(0.f),vec3f(1.f));
    vec3ul coords = vec3ul(center * maxValue);
    // PRINT(coords);
    bitmask_t _coords[3];
    _coords[0] = coords.x;
    _coords[1] = coords.y;
    _coords[2] = coords.z;
    return hilbert_c2i(3,numHilbertBits,_coords);
  }

  uint64_t UMeshQC::encodeTet(int primID)
  {
    const TetIndices indices = tetIndices[primID];
    return encodeBox(box4f()
                     .including(vertices[indices[0]])
                     .including(vertices[indices[1]])
                     .including(vertices[indices[2]])
                     .including(vertices[indices[3]]));
  }

  uint64_t UMeshQC::encodeHex(int primID)
  {
    const HexIndices indices = hexIndices[primID];
    return encodeBox(box4f()
                     .including(vertices[indices[0]])
                     .including(vertices[indices[1]])
                     .including(vertices[indices[2]])
                     .including(vertices[indices[3]])
                     .including(vertices[indices[4]])
                     .including(vertices[indices[5]])
                     .including(vertices[indices[6]])
                     .including(vertices[indices[7]]));
  }
#endif
    
  
  
  void UMeshQC::build(Volume *volume)
  {
    if (!geom) {
      std::cout << "UMeshQC: seems this is the first time we're building this thing ...." << std::endl;
      std::cout << "#bn.umesh: computing world bounds" << std::endl;
      worldBounds = box4f();
      for (int i=0;i<vertices.size();i++)
        worldBounds.extend((const vec4f&)vertices[i]);

      // ------------------------------------------------------------------
      std::cout << "#bn.umesh: uploading vertices and indices" << std::endl;
      PING; std::cout << "MEMORY LEAK!" << std::endl;
      OWLBuffer verticesBuffer
        = owlDeviceBufferCreate(getOWL(),
                                OWL_FLOAT4,
                                vertices.size(),
                                vertices.data());
      
      PING; std::cout << "MEMORY LEAK!" << std::endl;
      OWLBuffer tetIndicesBuffer
        = owlDeviceBufferCreate(getOWL(),
                                OWL_INT,
                                4*tetIndices.size(),
                                tetIndices.data());
      
      OWLBuffer hexIndicesBuffer
        = owlDeviceBufferCreate(getOWL(),
                                OWL_INT,
                                8*hexIndices.size(),
                                hexIndices.data());
      
      // ------------------------------------------------------------------
      std::cout << "umesh: creating list of elements" << std::endl;
      
#if CLUSTERS_FROM_QC
      std::cout << "umesh: building hilbert prims" << std::endl;
      std::vector<std::pair<uint64_t,uint32_t>> hilbertPrims
        (tetIndices.size()+hexIndices.size());
      owl::common::parallel_for_blocked
        (0,(int)tetIndices.size(),1024,
         [&](int begin, int end) {
           for (int i=begin;i<end;i++)
             hilbertPrims[i] = {encodeTet(i),(i<<3)|TET};
         });
      // for (int i=0;i<tetIndices.size();i++) 
      //   hilbertPrims[i] = {encodeTet(i),(i<<3)|TET};
      owl::common::parallel_for_blocked
        (0,(int)hexIndices.size(),1024,
         [&](int begin, int end) {
           for (int i=begin;i<end;i++)
             // for (int i=0;i<hexIndices.size();i++) 
             hilbertPrims[tetIndices.size()+i] = {encodeHex(i),(i<<3)|HEX};
         });
      std::cout << "umesh: sorting prims" << std::endl;
      std::sort(hilbertPrims.begin(),hilbertPrims.end());

      std::vector<Element> elements;
      for (auto prim : hilbertPrims) {
        Element elt;
        elt.ID = prim.second >> 3;
        elt.type = prim.second & 0x7;
        elements.push_back(elt);
      }
      PING; std::cout << "MEMORY LEAK!" << std::endl;
      OWLBuffer elementsBuffer
        = owlDeviceBufferCreate(getOWL(),
                                OWL_INT,
                                elements.size(),
                                elements.data());
      int numClusters = divRoundUp((int)elements.size(),clusterSize);

      PING; std::cout << "MEMORY LEAK!" << std::endl;
      OWLBuffer clustersBuffer
        = owlDeviceBufferCreate(getOWL(),
                                OWL_USER_TYPE(Cluster),
                                numClusters,nullptr);
#else
      std::vector<Element> elements;
      for (int i=0;i<tetIndices.size();i++) elements.push_back(Element(i, Element::TET));
      for (int i=0;i<hexIndices.size();i++) elements.push_back(Element(i, Element::HEX));

      PING; std::cout << "MEMORY LEAK!" << std::endl;
      OWLBuffer elementsBuffer
        = owlDeviceBufferCreate(getOWL(),
                                OWL_INT,
                                elements.size(),
                                elements.data());
      PRINT(elements.size());
      
      UMeshField::DD devMesh;
      devMesh.vertices    = (const float4*)owlBufferGetPointer(verticesBuffer,0);
      devMesh.tetIndices  = (const int4*)owlBufferGetPointer(tetIndicesBuffer,0);
      devMesh.hexIndices  = (const HexIndices*)owlBufferGetPointer(hexIndicesBuffer,0);
      devMesh.elements    = (const Element *)owlBufferGetPointer(elementsBuffer,0);
      devMesh.numElements = elements.size();

      cuBQL::BinaryBVH<float,3> bvh;
      cuBQL::BuildConfig buildConfig;
      box3f *d_primBounds = 0;
      PING;
      BARNEY_CUDA_CALL(MallocManaged(&d_primBounds,elements.size()*sizeof(box3f)));
      computeElementBounds<<<divRoundUp(devMesh.numElements,1024),1024>>>
        (d_primBounds,devMesh);

      PING;
      buildConfig.makeLeafThreshold = 8;
      static cuBQL::ManagedMemMemoryResource managedMem;
#if 1
      buildConfig.enableSAH();
#endif
      cuBQL::gpuBuilder(bvh,
                        (const cuBQL::box_t<float,3>*)d_primBounds,
                        (uint32_t)elements.size(),
                        buildConfig,
                        (hipStream_t)0,
                        managedMem);
      BARNEY_CUDA_SYNC_CHECK();
      PING;
      std::vector<Element> reorderedElements(devMesh.numElements);
      for (int i=0;i<devMesh.numElements;i++) {
        reorderedElements[i] = elements[bvh.primIDs[i]];
      }
      PING;
      owlBufferUpload(elementsBuffer,reorderedElements.data());
      PING;

      std::cout << "finding leaves to create clusters ..." << std::endl;
      std::vector<Cluster> h_clusters;
      PRINT(bvh.numNodes);
      for (int i=0;i<bvh.numNodes;i++) {
        auto node = bvh.nodes[i];
        if (node.count == 0) continue;
        Cluster c;
        c.begin = node.offset;
        c.end = node.offset + node.count;
        h_clusters.push_back(c);
      }

      int numClusters = h_clusters.size();

      PING; std::cout << "MEMORY LEAK!" << std::endl;
      OWLBuffer clustersBuffer
        = owlDeviceBufferCreate(getOWL(),
                                OWL_USER_TYPE(Cluster),
                                numClusters,h_clusters.data());
#endif
      
      OWLGeomType gt = owner->devGroup->getOrCreateGeomTypeFor
        ("UMeshQC",UMeshQC::createGeomType);
      geom = owlGeomCreate(getOWL(),gt);
      owlGeomSet1i(geom,"numElements",(int)elements.size());
      owlGeomSetBuffer(geom,"elements",elementsBuffer);
      owlGeomSetBuffer(geom,"vertices",verticesBuffer);
      owlGeomSetBuffer(geom,"tetIndices",tetIndicesBuffer);
      owlGeomSetBuffer(geom,"hexIndices",hexIndicesBuffer);


      // this is the first time we're building this!
      owlGeomSet1f(geom,"xf.baseDensity",0.f);
      owlGeomSet1i(geom,"xf.numValues",0);
      
      owlGeomSetBuffer(geom,"clusters",clustersBuffer);
      owlGeomSetPrimCount(geom,numClusters);

      OWLGroup group
        = owlUserGeomGroupCreate(getOWL(),1,&geom);
      std::cout << "performing initial accel build" << std::endl;
      owlGroupBuildAccel(group);
      volume->generatedGroups.push_back(group);
    }

    // std::cout << "updating volume transfer function" << std::endl;
    if (volume->xf.domain.lower < volume->xf.domain.upper) {
      owlGeomSet2f(geom,"xf.domain",volume->xf.domain.lower,volume->xf.domain.upper);
      // PRINT(volume->xf.domain);
    } else {
      owlGeomSet2f(geom,"xf.domain",worldBounds.lower.w,worldBounds.upper.w);
    }
    owlGeomSet1f(geom,"xf.baseDensity",volume->xf.baseDensity);
    PRINT(volume->xf.values.size());
    owlGeomSet1i(geom,"xf.numValues",(int)volume->xf.values.size());

    OWLBuffer xfValuesBuffer = volume->xf.valuesBuffer;
    owlGeomSetBuffer(geom,"xf.values",xfValuesBuffer);

    std::cout << "rebuilding volume accel after xf set" << std::endl;
    owlGroupBuildAccel(volume->generatedGroups[0]);
  }


#endif
}
