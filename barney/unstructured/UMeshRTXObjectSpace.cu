// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "UMeshRTXObjectSpace.h"
#include "cuBQL/bvh.h"

namespace barney {

  extern "C" char UMeshRTXObjectSpace_ptx[];
  
  OWLGeomType UMeshRTXObjectSpace::createGeomType(DevGroup *devGroup)
  {
    std::cout << OWL_TERMINAL_GREEN
              << "creating 'UMeshRTXObjectSpace' geometry type"
              << OWL_TERMINAL_DEFAULT << std::endl;
    
    static OWLVarDecl params[]
      = {
         { "mesh.worldBounds.lower", OWL_FLOAT4, OWL_OFFSETOF(DD,mesh.worldBounds.lower) },
         { "mesh.worldBounds.upper", OWL_FLOAT4, OWL_OFFSETOF(DD,mesh.worldBounds.upper) },
         { "mesh.vertices", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.vertices) },
         { "mesh.tetIndices", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.tetIndices) },
         { "mesh.pyrIndices", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.pyrIndices) },
         { "mesh.wedIndices", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.wedIndices) },
         { "mesh.hexIndices", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.hexIndices) },
         { "mesh.elements", OWL_BUFPTR, OWL_OFFSETOF(DD,mesh.elements) },
         { "mesh.numElements", OWL_INT, OWL_OFFSETOF(DD,mesh.numElements) },
         { "clusters", OWL_BUFPTR, OWL_OFFSETOF(DD,clusters) },
         { "xf.values", OWL_BUFPTR, OWL_OFFSETOF(DD,xf.values) },
         { "xf.domain", OWL_FLOAT2, OWL_OFFSETOF(DD,xf.domain) },
         { "xf.baseDensity", OWL_FLOAT, OWL_OFFSETOF(DD,xf.baseDensity) },
         { "xf.numValues", OWL_INT, OWL_OFFSETOF(DD,xf.numValues) },
         { nullptr }
    };
    OWLModule module = owlModuleCreate
      (devGroup->owl,UMeshRTXObjectSpace_ptx);
    OWLGeomType gt = owlGeomTypeCreate
      (devGroup->owl,OWL_GEOM_USER,sizeof(UMeshRTXObjectSpace::DD),
       params,-1);
    owlGeomTypeSetBoundsProg(gt,module,"UMeshRTXObjectSpaceBounds");
    owlGeomTypeSetIntersectProg(gt,/*ray type*/0,module,"UMeshRTXObjectSpaceIsec");
    owlGeomTypeSetClosestHit(gt,/*ray type*/0,module,"UMeshRTXObjectSpaceCH");
    owlBuildPrograms(devGroup->owl);
    
    return gt;
  }
  
  void UMeshRTXObjectSpace::createClusters()
  {
    assert(clusters.empty());
    assert(!clustersBuffer);

    SetActiveGPU forDuration(devGroup->devices[0]);
    // ==================================================================
    
    cuBQL::BinaryBVH<float,3> bvh;
    box3f *d_primBounds = 0;
    PING;
    BARNEY_CUDA_CALL(MallocManaged(&d_primBounds,mesh->elements.size()*sizeof(box3f)));
    
    auto d_mesh = mesh->getDD(0);
    computeElementBoundingBoxes
      <<<divRoundUp((int)mesh->elements.size(),1024),1024>>>
      (d_primBounds,d_mesh);
    
    cuBQL::BuildConfig buildConfig;
    buildConfig.makeLeafThreshold = 8;
    buildConfig.enableSAH();
    static cuBQL::ManagedMemMemoryResource managedMem;
    cuBQL::gpuBuilder(bvh,
                      (const cuBQL::box_t<float,3>*)d_primBounds,
                      (uint32_t)mesh->elements.size(),
                      buildConfig,
                      (hipStream_t)0,
                      managedMem);
    std::vector<Element> reorderedElements(mesh->elements.size());
    for (int i=0;i<mesh->elements.size();i++) {
      reorderedElements[i] = mesh->elements[bvh.primIDs[i]];
    }
    mesh->elements = reorderedElements;
    owlBufferUpload(mesh->elementsBuffer,reorderedElements.data());
    BARNEY_CUDA_CALL(Free(d_primBounds));

    for (int i=0;i<bvh.numNodes;i++) {
      auto node = bvh.nodes[i];
      if (node.count == 0) continue;
      Cluster c;
      c.begin = node.offset;
      c.end = node.offset + node.count;
      clusters.push_back(c);
    }
    cuBQL::free(bvh,0,managedMem);
    
    // ==================================================================

    clustersBuffer = owlDeviceBufferCreate(devGroup->owl,OWL_USER_TYPE(Cluster),
                                           clusters.size(),clusters.data());
  }
  
  void UMeshRTXObjectSpace::build()
  {
    BARNEY_CUDA_SYNC_CHECK();
    
    if (!group) {
      createClusters();
      
      std::string gtTypeName = "UMeshRTXObjectSpace";
      OWLGeomType gt = devGroup->getOrCreateGeomTypeFor
        (gtTypeName,createGeomType);
      geom
        = owlGeomCreate(devGroup->owl,gt);
      owlGeomSetPrimCount(geom,(int)clusters.size());

      // ------------------------------------------------------------------
      assert(mesh->tetIndicesBuffer);
      owlGeomSet4fv(geom,"mesh.worldBounds.lower",&mesh->worldBounds.lower.x);
      owlGeomSet4fv(geom,"mesh.worldBounds.upper",&mesh->worldBounds.upper.x);
      owlGeomSetBuffer(geom,"mesh.vertices",mesh->verticesBuffer);
      
      owlGeomSetBuffer(geom,"mesh.tetIndices",mesh->tetIndicesBuffer);
      owlGeomSetBuffer(geom,"mesh.pyrIndices",mesh->pyrIndicesBuffer);
      owlGeomSetBuffer(geom,"mesh.wedIndices",mesh->wedIndicesBuffer);
      owlGeomSetBuffer(geom,"mesh.hexIndices",mesh->hexIndicesBuffer);
      owlGeomSetBuffer(geom,"mesh.elements",mesh->elementsBuffer);
      
      // ------------------------------------------------------------------      
      owlGeomSetBuffer(geom,"clusters",clustersBuffer);
      
      // ------------------------------------------------------------------      
      
      if (volume->xf.domain.lower < volume->xf.domain.upper) {
        owlGeomSet2f(geom,"xf.domain",volume->xf.domain.lower,volume->xf.domain.upper);
      } else {
        owlGeomSet2f(geom,"xf.domain",mesh->worldBounds.lower.w,mesh->worldBounds.upper.w);
      }
      owlGeomSet1f(geom,"xf.baseDensity",volume->xf.baseDensity);
      owlGeomSet1i(geom,"xf.numValues",(int)volume->xf.values.size());
      // intentionally set to null for first-time build
      owlGeomSetBuffer(geom,"xf.values",0/*volume->xf.valuesBuffer*/);
      
      // ------------------------------------------------------------------      
      group
        = owlUserGeomGroupCreate(devGroup->owl,1,&geom,OPTIX_BUILD_FLAG_ALLOW_UPDATE);
      owlGroupBuildAccel(group);
      volume->generatedGroups.push_back(group);
    }
    
    if (volume->xf.domain.lower < volume->xf.domain.upper) {
      owlGeomSet2f(geom,"xf.domain",volume->xf.domain.lower,volume->xf.domain.upper);
    } else {
      owlGeomSet2f(geom,"xf.domain",mesh->worldBounds.lower.w,mesh->worldBounds.upper.w);
    }
    owlGeomSet1f(geom,"xf.baseDensity",volume->xf.baseDensity);
    owlGeomSet1i(geom,"xf.numValues",(int)volume->xf.values.size());
    owlGeomSetBuffer(geom,"xf.values",volume->xf.valuesBuffer);

    std::cout << "refitting ... umesh mc geom" << std::endl;
    owlGroupRefitAccel(volume->generatedGroups[0]);
  }


}

