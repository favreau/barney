#include "hip/hip_runtime.h"
// Copyright 2025 Ingo Wald
// SPDX-License-Identifier: Apache-2.0

#include "barney/MPIContext.h"
#include "barney/globalTrace/TwoStage.h"
#include "barney/DeviceGroup.h"
#include "barney/render/RayQueue.h"
#include "rtcore/ComputeInterface.h"

namespace BARNEY_NS {

  extern void (*profHook)();
  
  std::vector<std::tuple<double,double,int,const char *>> kernelTimes;
  
#define ENTER() const double prof_t0 = getCurrentTime();
#define LEAVE(count,name)                                   \
  const double prof_t1 = getCurrentTime();                    \
  kernelTimes.push_back(std::make_tuple<double,double,int,const char *>((double)prof_t0,(double)prof_t1,(int)count,(const char *)name));

  int prof_rank;

  void twoStageProfHook()
  {
    std::stringstream ss;
    static double t00 = std::get<0>(kernelTimes[0]);
    for (auto kernel : kernelTimes) {
      double t0 = std::get<0>(kernel)-t00;
      double t1 = std::get<1>(kernel)-t00;
      int numItems = std::get<2>(kernel);
      const char *name = std::get<3>(kernel);

      ss << "r" << prof_rank << " [" << prettyDouble(t0) << "s.."
         << prettyDouble(t1) << "s = "
         << prettyDouble(t1-t0) << "s]: "
         << prettyNumber(numItems) << " items in "
         << name
         << " -> " << prettyDouble(1000000.f*(t1-t0)/numItems) << "s per mio items"
         << std::endl;
    }
    kernelTimes.clear();
    std::cout << ss.str();
  }
  
  __rtc_global
  void buildHitsOnly(const rtc::ComputeInterface &ci,
                      HitOnly *hitOnly,
                      Ray *rayQueue,
                     int N);
  
  __rtc_global
  void reduceReceivedHitsKernel_intraNode(const rtc::ComputeInterface &ci,
                                          HitOnly *hitOnly,
                                          int nRays,
                                          int reduceFactor)
  {
    int tid = ci.launchIndex().x;
    if (tid >= nRays) return;

    HitOnly reduced = hitOnly[tid];
    for (int peer=1;peer<reduceFactor;peer++) {
      HitOnly *hit = hitOnly+peer*nRays+tid;
      
      if (hit->tHit >= reduced.tHit) continue;
      
      reduced = *hit;
    }
    hitOnly[tid] = reduced;
    
  }
  
  __rtc_global
  void reduceReceivedHitsKernel_crossNodes(const rtc::ComputeInterface &ci,
                                           Ray *rayQueueThisRank,
                                           HitOnly *hitOnlyAllRanks,
                                           int nRays,
                                           int reduceFactor)
  {
    int tid = ci.launchIndex().x;
    if (tid >= nRays) return;
    
    Ray ray = rayQueueThisRank[tid];
    for (int peer=0;peer<reduceFactor;peer++) {
      HitOnly *hit = hitOnlyAllRanks+peer*nRays+tid;

      if (hit->tHit >= ray.tMax) continue;

      ray.tMax     = hit->tHit;
      ray.bsdfType = hit->bsdfType;
      ray.hitBSDF  =  hit->hitBSDF;
      ray.P        =  hit->P;
      ray.N        =  hit->N;
    }
    rayQueueThisRank[tid] = ray;
  }
  
  
  __rtc_global
  void createRayOnly(const rtc::ComputeInterface &ci,
                     RayOnly *rayOnly,
                     Ray *rayQueue,
                     int N);
  __rtc_global
  void buildStagedRayQueue(const rtc::ComputeInterface &ci,
                           Ray *rayQueue,
                           RayOnly *rayOnly,
                           int N);
  
  TwoStage::TwoStage(MPIContext *context)
    : GlobalTraceImpl(context),
      context(context),
      world(context->world),
      topo(context->topo.get())
  {
    prof_rank = world.rank;
    profHook = twoStageProfHook;
    
    if (context->devices->size() != 1)
      throw std::runtime_error
        ("twostage all2all currently only works for one device per rank");
    this->device = context->devices->get(0);
    
    if (topo->islands.size() != 1)
      throw std::runtime_error
        ("twostage all2all currently only works for a single island");

    myGID = device->globalRank();
    numGlobal = topo->allDevices.size();
    rayCounts.resize(numGlobal);
    // sanity check that all physical nodes have same number of GPUs
    std::map<size_t,int> gpuCountInHost;
    int numHosts = 0;
    for (int gid=0; gid<context->topo->allDevices.size(); gid++) {
      auto &dev = context->topo->allDevices[gid];
      gpuCountInHost[dev.hostNameHash]++;
      numHosts = std::max(numHosts,topo->physicalHostIndexOf[gid]+1);
    }
    gpusPerHost = gpuCountInHost.begin()->second;
    for (auto count : gpuCountInHost)
      if (count.second != gpusPerHost)
        throw std::runtime_error
          ("twostage all2all currently requires same number of GPUs on all ranks");
    assert(numHosts * gpusPerHost == context->topo->allDevices.size());

#if 1
    this->hostIdx = topo->physicalHostIndexOf[myGID];
# if 1
    // allows oversubscription - we enumerate based on (host:process)
    // instead of (host.physialGPU)
    this->gpuIdx = topo->rankOnHost[myGID];
# else
    this->gpuIdx = topo->physicalGpuIndexOf[myGID];
# endif
    _rankOf.resize(numGlobal);

    std::vector<int> gidOfRank(numGlobal);
    world.allGather(gidOfRank.data(),&myGID,1);
    for (int r=0;r<numGlobal;r++)
      _rankOf[gidOfRank[r]] = r;
#else
    this->hostIdx = myGID / gpusPerHost;
    this->gpuIdx = myGID % gpusPerHost;
#endif
    this->numHosts = numGlobal / gpusPerHost;

    if (FromEnv::get()->logTopo) {
      world.barrier();
      if (context->myRank() == 0) {
        std::cout << "=========== TwoStage All2all ===========" << std::endl;
        std::cout << "- num MPI ranks (w/ one gpu each) " << numGlobal << std::endl;
        std::cout << "- detected num physical hosts " << numHosts << std::endl;
        std::cout << "- detected num (active) GPUs per host " << gpusPerHost << std::endl;
        for (int h=0;h<numHosts;h++)
          for (int g=0;g<gpusPerHost;g++) {
            std::cout << "- gpu on rank " << (rankOf(h,g))
                      << " is logical h" << h << "g" << g << " {"
                      << topo->toString(rankOf(h,g)) << "}" << std::endl;
          }
      }
      world.barrier();
    }
  }


  void TwoStage::ensureAllOurQueuesAreLargeEnough()
  {
    auto rtc = device->rtc;
    size_t ourRequiredQueueSize
      = device->rayQueue->size * numGlobal;
    if (ourRequiredQueueSize > currentReservedSize) {
      if (FromEnv::get()->logQueues) {
        std::cout << "resizing ray queues from " << currentReservedSize
                  << " to " << ourRequiredQueueSize << std::endl;
      }
      for (int i=0;i<2;i++)
        if (raysOnly[i]) rtc->freeMem(raysOnly[i]);
      for (int i=0;i<2;i++)
        if (hitsOnly[i]) rtc->freeMem(hitsOnly[i]);
      
      if (stagedRayQueue) rtc->freeMem(stagedRayQueue);
      
      size_t N = ourRequiredQueueSize+1024;
      for (int i=0;i<2;i++)
        raysOnly[i] = (RayOnly*)rtc->allocMem(N*sizeof(RayOnly));
      for (int i=0;i<2;i++)
        hitsOnly[i] = (HitOnly*)rtc->allocMem(N*sizeof(HitOnly));
      stagedRayQueue = (Ray *)rtc->allocMem(N*sizeof(Ray));
      
      currentReservedSize = N;
    }
  }

  // step 1: have all ranks exchange which (global) device has how
  // many rays (needed to set up the send/receives)
  void TwoStage::exchangeHowManyRaysEachDeviceHas()
  {
    ENTER();
    
    int myRayCount = device->rayQueue->numActive;
    // world.barrier();
    // BN_MPI_CALL(Alltoall(/* sendbuf */&myRayCount,
    //                      /* OUR count */1,MPI_INT,
    //                      /*recvbuf*/rayCounts.data(),
    //                      1,MPI_INT,
    //                      world.comm));
    // PING; world.barrier(); PING;
    world.allGather(rayCounts.data(),&myRayCount,1);

    
    if (FromEnv::get()->logQueues)  {
      if (myGID == 0) {
        std::cout << "ray counts (" << rayCounts.size() << "):";
        for (auto rc : rayCounts) std::cout << " " << rc;
        std::cout << std::endl;
      }
      // for (int i=0;i<numGlobal;i++) {
      //   world.barrier();
      //   if (myGID == i) {
      //     std::cout << "ray counts (" << rayCounts.size() << "):";
      //     for (auto rc : rayCounts) std::cout << " " << rc;
      //     std::cout << std::endl;
      //   }
      //   world.barrier();
      // }
    }
    // PING; world.barrier(); PING;
    LEAVE(1,"exchangeHowManyRaysEachDeviceHas");
  }
  
  
  /*! in this stage we have all each GPU exchange its rays with
    all GPUs that have same phsycail ID in all OTHER ranks, but NOT
    with other GPUs in same rank
  */
  void TwoStage::sendAndReceiveRays_crossNodes()
  {
    ENTER();
    // PING; world.barrier(); PING;
    
    // -----------------------------------------------------------------------------
    // first, create 'raysOnly[]' array, for each local device
    // -----------------------------------------------------------------------------
    int myRayCount = device->rayQueue->numActive;
    {
      SetActiveGPU forDuration(device);
      int bs = 128;
      int nb = divRoundUp(myRayCount,bs);
      __rtc_launch(device->rtc,
                   createRayOnly,
                   nb,bs,
                   // args
                   raysOnly[0],
                   device->rayQueue->traceAndShadeReadQueue.rays,
                   myRayCount);
    }

    device->rtc->sync();

    // world.barrier();
    std::vector<MPI_Request> requests;
    int recvOfs = 0;
    for (int h=0;h<numHosts;h++) {
      MPI_Request req;
      int recvCount = rayCounts[rankOf(h,gpuIdx)];
      if (FromEnv::get()->logQueues) 
        printf("splat-cross r%i receiving %i from %i (q 0->1)\n",
               myGID,recvCount,rankOf(h,gpuIdx));
      world.recv(rankOf(h,gpuIdx),0,raysOnly[1]+recvOfs,
                 recvCount,req);
      recvOfs += recvCount;
      requests.push_back(req);
    }
    intraNodes.numRaysReceived = recvOfs;
    if (FromEnv::get()->logQueues) 
      printf("splat-cross r%i total received %i\n",
             myGID,intraNodes.numRaysReceived);
    
    // world.barrier();
    for (int h=0;h<numHosts;h++) {
      MPI_Request req;
      if (FromEnv::get()->logQueues) 
        printf("splat-cross r%i sending %i to %i (q 0->1)\n",
               myGID,myRayCount,rankOf(h,gpuIdx));
      world.send(rankOf(h,gpuIdx),0,raysOnly[0],myRayCount,req);
      requests.push_back(req);
    }
    // world.barrier();
    
    BN_MPI_CALL(Waitall(requests.size(),requests.data(),MPI_STATUSES_IGNORE));
    requests.clear();
    // PING; world.barrier(); PING;
    LEAVE(recvOfs,"sendAndReceiveRays_crossNodes");
  }
  

  /*! in this stage we have all each GPU exchange its rays with
    all GPUs that have same phsycail ID in all OTHER ranks, but NOT
    with other GPUs in same rank
  */
  void TwoStage::sendAndReceiveRays_intraNode()
  {
    ENTER();
    // PING; world.barrier(); PING;

    std::vector<MPI_Request> requests;
    int recvOfs = 0;
    for (int g=0;g<gpusPerHost;g++) {
      MPI_Request req;
      int raysOnPeer = 0;
      for (int h=0;h<numHosts;h++)
        raysOnPeer += rayCounts[rankOf(h,g)];
      if (FromEnv::get()->logQueues) 
        printf("splat-intra r%i receiving %i from %i (q 1->0)\n",
               myGID,raysOnPeer,rankOf(hostIdx,g));
      world.recv(rankOf(hostIdx,g),0,
                 raysOnly[0]+recvOfs,raysOnPeer,
                 req);
      recvOfs += raysOnPeer;
      requests.push_back(req);
    }
    bothStages.numRaysReceived = recvOfs;
    if (FromEnv::get()->logQueues) 
      printf("splat-intra r%i total received %i\n",
             myGID,bothStages.numRaysReceived);

    // world.barrier();
    // PING; world.barrier(); PING;
    
    int numRaysWeHave = 0;
    for (int h=0;h<numHosts;h++)
      numRaysWeHave += rayCounts[rankOf(h,gpuIdx)];
    for (int g=0;g<gpusPerHost;g++) {
      MPI_Request req;
      if (FromEnv::get()->logQueues) 
        printf("splat-intra r%i sending %i to %i (q 1->0)\n",
               myGID,numRaysWeHave,rankOf(hostIdx,g));
      world.send(rankOf(hostIdx,g),0,
                 raysOnly[1],numRaysWeHave,
                 req);
      requests.push_back(req);
    }

    // world.barrier();
    // PING; world.barrier(); PING;
    
    BN_MPI_CALL(Waitall(requests.size(),requests.data(),MPI_STATUSES_IGNORE));
    requests.clear();
    // PING; world.barrier(); PING;
    LEAVE(recvOfs,"sendAndReceiveRays_intraNode");
  }

  

  void TwoStage::traceRays(GlobalModel *model,
                           uint32_t rngSeed,
                           bool needHitIDs) 
  {
    // std::cout << "==================================================================\n";
    // world.barrier();
    assert(needHitIDs == false); // not implemented right now
    // world.barrier();
    ensureAllOurQueuesAreLargeEnough();
    // world.barrier();
    exchangeHowManyRaysEachDeviceHas();
    // world.barrier();
    sendAndReceiveRays_crossNodes();
    // world.barrier();
    sendAndReceiveRays_intraNode();
    // world.barrier();

    traceAllReceivedRays(model,rngSeed,needHitIDs);
    // world.barrier();

    exchangeHits_intraNode();
     // world.barrier();
    reduceHits_intraNode();
    // world.barrier();
    exchangeHits_crossNodes();
    // world.barrier();
    reduceHits_crossNodes();
    // world.barrier();
  }


  void TwoStage::traceAllReceivedRays(GlobalModel *model,
                                      uint32_t rngSeed,
                                      bool needHitIDs)
  {
    // PING; world.barrier(); PING;
    
    SetActiveGPU forDuration(device);
    int numRaysWeHaveTotal = bothStages.numRaysReceived;
    {
      ENTER();
      if (FromEnv::get()->logQueues) 
        printf("buildlocalrays r%i total rays %i (q0)\n",
               myGID,numRaysWeHaveTotal);
      __rtc_launch(device->rtc,
                   buildStagedRayQueue,
                   divRoundUp(numRaysWeHaveTotal,1024),1024,
                   // args
                   stagedRayQueue,
                   raysOnly[0],
                   numRaysWeHaveTotal);
      
      device->rtc->sync();
      LEAVE(numRaysWeHaveTotal,"buildStagedRayQueue");
    }
    
    auto savedOriginalRayCount = device->rayQueue->numActive;
    auto savedOriginalRayQueue = device->rayQueue->traceAndShadeReadQueue.rays;
    device->rayQueue->traceAndShadeReadQueue.rays = stagedRayQueue;
    device->rayQueue->numActive = numRaysWeHaveTotal;

    {
      ENTER()
        if (FromEnv::get()->logQueues) 
          printf("localtrace r%i total rays %i\n",
                 myGID,numRaysWeHaveTotal);
      context->traceRaysLocally(model,rngSeed,needHitIDs);
      device->rtc->sync();
      LEAVE(numRaysWeHaveTotal,"localTrace");
    }
    
    if (FromEnv::get()->logQueues) 
      printf("buildhits r%i total rays %i (q0)\n",
             myGID,numRaysWeHaveTotal);
    {
      ENTER();
      __rtc_launch(device->rtc,
                   buildHitsOnly,
                   divRoundUp(numRaysWeHaveTotal,1024),1024,
                   // args
                   hitsOnly[0],
                   stagedRayQueue,
                   numRaysWeHaveTotal);
      device->rtc->sync();
      LEAVE(numRaysWeHaveTotal,"buildHitsOnly");
    }
    device->rayQueue->numActive = savedOriginalRayCount;
    device->rayQueue->traceAndShadeReadQueue.rays = savedOriginalRayQueue;

    // PING; world.barrier(); PING;
    
  }
  
  void TwoStage::exchangeHits_intraNode()
  {
    ENTER();
    std::vector<MPI_Request> requests;
    int recvOfs = 0;
    for (int g=0;g<gpusPerHost;g++) {
      MPI_Request req;
      int recvCount = 0;
      for (int h=0;h<numHosts;h++)
        recvCount += rayCounts[rankOf(h,gpuIdx)];
      world.recv(rankOf(hostIdx,g),0,
                 hitsOnly[1]+recvOfs,recvCount,req);
      if (FromEnv::get()->logQueues) 
        printf("xchg-intra r%i receiving %i from %i (q0->1)\n",
               myGID,recvCount,rankOf(hostIdx,g));
      requests.push_back(req);
      recvOfs += recvCount;
    }

    // and matching sends
    int sendOfs = 0;
    for (int g=0;g<gpusPerHost;g++) {
      int sendCount = 0;
      for (int h=0;h<numHosts;h++)
        sendCount += rayCounts[rankOf(h,g)];
      MPI_Request req;
      world.send(rankOf(hostIdx,g),0,
                 hitsOnly[0]+sendOfs,sendCount,req);
      if (FromEnv::get()->logQueues) 
        printf("xchg-intra r%i sending %i to %i (q0->1)\n",
               myGID,sendCount,rankOf(hostIdx,g));
      requests.push_back(req);
      sendOfs += sendCount;
    }
    BN_MPI_CALL(Waitall(requests.size(),requests.data(),MPI_STATUSES_IGNORE));
    requests.clear();
    LEAVE(recvOfs,"exchangeHits_intraNode");
  }


  void TwoStage::reduceHits_intraNode()
  {
    ENTER();
    SetActiveGPU forDuration(device);
    int g = gpuIdx;
    int numUniqueRaysThisGPU = 0;
    for (int h=0;h<numHosts;h++)
      numUniqueRaysThisGPU += rayCounts[rankOf(h,g)];

    if (FromEnv::get()->logQueues) 
      printf("r%i intra-reducing %i sets of %i hits (q1)\n",
             myGID,
             gpusPerHost,
             numUniqueRaysThisGPU);
    __rtc_launch(device->rtc,
                 reduceReceivedHitsKernel_intraNode,
                 divRoundUp(numUniqueRaysThisGPU,128),128,
                 // args
                 hitsOnly[1],
                 numUniqueRaysThisGPU,
                 gpusPerHost);
    device->rtc->sync();
    LEAVE(numUniqueRaysThisGPU*gpusPerHost,"reduceHits_intraNode");
  }
  
  void TwoStage::exchangeHits_crossNodes()
  {
    ENTER();
    std::vector<MPI_Request> requests;
    int recvOfs = 0;
    int recvCount = rayCounts[rankOf(hostIdx,gpuIdx)];
    for (int h=0;h<numHosts;h++) {
      MPI_Request req;

      if (FromEnv::get()->logQueues) 
        printf("xchg-intra r%i receiving %i from %i (q1->0)\n",
               myGID,recvCount,rankOf(h,gpuIdx));
      world.recv(rankOf(h,gpuIdx),0,
                 hitsOnly[0]+recvOfs,recvCount,req);
      requests.push_back(req);
      recvOfs += recvCount;
    }

    // and matching sends
    int sendOfs = 0;
    for (int h=0;h<numHosts;h++) {
      MPI_Request req;
      int sendCount = rayCounts[rankOf(h,gpuIdx)];
      if (FromEnv::get()->logQueues) 
        printf("xchg-intra r%i sending %i to %i (q1->0)\n",
               myGID,sendCount,rankOf(h,gpuIdx));
      world.send(rankOf(h,gpuIdx),0,
                 hitsOnly[1]+sendOfs,sendCount,req);
      requests.push_back(req);
      sendOfs += sendCount;
    }
    
    BN_MPI_CALL(Waitall(requests.size(),requests.data(),MPI_STATUSES_IGNORE));
    requests.clear();
    LEAVE(recvOfs,"exchangeHits_crossNodes");
  }
  
  void TwoStage::reduceHits_crossNodes()
  {
    ENTER();
    SetActiveGPU forDuration(device);
    int numUniqueRaysThisGPU = rayCounts[rankOf(hostIdx,gpuIdx)];
    if (FromEnv::get()->logQueues) 
      printf("r%i cross-reducing %i sets of %i hits (q0)\n",
             myGID,numHosts,numUniqueRaysThisGPU);
    __rtc_launch(device->rtc,
                 reduceReceivedHitsKernel_crossNodes,
                 divRoundUp(numUniqueRaysThisGPU,128),128,
                 // args
                 device->rayQueue->traceAndShadeReadQueue.rays,
                 hitsOnly[0],
                 numUniqueRaysThisGPU,
                 numHosts);
    device->rtc->sync();
    LEAVE(numUniqueRaysThisGPU*numHosts,"reduceHits_crossNodes");
  }
  
}
