#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/DeviceContext.h"
#include "barney/fb/FrameBuffer.h"
#include "barney/fb/TiledFB.h"
#include "barney/render/World.h"
#include "barney/render/Renderer.h"
#include "barney/GlobalModel.h"

namespace barney {
  namespace render {

#define MAX_DIFFUSE_BOUNCES 1
    
#define ENV_LIGHT_SAMPLING 1

#define USE_MIS 1
    
    // inline __device__ float abs(float f) { return fabsf(f); }
    
  inline __device__ float square(float f) { return f*f; }
  
    
    // enum { MAX_PATH_DEPTH = 10 };

    inline __device__
    float safe_eps(float f, vec3f v)
    {
      return max(f,1e-5f*reduce_max(abs(v)));
    }

    
    typedef enum {
      RENDER_MODE_UNDEFINED,
      RENDER_MODE_LOCAL,
      RENDER_MODE_AO,
      RENDER_MODE_PT
    } RenderMode;
  
    inline __device__
    vec3f randomDirection(Random &rng)
    {
      vec3f v;
      while (true) {
        v.x = 1.f-2.f*rng();
        v.y = 1.f-2.f*rng();
        v.z = 1.f-2.f*rng();
        if (dot(v,v) <= 1.f)
          return normalize(v);
      }
    }

    inline __device__
    bool sampleAreaLights(Light::Sample &ls,
                          const render::World::DD &world,
                          const vec3f P,
                          const vec3f N,
                          Random &random,
                          bool dbg)
    {
      if (world.numQuadLights == 0) return false;
      static const int RESERVOIR_SIZE = 8;
      int   lID[RESERVOIR_SIZE];
      float u[RESERVOIR_SIZE];
      float v[RESERVOIR_SIZE];
      float weights[RESERVOIR_SIZE];
      float sumWeights = 0.f;
      QuadLight::DD light;
      for (int i=0;i<RESERVOIR_SIZE;i++) {
        lID[i] = min(int(random()*world.numQuadLights),
                     world.numQuadLights-1);
        weights[i] = 0.f;
        light = world.quadLights[lID[i]];
        u[i] = random();
        v[i] = random();
        float lightArea = light.area;
        if (lightArea < 0.f)
          printf("INVALID NEGATIVE LIGHT AREA on light %i/%i : %f\n",
                 lID[i],world.numQuadLights,lightArea);
        vec3f LN = light.normal;
        vec3f LP = light.corner + u[i]*light.edge0 + v[i]*light.edge1;
        vec3f lightDir = LP - P;
        float lightDist = length(lightDir);
        if (lightDist < 1e-3f) continue;
      
        lightDir *= 1.f/lightDist;

        float weight = dot(lightDir,N);
        if (weight <= 1e-3f) continue;
        weight *= -dot(lightDir,LN);
        if (weight <= 1e-3f) continue;
        if (lightArea == 0.f || reduce_max(light.emission) == 0)
          printf("invalid light! %f : %f %f %f\n",
                 lightArea,
                 light.emission.x,
                 light.emission.y,
                 light.emission.z);
        weight *= (1.f/(lightDist*lightDist)) * lightArea * reduce_max(light.emission);
        if (isnan(sumWeights) || weight < 0.f)
          printf("area lights: weight[%i:%i] is nan or negative: dist  %f area %f emission %f %f %f\n",
                 i,lID[i],lightDist,lightArea,
                 light.emission.x,
                 light.emission.y,
                 light.emission.z);
        sumWeights += weight;
        weights[i] = weight;
      }
      if (isnan(sumWeights))
        printf("area lights: sumWeights is nan!\n");
      if (sumWeights == 0.f) return false;
      float r = random()*sumWeights;
      int i=0;
      while (i<RESERVOIR_SIZE && r >= weights[i]) { r-= weights[i]; ++i; }
      if (i == RESERVOIR_SIZE) return false;
    
      light = world.quadLights[lID[i]];
      vec3f LP = light.corner + u[i]*light.edge0 + v[i]*light.edge1;
      vec3f LD = LP-P;
      ls.direction
        = normalize(LD);
      ls.distance
        = length(LD);
      ls.radiance
        = light.emission
        * (light.area * -dot(light.normal,ls.direction)
           / square(ls.distance));
      ls.pdf
        = weights[i]/sumWeights
        * (float(RESERVOIR_SIZE)/float(world.numQuadLights));
      if (ls.pdf <= 0.f)
        printf("invalid area light PDF %f from i %i weight %f sum %f\n",
               ls.pdf,i,weights[i],sumWeights);
      return true;
    }

    inline __device__
    bool sampleDirLights(Light::Sample &ls,
                         const World::DD &world,
                         const Renderer::DD &renderer,
                         const vec3f P,
                         const vec3f N,
                         Random &random,
                         bool dbg)
    {
      if (world.numDirLights == 0) return false;
      static const int RESERVOIR_SIZE = 8;
      int   lID[RESERVOIR_SIZE];
      float weights[RESERVOIR_SIZE];
      float sumWeights = 0.f;
      DirLight::DD light;
    
      for (int i=0;i<RESERVOIR_SIZE;i++) {
        lID[i] = min(int(random()*world.numDirLights),
                     world.numDirLights-1);
        weights[i] = 0.f;
        light = world.dirLights[lID[i]];
        vec3f light_radiance
          = light.color
          * light.radiance;
        
        vec3f lightDir = -light.direction;
        float weight = dot(lightDir,N);
        if (1 && dbg) printf("light #%i, dir %f %f %f weight %f\n",lID[i],lightDir.x,lightDir.y,lightDir.z,weight);
        if (weight <= 1e-3f) continue;
        weight *= reduce_max(light_radiance);
        if (weight <= 1e-3f) continue;
        // if (0 && dbg) printf("radiance %f %f color %f %f %f weight %f\n",light.radiance.x,light.radiance.y,light.radiance.z,weight);
        weights[i] = weight;
        sumWeights += weight;
      }
      if (sumWeights == 0.f) return false;
      float r = random()*sumWeights;
      int i=0;
      while (i<RESERVOIR_SIZE && r >= weights[i]) { r-= weights[i]; ++i; }
      if (i == RESERVOIR_SIZE) return false;
    
      light = world.dirLights[lID[i]];
      ls.direction
        = -light.direction;
      ls.distance
        = INFINITY;
      ls.radiance
        = light.radiance;
      ls.pdf
        = weights[i]/sumWeights
        * (float(RESERVOIR_SIZE)/float(world.numDirLights));
      return weights[i] != 0.f;
    }

    inline __device__
    bool sampleEnvLight(Light::Sample &ls,
                        const World::DD &world,
                        const Renderer::DD &renderer,
                        const vec3f P,
                        const vec3f N,
                        Random &random,
                        bool dbg)
    {
      /* in barney, the environment is either a explicit hdri map (in
         EnvMapLight); or a uniform brightness of 'renderer.ambientRadiance' */
      if (world.envMapLight.texture)
        ls = world.envMapLight.sample(random,dbg);
      else {
        ls.direction = randomDirection(random);
        ls.radiance  = renderer.ambientRadiance;
        ls.pdf       = ONE_OVER_FOUR_PI;
        ls.distance  = INFINITY;
      }
      return true;
    }

    inline __device__
    bool sampleLights(Light::Sample &ls,
                      const World::DD &world,
                      const Renderer::DD &renderer,
                      const vec3f P,
                      const vec3f Ng,
                      Random &random,
#if USE_MIS
                      bool &lightNeedsMIS,
                      bool &lightIsDirLight,
#endif
                      bool dbg)
    {
#if USE_MIS
# if 1
      // huh ... not sure this is correct; setting this to true means
      // we'll always compute MIS weights for shadow and bounce ray as
      // if there was only an env-map light; even though we may
      // acutally have sampled a dir-light. that _may_ be true because
      // even if we did sample a dirlight there still _is_ a pdf for
      // the env-map light... but it's a bit iffy.
       lightNeedsMIS = true;
# else
      lightNeedsMIS = false;
# endif
#endif

#if ENV_LIGHT_SAMPLING
      Light::Sample els;
      float elsWeight
        = (sampleEnvLight(els,world,renderer,P,Ng,random,dbg)
           ? (reduce_max(els.radiance)/els.pdf)
           : 0.f);
        // = world.envMapLight.sample(random,dbg);
#else
      float elsWeight = 0.f;
#endif

      Light::Sample als;
      float alsWeight
        = (sampleAreaLights(als,world,P,Ng,random,dbg)
           ? (reduce_max(als.radiance)/als.pdf)
           : 0.f);
      Light::Sample dls;
      float dlsWeight
        = (sampleDirLights(dls,world,renderer,P,Ng,random,dbg)
           ? (reduce_max(dls.radiance)/dls.pdf)
           : 0.f);

      if (dbg) printf("sampling lights dls %f els %f\n",
                      dlsWeight,elsWeight);
      
      float sumWeights
        = alsWeight+dlsWeight+elsWeight;
      if (sumWeights == 0.f) return false;

      elsWeight *= 1.f/sumWeights;
      alsWeight *= 1.f/sumWeights;
      dlsWeight *= 1.f/sumWeights;
      
      float r = random();
      // if (dbg) printf("r %f els %f dls %f\n",r, elsWeight,dlsWeight);
      if (r <= alsWeight) {
        ls = als;
        ls.pdf *= alsWeight;
#if ENV_LIGHT_SAMPLING
      } else if (r <= alsWeight+elsWeight) {
        ls = els;
        ls.pdf *= elsWeight;
        if (dbg) printf(" ->  picked env light sample\n");
# if USE_MIS
        lightNeedsMIS = true;
# endif
#endif
      } else {
        if (dbg) printf(" ->  picked DIR light sample\n");
        ls = dls;
        ls.pdf *= dlsWeight;
# if USE_MIS
        lightIsDirLight = true;
# endif
      }
      // if (dbg)
      //   printf(" light weights %f %f\n",
      //          alsWeight,dlsWeight);
      if (isnan(ls.pdf) || (ls.pdf <= 0.f)) return false;
      
      return true;
    }




    inline __device__
    float schlick(float cosine,
                  float ref_idx)
    {
      float r0 = (1.0f - ref_idx) / (1.0f + ref_idx);
      r0 = r0 * r0;
      return r0 + (1.0f - r0)*powf((1.0f - cosine), 5.0f);
    }
  
  

    inline __device__
    bool refract(const vec3f& v,
                 const vec3f& n,
                 float ni_over_nt,
                 vec3f &refracted)
    {
      vec3f uv = normalize(v);
      float dt = dot(uv, n);
      float discriminant = 1.0f - ni_over_nt * ni_over_nt*(1 - dt * dt);
      if (discriminant > 0.f) {
        refracted = ni_over_nt * (uv - n * dt) - n * sqrtf(discriminant);
        return true;
      }
      else
        return false;
    }
  
    // inline __device__
    // vec3f reflect(const vec3f &v,
    //               const vec3f &n)
    // {
    //   return v - 2.0f*dot(v, n)*n;
    // }
  


    // inline __device__
    // bool scatter_glass(vec3f &scattered_direction,
    //                    Random &random,
    //                    // const vec3f &org,
    //                    const vec3f &dir,
    //                    // const vec3f &P,
    //                    vec3f N,
    //                    const float ior
    //                    // ,
    //                    // PerRayData &prd
    //                    )
    // {
    //   // const vec3f org   = optixGetWorldRayOrigin();
    //   // const vec3f dir   = normalize((vec3f)optixGetWorldRayDirection());

    //   // N = normalize(N);
    //   vec3f outward_normal;
    //   vec3f reflected = reflect(dir,N);
    //   float ni_over_nt;
    //   // prd.out.attenuation = vec3f(1.f, 1.f, 1.f); 
    //   vec3f refracted;
    //   float reflect_prob;
    //   float cosine;
  
    //   if (dot(dir,N) > 0.f) {
    //     outward_normal = -N;
    //     ni_over_nt = ior;
    //     cosine = dot(dir, N);// / vec3f(dir).length();
    //     cosine = sqrtf(1.f - ior*ior*(1.f-cosine*cosine));
    //   }
    //   else {
    //     outward_normal = N;
    //     ni_over_nt = 1.0 / ior;
    //     cosine = -dot(dir, N);// / vec3f(dir).length();
    //   }
    //   if (refract(dir, outward_normal, ni_over_nt, refracted)) 
    //     reflect_prob = schlick(cosine, ior);
    //   else 
    //     reflect_prob = 1.f;

    //   // prd.out.scattered_origin = P;
    //   if (random() < reflect_prob) 
    //     // prd.out.
    //     scattered_direction = reflected;
    //   else 
    //     // prd.out.
    //     scattered_direction = refracted;
  
    //   return true;
    // }


  
    inline __device__
    vec3f radianceFromEnv(const World::DD &world,
                          const Renderer::DD &renderer,
                          Ray &ray)
    {
      auto &env = world.envMapLight;
      if (env.texture) {
        vec3f d = xfmVector(env.toLocal,normalize(ray.dir));
        float theta = pbrtSphericalTheta(d);
        float phi   = pbrtSphericalPhi(d);
        const float invPi  = 1.f/M_PI;
        const float inv2Pi = 1.f/(2.f*M_PI);
        vec2f uv(phi * inv2Pi, theta * invPi);

        float4 color = tex2D<float4>(env.texture,uv.x,uv.y);
        float envLightPower = 1.f;
        return envLightPower*vec3f(color.x,color.y,color.z);
      } else {
        return renderer.ambientRadiance;
      }
    }

    /*! if there _is_ a dedicated env-map light specified, this looks
        up the background color from that map; otherwise, it returns
        the 'ray.misscolor' that the primary ray generation has set as
        default color for this ray */
    inline __device__
    vec3f primaryRayMissColor(const World::DD &world,
                          const Renderer::DD &renderer,
                          Ray &ray)
    {
      if (world.envMapLight.texture)
        return radianceFromEnv(world,renderer,ray);
      return
        // primary rays do store a default misscolor in the ray itself
        // - we simply return this if there's no env-map.
        ray.missColor;
    }

    /*! ugh - that should all go into material::AnariPhysical .... */
    template<int MAX_PATH_DEPTH>
    inline __device__
    void bounce(const World::DD &world,
                const Renderer::DD &renderer,
                vec3f &fragment,
                Ray &path,
                Ray &shadowRay,
                int pathDepth)
    {
      const float EPS = 1e-4f;

      const bool  hadNoIntersection  = !path.hadHit();
      const vec3f incomingThroughput = path.throughput;
      
      bool fire = path.dbg;//0 && (path.pixelID == 969722);
      // bool fire = 0 && (path.pixelID == 963212);
      // bool fire = 1 && (path.pixelID == 428428);

      if (fire || 0 && path.dbg)
        printf("(%i) ------------------------------------------------------------------\n -> incoming %f %f %f dir %f %f %f t %f\n  tp %f %f %f ismiss %i, bsdf %i\n",
               pathDepth,
               path.org.x,
               path.org.y,
               path.org.z,
               (float)path.dir.x,
               (float)path.dir.y,
               (float)path.dir.z,
               path.tMax,
               (float)path.throughput.x,
               (float)path.throughput.y,
               (float)path.throughput.z,
               int(hadNoIntersection),(int)path.bsdfType);
      
      if (path.isShadowRay) {
        // ==================================================================
        // shadow ray = all we have to do is add carried radiance if it
        // reached the light, and discards
        // ==================================================================
                 
        if (hadNoIntersection) {
          // fragment = clamp((vec3f)path.throughput,vec3f(0.f),vec3f(1.f));
          fragment =
# if USE_MIS
            path.misWeight *
#endif
            (vec3f)path.throughput;
          if (fire)
            printf("_shadow_ ray reaches light: tp %f %f %f misweight %f frag %f %f %f\n",
                   (float)path.throughput.x,
                   (float)path.throughput.y,
                   (float)path.throughput.z,
                   (float)path.misWeight,
                   fragment.x,
                   fragment.y,
                   fragment.z);
          if (0 && path.dbg) printf("shadow miss, frag %f %f %f\n",
                               fragment.x,
                               fragment.y,
                               fragment.z);
        }

        // this path is done.
        shadowRay.tMax = -1.f;
        path.tMax = -1.f;
        return;
      }

      vec3f Ng = path.getN();
      const bool  isVolumeHit        = (Ng == vec3f(0.f));
      if (!isVolumeHit)
        Ng = normalize(Ng);
      const bool  hitWasOnFront      = dot((vec3f)path.dir,Ng) < 0.f;
      vec3f Ngff = Ng;
      if (!hitWasOnFront)
        Ngff = - Ng;

      if (hadNoIntersection) {
        // ==================================================================
        // regular ray that did NOT hit ANYTHING 
        // ==================================================================
        if (pathDepth == 0) {
          // ----------------------------------------------------------------
          // PRIMARY ray that didn't hit anything -> background
          // ----------------------------------------------------------------
          // if (path.dbg)
          //   printf("miss primary %f %f %f\n",
          //          path.missColor.x,
          //          path.missColor.y,
          //          path.missColor.z);
          // fragment = path.missColor;
          fragment = primaryRayMissColor(world,renderer,path);
          // fragment = path.throughput * backgroundOrEnv(world,path);
          
          // const vec3f fromEnv
          //   = // 1.5f*
          //   backgroundOrEnv(world,path);

          // const vec3f tp = path.throughput;
          // const vec3f addtl = tp
          //   * fromEnv;
          // fragment = addtl;
        } else {
          // ----------------------------------------------------------------
          // SECONDARY ray that didn't hit anything -> env-light
          // ----------------------------------------------------------------
          // this path had at least one bounce, but now bounced into
          // nothingness - compute env-light contribution, and weigh it
          // with the path's carried throughput.
#if ENV_LIGHT_SAMPLING
# if USE_MIS
          const vec3f fromEnv = radianceFromEnv(world,renderer,path);
          fragment = path.throughput * fromEnv * path.misWeight;

          if (fire)
            printf("bounce ray hits env light: tp %f %f %f misweight %f fromEnv %f %f %f\n",
                   (float)path.throughput.x,
                   (float)path.throughput.y,
                   (float)path.throughput.z,
                   (float)path.misWeight,
                   fromEnv.x,
                   fromEnv.y,
                   fromEnv.z);
# else
          fragment = vec3f(0.f);
# endif
#else
          const vec3f fromEnv = radianceFromEnv(world,renderer,path);
          if (0 && path.dbg)
            printf("fromenv %f %f %f\n",
                   fromEnv.x,
                   fromEnv.y,
                   fromEnv.z);
          fragment = path.throughput * fromEnv;
#endif
        }
        // no outgoing rays; this path is done.
        path.tMax = -1.f;
        return;
      }
    

      // ==================================================================
      // this ray DID hit something: compute its local frame buffer
      // contribution at this hit point (if any), and generate secondary
      // ray and shadow ray (if applicable), with proper weights.
      // ==================================================================    
      Random &random = (Random &)path.rngSeed;
      const PackedBSDF bsdf = path.getBSDF();
      // bool doTransmission = false;
        // =  ((float)path.mini.transmission > 0.f)
        // && (random() < (float)path.mini.transmission);
      render::DG dg;
      dg.P  = path.P;
      dg.Ng = Ng;
      dg.Ns = Ng;
      dg.wo = -normalize((vec3f)path.dir);
      dg.insideMedium = path.isInMedium;
      // for volumes:
      // if (dg.Ng == vec3f(0.f))
      //   dg.Ng = dg.Ns = -path.dir;
      
      // if (1 && path.dbg)
      //   printf("dg.N %f %f %f\n",
      //          dg.Ns.x,
      //          dg.Ns.y,
      //          dg.Ns.z);

      // if the ray is a volume hit we want it offset it into the
      // direction the ray came from (otherwise we have a chance of
      // the shadow ray or boucne ray to terminate right where the
      // original ray ended; for others we want to offset based on
      // normal */
      const float offsetEpsilon = safe_eps(EPS,dg.P);
      vec3f frontFacingSurfaceOffset
        = offsetEpsilon*(isVolumeHit?dg.wo:Ngff);
      // vec3f dg_P
      //   = path.P+frontFacingSurfaceOffset;
// if (path.dbg)
      //   printf("(%i) hit trans %f ior %f, dotrans %i\n",
      //          pathDepth,
      //          (float)path.transmission,
      //          (float)path.ior,
      //          int(doTransmission));
// #if 1
      // if (path.dbg) printf("mattype %i\n",path.materialType);




      // ==================================================================
      // FIRST, let us look at generating any shadow rays, if
      // applicable; this way we can later modify the incoming ray in
      // place when we generate the outgoing ray.
      // ==================================================================
      Light::Sample ls;
      // todo check if BSDF is perfectly specular
#if USE_MIS
      bool lightNeedsMIS = false;
      bool lightIsDirLight = false;
#endif
      if (sampleLights(ls,world,renderer,dg.P,Ngff,random,
#if USE_MIS
                       lightNeedsMIS,
                       lightIsDirLight,
#endif
                       fire || 0 && path.dbg)
          // && 
          // (path.materialType != GLASS)
          ) {
        if (fire || 0 && path.dbg)
          printf("sample light dir %f %f %f rad %f %f %f pdf %f spike %f\n",
                 ls.direction.x,
                 ls.direction.y,
                 ls.direction.z,
                 ls.radiance.x,
                 ls.radiance.y,
                 ls.radiance.z,
                 ls.pdf,
                 reduce_max(ls.radiance)/ls.pdf);
        EvalRes f_r
          = bsdf.eval(dg,ls.direction,fire)
          // * fabsf(dot(dg.Ng,ls.direction))
          ;
        if (fire || 0 && path.dbg) printf("eval light res %f %f %f: %f\n",
                                  f_r.value.x,
                                  f_r.value.y,
                                  f_r.value.z,
                                  f_r.pdf);
        
        if (!f_r.valid() || reduce_max(f_r.value) < 1e-4f) {
          if (fire || 0 && path.dbg) printf(" no f_r, killing shadow ray\n");
          shadowRay.tMax = -1.f;
        } else {
          vec3f tp_sr
            = (incomingThroughput)
            * (1.f/ls.pdf)
            * f_r.value
            * ls.radiance
            * (isVolumeHit?1.f:fabsf(dot(dg.Ng,ls.direction)));
          if (fire) {
            printf(" -> inc tp %f %f %f, dot %f\n",
                   incomingThroughput.x,
                   incomingThroughput.y,
                   incomingThroughput.z,
                   (isVolumeHit?1.f:fabsf(dot(dg.Ng,ls.direction))));
            printf(" -> shadow f_r %f %f %f ls.rad %f %f %f pdf %f\n",
                   f_r.value.x,
                   f_r.value.y,
                   f_r.value.z,
                   ls.radiance.x,
                   ls.radiance.y,
                   ls.radiance.z,
                   ls.pdf);
            printf(" -> shadow ray tp %f %f %f\n",
                   tp_sr.x,
                   tp_sr.y,
                   tp_sr.z);
          }
          shadowRay.makeShadowRay
            (/* thrghhpt */tp_sr,
             /* surface: */dg.P + frontFacingSurfaceOffset,
             /* to light */ls.direction,
             /* length   */ls.distance * (1.f-2.f*offsetEpsilon));
          // if (path.dbg) printf("new shadow ray len %f %f\n",ls.dist,shadowRay.tMax);
          shadowRay.rngSeed = path.rngSeed + 1; random();
          shadowRay.dbg = path.dbg;
          shadowRay.pixelID = path.pixelID;

          shadowRay.misWeight = 1.f;
#if USE_MIS
          if (!lightIsDirLight && lightNeedsMIS) {
            float pdf_lightRay_lightDir
              = world.envMapLight.pdf(ls.direction);
            float pdf_scatterRay_lightDir
              = bsdf.pdf(dg,ls.direction);
            // compute MIS weight weight that shadow direction
            shadowRay.misWeight
              = pdf_lightRay_lightDir
              / (pdf_lightRay_lightDir + pdf_scatterRay_lightDir + 1e-10f);
            // and if it's too small for any reason, kill the shadow
            // ray
            if ((float)shadowRay.misWeight < 1e-5f)
              shadowRay.tMax  = -1.f;
          }
#endif
        }
      }




      // ==================================================================
      // now, let's decide what to do with the ray itself
      // ==================================================================
      path.tMax = -1.f;
      if (pathDepth >= MAX_PATH_DEPTH)
        return;
      
      ScatterResult scatterResult;
      bsdf.scatter(scatterResult,dg,random,fire || 0 && path.dbg);
      if (fire || 0 && path.dbg)
        printf("scatter result.valid ? %i\n",
               int(scatterResult.valid()));
      if (!scatterResult.valid() || scatterResult.pdf <= 1e-6f)
        return;
      
      bool isDiffuseBounce
        = scatterResult.wasDiffuse;
        //        = !isinf(scatterResult.pdf);
      if (isDiffuseBounce && (path.numDiffuseBounces+1)>MAX_DIFFUSE_BOUNCES) 
        return;
      
      path.numDiffuseBounces = path.numDiffuseBounces + 1;
      
      if (fire || 0 && path.dbg)
        printf("offsetting into sign %f, direction %f %f %f\n",
               scatterResult.offsetDirection,
               frontFacingSurfaceOffset.x,
               frontFacingSurfaceOffset.y,
               frontFacingSurfaceOffset.z);
      path.org
        = dg.P + scatterResult.offsetDirection * frontFacingSurfaceOffset;
      if (fire || 0 && path.dbg)
        printf("path scattered, bsdf in scatter dir is %f %f %f, pdf %f\n",
               (float)scatterResult.f_r.x, 
               (float)scatterResult.f_r.y, 
               (float)scatterResult.f_r.z,
               scatterResult.pdf);
      path.dir
        = normalize(scatterResult.dir);
      
      vec3f scatterFactor
        = scatterResult.f_r
        // ONE_PI *
        * (isVolumeHit?1.f:fabsf(dot(dg.Ng,path.dir)))
        / (isinf(scatterResult.pdf)? 1.f : (scatterResult.pdf + 1e-10f));
      path.throughput
        = path.throughput * scatterFactor;
      path.clearHit();
      if ((fire || 0 && path.dbg) && scatterResult.changedMedium)
        printf("path DID change medium\n");
      if (scatterResult.changedMedium)
        path.isInMedium = !path.isInMedium;
      
      if (0 && path.dbg)
        printf("scatter dir %f %f %f tp %f %f %f\n",
               (float)path.dir.x,
               (float)path.dir.y,
               (float)path.dir.z,
               (float)path.throughput.x,
               (float)path.throughput.y,
               (float)path.throughput.z);
      
      
#if USE_MIS
      if (lightNeedsMIS && !isinf(scatterResult.pdf)) {
        float pdf_scatterRay_scatterDir = bsdf.pdf(dg,path.dir);
        float pdf_lightRay_scatterDir   = world.envMapLight.pdf(path.dir);
        
        path.misWeight
          = pdf_scatterRay_scatterDir
          / (pdf_scatterRay_scatterDir + pdf_lightRay_scatterDir);
        // if (fire || 0 && path.dbg)
        //   printf("path mis %f shadow mis %f (tmax %f)\n",
        //          (float)path.misWeight,
        //          (float)shadowRay.misWeight,shadowRay.tMax);
        // }
      } else {
        path.misWeight = 1.f;
      }
#endif
    }
  

    template<int MAX_PATH_DEPTH>
    __global__
    void g_shadeRays_pt(World::DD world,
                        Renderer::DD renderer,
                        AccumTile *accumTiles,
                        int accumID,
                        Ray *readQueue,
                        int numRays,
                        Ray *writeQueue,
                        int *d_nextWritePos,
                        int generation)
    {
      int tid = threadIdx.x + blockIdx.x*blockDim.x;
      if (tid >= numRays) return;

      // pixel 105798 frag 8.085938 11.882812 18.906250
      // pixel 864686 frag 7.183594 10.132812 11.132812
      // resetting accumid
      
      Ray path = readQueue[tid];

      float alpha
        = (generation == 0)
        ? (path.hadHit()? 1.f : 0.f)
        : 0.f;
#if DENOISE
      vec3f incomingN
        = path.hadHit()
        ? path.getN()
        : vec3f(0.f);
      if (incomingN == vec3f(0.f))
        incomingN = vec3f(1.f,0.f,0.f);
#endif
      // what we'll add into the frame buffer
      vec3f fragment = 0.f;
      float z = path.tMax;
      // create a (potential) shadow ray, and init to 'invalid'
      Ray shadowRay;
      shadowRay.tMax = -1.f;
      
        // printf("sammpling dir for N %f %f %f\n",dg.N.x,dg.N.y,dg.N.z);

      // bounce that ray on the scene, possibly generating a) a fragment
      // to add to frame buffer; b) a outgoing ray (in-place
      // modification of 'path'); and/or c) a shadow ray
      bounce<MAX_PATH_DEPTH>(world,renderer,
                             fragment,
                             path,shadowRay,
                             generation);
    
      // write shadow and bounce ray(s), if any were generated
      // if (path.dbg)
      //   printf("path.tmax %f shadowray.tmax %f frag %f %f %f\n",
      //          path.tMax,shadowRay.tMax,
      //          fragment.x,fragment.y,fragment.z);
      if (shadowRay.tMax > 0.f) {
        writeQueue[atomicAdd(d_nextWritePos,1)] = shadowRay;
      }
      if (path.tMax > 0.f) {
        writeQueue[atomicAdd(d_nextWritePos,1)] = path;
      }

      // and write the shade fragment, if generated
      int tileID  = path.pixelID / pixelsPerTile;
      int tileOfs = path.pixelID % pixelsPerTile;
      float4 &valueToAccumInto
        = accumTiles[tileID].accum[tileOfs];

#if DENOISE
      vec3f &valueToAccumNormalInto
        = accumTiles[tileID].normal[tileOfs];
      // if (generation == 0)
      //   accumTiles[tileID].normal[tileOfs] = incomingN;
#endif
      
      // ==================================================================
      // add to accum buffer. be careful of two things:
      //
      // a) since each pixel could have two DIFFERENT rays in the
      // queue (shadow ray and bounce ray) we cannot simply 'add', but
      // have to use an atomic add, because these could be in the same
      // warp.
      //
      // b) since we don't have an explicit frame buffer clear we
      // still have to make sure each pixel is written - not added -
      // exactly once in the first generation of the first frame.
      // ==================================================================

      // clamping ...
      float clampMax = 10.f*(1+accumID);
      fragment = min(fragment,vec3f(clampMax));
      
      if (accumID == 0 && generation == 0) {
        // if (path.dbg) printf("init frag %f %f %f\n",fragment.x,fragment.y,fragment.z);
        valueToAccumInto = make_float4(fragment.x,fragment.y,fragment.z,alpha);
      } else {
        // if (path.dbg) printf("adding frag %f %f %f\n",fragment.x,fragment.y,fragment.z);
        if (generation == 0 && alpha) 
          atomicAdd(&valueToAccumInto.w,alpha);

        if (fragment.x > 0.f)
          atomicAdd(&valueToAccumInto.x,fragment.x);
        if (fragment.y > 0.f)
          atomicAdd(&valueToAccumInto.y,fragment.y);
        if (fragment.z > 0.f)
          atomicAdd(&valueToAccumInto.z,fragment.z);
#if DENOISE
        if (incomingN.x > 0.f)
          atomicAdd(&valueToAccumNormalInto.x,incomingN.x);
        if (incomingN.y > 0.f)
          atomicAdd(&valueToAccumNormalInto.y,incomingN.y);
        if (incomingN.z > 0.f)
          atomicAdd(&valueToAccumNormalInto.z,incomingN.z);
#endif
      }

      // and for apps that need a depth buffer, write z
      if (generation == 0) {
        float &tile_z = accumTiles[tileID].depth[tileOfs];
        if (accumID == 0) 
          tile_z = z;
        else
          tile_z = min(tile_z,z);
      }
    }
  }  

  using namespace render;
  
  void DeviceContext::shadeRays_launch(Renderer *renderer,
                                       GlobalModel *model,
                                       TiledFB *fb,
                                       int generation)
  {
    SetActiveGPU forDuration(device);
    int numRays = rays.numActive;
    int bs = 128;
    int nb = divRoundUp(numRays,bs);

    static RenderMode renderMode = RENDER_MODE_UNDEFINED;
    if (renderMode == RENDER_MODE_UNDEFINED) {
      const char *_fromEnv = getenv("BARNEY_RENDER");
      if (!_fromEnv)
        _fromEnv = "pt";
      const std::string mode = _fromEnv;
      if (mode == "AO" || mode == "ao")
        renderMode = RENDER_MODE_AO;
      else if (mode == "PT" || mode == "pt")
        renderMode = RENDER_MODE_PT;
      else if (mode == "local" || mode == "LOCAL")
        renderMode = RENDER_MODE_LOCAL;
      else
        throw std::runtime_error("unknown barney render mode '"+mode+"'");
    }

    DevGroup *dg = device->devGroup;
    World *world = model->getSlot(dg->lmsIdx)->world.get();

    if (nb) {
      World::DD devWorld
        = world->getDD(device);
      Renderer::DD devRenderer
        = renderer->getDD(device.get());
           
      switch(renderMode) {
#if 0
      case RENDER_MODE_LOCAL:
        g_shadeRays_pt<0>
          <<<nb,bs,0,device->launchStream>>>
          (world->getDD(device),
           fb->accumTiles,fb->owner->accumID,
           rays.traceAndShadeReadQueue,numRays,
           rays.receiveAndShadeWriteQueue,rays._d_nextWritePos,generation);
        break;
      case RENDER_MODE_AO:
        g_shadeRays_pt<1>
          <<<nb,bs,0,device->launchStream>>>
          (world->getDD(device),
           fb->accumTiles,fb->owner->accumID,
           rays.traceAndShadeReadQueue,numRays,
           rays.receiveAndShadeWriteQueue,rays._d_nextWritePos,generation);
        break;
      case RENDER_MODE_PT:
#else
      default:
#endif

#if 1
        CHECK_CUDA_LAUNCH(g_shadeRays_pt<12>,
                          nb,bs,0,device->launchStream,
                          /* args */
                          devWorld,devRenderer,
                          fb->accumTiles,fb->owner->accumID,
                          rays.traceAndShadeReadQueue,numRays,
                          rays.receiveAndShadeWriteQueue,
                          rays._d_nextWritePos,generation);
#else
        g_shadeRays_pt<12><<<nb,bs,0,device->launchStream>>>
          (devWorld,devRenderer,
         fb->accumTiles,fb->owner->accumID,
           rays.traceAndShadeReadQueue,numRays,
           rays.receiveAndShadeWriteQueue,rays._d_nextWritePos,generation);
#endif
        break;

      }
    }
  }

}
