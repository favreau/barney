#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "barney/LocalContext.h"
#include "barney/LocalFB.h"

namespace barney {

  LocalContext::LocalContext(const std::vector<int> &dataGroupIDs,
                             const std::vector<int> &gpuIDs)
    : Context(dataGroupIDs,gpuIDs)
  {
  }
  
  FrameBuffer *LocalContext::createFB(int owningRank) 
  {
    assert(owningRank == 0);
    return initReference(LocalFB::create(this));
  }

  __global__ void g_renderTiles(mori::AccumTile *tiles,
                                mori::TileDesc  *tileDescs,
                                int numTiles,
                                vec2i fbSize)
  {
    int tileID = blockIdx.x;
    vec2i tileOffset = tileDescs[tileID].lower;
    int ix = threadIdx.x + tileOffset.x;
    int iy = threadIdx.y + tileOffset.y;
    
    
    if (ix >= fbSize.x) return;
    if (iy >= fbSize.y) return;
    mori::AccumTile &tile = tiles[tileID];

    // int sx = 13*17;
    // int sy = 11*19;
    // float r = (ix % sx)/(sx-1.f);
    // float g = (iy % sy)/(sy-1.f);
    float r = ix / (fbSize.x-1.f);
    float g = iy / (fbSize.y-1.f);
    float b = 1.f - (ix+iy)/(fbSize.x+fbSize.y-1.f);

    bool dbg = (ix == 118 && iy == 123);

    tile.accum[threadIdx.y*mori::tileSize+threadIdx.x] = make_float4(r,g,b,1.f);
  }
  
  void renderTiles(Context *context,
                   int localID,
                   Model *model,
                   FrameBuffer *fb,
                   const BNCamera *camera)
  {
    auto &devFB = *fb->perGPU[localID];
    auto device = devFB.device;
    
    SetActiveGPU forDuration(device->gpuID);
    g_renderTiles
      <<<devFB.numActiveTiles,vec2i(mori::tileSize),0,device->stream>>>
      (devFB.accumTiles,
       devFB.tileDescs,
       devFB.numActiveTiles,
       devFB.numPixels);
  }
  
  void LocalContext::render(Model *model,
                            const BNCamera *camera,
                            FrameBuffer *fb)
  {
    // ------------------------------------------------------------------
    // tell each device to start rendering accum tiles
    // ------------------------------------------------------------------
    for (int localID = 0; localID < gpuIDs.size(); localID++) {
      auto &devFB = *fb->perGPU[localID];
      SetActiveGPU forDuration(devFB.device);
      renderTiles(this,localID,model,fb,camera);
    }
    
    // ------------------------------------------------------------------
    // tell each device to finalize its rendered accum tiles
    // ------------------------------------------------------------------
    for (int localID = 0; localID < gpuIDs.size(); localID++)
      // (will set active GPU internally)
      fb->perGPU[localID]->finalizeTiles();

    // ------------------------------------------------------------------
    // 
    // ------------------------------------------------------------------
    for (int localID = 0; localID < gpuIDs.size(); localID++) {
      auto &devFB = *fb->perGPU[localID];
      SetActiveGPU forDuration(devFB.device);
      mori::TiledFB::writeFinalPixels(fb->finalFB,
                                      fb->numPixels,
                                      devFB.finalTiles,
                                      devFB.tileDescs,
                                      devFB.numActiveTiles,
                                      devFB.device->stream);
    }
    
    for (int localID = 0; localID < gpuIDs.size(); localID++)
      fb->perGPU[localID]->sync();

    // ------------------------------------------------------------------
    // copy final frame buffer to app's frame buffer memory
    // ------------------------------------------------------------------
    if (fb->hostFB != fb->finalFB)
      MORI_CUDA_CALL(Memcpy(fb->hostFB,fb->finalFB,
                            fb->numPixels.x*fb->numPixels.y*sizeof(uint32_t),
                            hipMemcpyDefault));
    MORI_CUDA_SYNC_CHECK();
  }
  
}
